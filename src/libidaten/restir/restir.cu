#include "hip/hip_runtime.h"
#include <utility>

#include "restir/restir.h"

#include "aten4idaten.h"
#include "kernel/accelerator.cuh"
#include "kernel/device_scene_context.cuh"
#include "kernel/intersect.cuh"
#include "kernel/pt_common.h"

#include "cuda/cudadefs.h"
#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

#include "renderer/pathtracing/pathtracing_impl.h"
#include "renderer/restir/restir_impl.h"

__global__ void initReSTIRParameters(
    int32_t width, int32_t height,
    idaten::Reservoir* reservoirs,
    idaten::ReSTIRInfo* restir_infos)
{
    auto ix = blockIdx.x * blockDim.x + threadIdx.x;
    auto iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix >= width || iy >= height) {
        return;
    }

    const auto idx = getIdx(ix, iy, width);

    reservoirs[idx].clear();
    restir_infos[idx].clear();
}

__global__ void shade(
    idaten::Reservoir* reservoirs,
    idaten::ReSTIRInfo* restir_infos,
    float4* aovNormalDepth,
    float4* aovTexclrMeshid,
    aten::mat4 mtx_W2C,
    int32_t width, int32_t height,
    idaten::Path paths,
    const int32_t* __restrict__ hitindices,
    int32_t* hitnum,
    const aten::Intersection* __restrict__ isects,
    aten::ray* rays,
    int32_t sample,
    int32_t frame,
    int32_t bounce, int32_t rrBounce,
    idaten::context ctxt,
    const aten::ObjectParameter* __restrict__ shapes,
    const aten::MaterialParameter* __restrict__ mtrls,
    const aten::LightParameter* __restrict__ lights,
    const aten::TriangleParameter* __restrict__ prims,
    const aten::mat4* __restrict__ matrices,
    uint32_t* random)
{
    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= *hitnum) {
        return;
    }

    idx = hitindices[idx];

    ctxt.shapes = shapes;
    ctxt.mtrls = mtrls;
    ctxt.lights = lights;
    ctxt.prims = prims;
    ctxt.matrices = matrices;

    __shared__ aten::MaterialParameter shMtrls[64];

    const auto ray = rays[idx];

#if IDATEN_SAMPLER == IDATEN_SAMPLER_SOBOL
    auto scramble = random[idx] * 0x1fe3434f;
    paths.sampler[idx].init(frame + sample, 4 + bounce * 300, scramble);
#elif IDATEN_SAMPLER == IDATEN_SAMPLER_CMJ
    auto rnd = random[idx];
    auto scramble = rnd * 0x1fe3434f
        * (((frame + sample) + 331 * rnd) / (aten::CMJ::CMJ_DIM * aten::CMJ::CMJ_DIM));
    paths.sampler[idx].init(
        (frame + sample) % (aten::CMJ::CMJ_DIM * aten::CMJ::CMJ_DIM),
        4 + bounce * 300,
        scramble);
#endif

    aten::hitrecord rec;

    const auto& isect = isects[idx];

    const auto& obj = ctxt.GetObject(static_cast<uint32_t>(isect.objid));
    AT_NAME::evaluate_hit_result(rec, obj, ctxt, ray, isect);

    bool isBackfacing = dot(rec.normal, -ray.dir) < 0.0f;

    // 交差位置の法線.
    // 物体からのレイの入出を考慮.
    aten::vec3 orienting_normal = rec.normal;

    AT_NAME::FillMaterial(
        shMtrls[threadIdx.x],
        ctxt,
        rec.mtrlid,
        rec.isVoxel);

    auto albedo = AT_NAME::sampleTexture(shMtrls[threadIdx.x].albedoMap, rec.u, rec.v, aten::vec4(1), bounce);

    // Apply normal map.
    int32_t normalMap = shMtrls[threadIdx.x].normalMap;
    const auto pre_sampled_r = AT_NAME::material::applyNormal(
        &shMtrls[threadIdx.x],
        normalMap,
        orienting_normal, orienting_normal,
        rec.u, rec.v,
        ray.dir,
        &paths.sampler[idx]);

    if (!shMtrls[threadIdx.x].attrib.isTranslucent
        && !shMtrls[threadIdx.x].attrib.isEmissive
        && isBackfacing)
    {
        orienting_normal = -orienting_normal;
    }

    auto& restir_info = restir_infos[idx];
    {
        restir_info.clear();
        restir_info.nml = orienting_normal;
        restir_info.is_voxel = rec.isVoxel;
        restir_info.mtrl_idx = rec.mtrlid;
        restir_info.throughput = paths.throughput[idx].throughput;
        restir_info.wi = ray.dir;
        restir_info.u = rec.u;
        restir_info.v = rec.v;
        restir_info.p = rec.p;
        restir_info.pre_sampled_r = pre_sampled_r;
    }

    if (bounce == 0) {
        // Store AOV.
        int32_t ix = idx % width;
        int32_t iy = idx / width;

        const auto _idx = getIdx(ix, iy, width);

        // World coordinate to Clip coordinate.
        aten::vec4 pos = aten::vec4(rec.p, 1);
        pos = mtx_W2C.apply(pos);

        aovNormalDepth[_idx] = make_float4(orienting_normal.x, orienting_normal.y, orienting_normal.z, pos.w);
        aovTexclrMeshid[_idx] = make_float4(albedo.x, albedo.y, albedo.z, isect.mtrlid);
    }

    // Implicit conection to light.
    auto is_hit_implicit_light = AT_NAME::HitImplicitLight(
        isBackfacing,
        bounce,
        paths.contrib[idx], paths.attrib[idx], paths.throughput[idx],
        ray,
        rec.p, orienting_normal,
        rec.area,
        shMtrls[threadIdx.x]);
    if (is_hit_implicit_light) {
        return;
    }

    // Generate initial candidates.
    if (!(shMtrls[threadIdx.x].attrib.isSingular || shMtrls[threadIdx.x].attrib.isTranslucent))
    {
        auto& reservoir = reservoirs[idx];

        AT_NAME::restir::SampleLightByStreamingRIS(
            reservoir,
            shMtrls[threadIdx.x],
            ctxt,
            rec.p, orienting_normal,
            ray.dir,
            rec.u, rec.v,
            &paths.sampler[idx],
            bounce);
    }

    const auto russianProb = AT_NAME::ComputeRussianProbability(
        bounce, rrBounce,
        paths.attrib[idx],
        paths.throughput[idx],
        paths.sampler[idx]);

    AT_NAME::MaterialSampling sampling;

    AT_NAME::material::sampleMaterialWithExternalAlbedo(
        &sampling,
        &shMtrls[threadIdx.x],
        orienting_normal,
        ray.dir,
        rec.normal,
        &paths.sampler[idx],
        pre_sampled_r,
        rec.u, rec.v,
        albedo);

    auto nextDir = normalize(sampling.dir);
    auto pdfb = sampling.pdf;
    auto bsdf = sampling.bsdf;

    // Get normal to add ray offset.
    // In refraction material case, new ray direction might be computed with inverted normal.
    // For example, when a ray go into the refraction surface, inverted normal is used to compute new ray direction.
    auto rayBasedNormal = (!isBackfacing && shMtrls[threadIdx.x].attrib.isTranslucent)
        ? -orienting_normal
        : orienting_normal;

    auto c = dot(orienting_normal, nextDir);

    if (pdfb > 0 && c > 0) {
        paths.throughput[idx].throughput *= bsdf * c / pdfb;
        paths.throughput[idx].throughput /= russianProb;
    }
    else {
        paths.attrib[idx].isTerminate = true;
        return;
    }

    // Make next ray.
    rays[idx] = aten::ray(rec.p, nextDir, rayBasedNormal);

    paths.throughput[idx].pdfb = pdfb;
    paths.attrib[idx].isSingular = shMtrls[threadIdx.x].attrib.isSingular;
    paths.attrib[idx].mtrlType = shMtrls[threadIdx.x].type;
}

__global__ void EvaluateVisibility(
    int32_t bounce,
    int32_t width, int32_t height,
    idaten::Path paths,
    int32_t* hitindices,
    int32_t* hitnum,
    idaten::Reservoir* reservoirs,
    const idaten::ReSTIRInfo* __restrict__ restir_infos,
    idaten::ShadowRay* shadowRays,
    idaten::context ctxt,
    const aten::ObjectParameter* __restrict__ shapes,
    const aten::MaterialParameter* __restrict__ mtrls,
    const aten::LightParameter* __restrict__ lights,
    const aten::TriangleParameter* __restrict__ prims,
    const aten::mat4* __restrict__ matrices)
{
    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= *hitnum) {
        return;
    }

    ctxt.shapes = shapes;
    ctxt.mtrls = mtrls;
    ctxt.lights = lights;
    ctxt.prims = prims;
    ctxt.matrices = matrices;

    idx = hitindices[idx];

    const auto size = width * height;

    aten::span reservoirs_as_span(reservoirs, size);
    aten::const_span resitr_infos_as_span(restir_infos, size);
    aten::span shadow_rays(shadowRays, size);

    AT_NAME::restir::EvaluateVisibility(
        idx,
        bounce,
        paths,
        ctxt,
        reservoirs_as_span,
        resitr_infos_as_span,
        shadow_rays);
}

__global__ void ComputePixelColor(
    const idaten::Reservoir* __restrict__ reservoirs,
    const idaten::ReSTIRInfo* __restrict__ restir_infos,
    idaten::Path paths,
    int32_t* hitindices,
    int32_t* hitnum,
    const float4* __restrict__ aovTexclrMeshid,
    const aten::LightParameter* __restrict__ lights, int32_t lightnum,
    const aten::MaterialParameter* __restrict__ mtrls,
    hipTextureObject_t* textures,
    const idaten::ShadowRay* __restrict__ shadowRays)
{
    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= *hitnum) {
        return;
    }

    idx = hitindices[idx];

    if (paths.attrib[idx].isTerminate) {
        return;
    }

    if (lightnum <= 0) {
        return;
    }

    idaten::context ctxt;
    {
        ctxt.mtrls = mtrls;
        ctxt.textures = textures;
    }

    __shared__ aten::MaterialParameter shMtrls[64];

    const auto& reservoir = reservoirs[idx];
    const auto& restir_info = restir_infos[idx];

    AT_NAME::FillMaterial(
        shMtrls[threadIdx.x],
        ctxt,
        restir_info.mtrl_idx,
        restir_info.is_voxel);

    aten::const_span lights_as_span(lights, lightnum);

    auto contrib = AT_NAME::restir::ComputeContribution(
        reservoir, restir_info,
        shMtrls[threadIdx.x],
        aovTexclrMeshid[idx],
        shadowRays[idx],
        lights_as_span);
    if (contrib) {
        const auto pixel_color = contrib.value() * paths.throughput[idx].throughput;
        paths.contrib[idx].contrib += make_float3(pixel_color.x, pixel_color.y, pixel_color.z);
    }
}

namespace idaten
{
    void ReSTIRPathTracing::initReSTIR(int32_t width, int32_t height)
    {
        dim3 block(BLOCK_SIZE, BLOCK_SIZE);
        dim3 grid(
            (width + block.x - 1) / block.x,
            (height + block.y - 1) / block.y);

        initReSTIRParameters << < grid, block, 0, m_stream >> > (
            width, height,
            m_reservoirs[m_curReservoirPos].data(),
            m_restir_infos.data());

        checkCudaKernel(initReSTIRParameters);
    }

    void ReSTIRPathTracing::onShadeReSTIR(
        hipSurfaceObject_t outputSurf,
        int32_t width, int32_t height,
        int32_t sample,
        int32_t bounce, int32_t rrBounce)
    {
        m_mtx_W2V.lookat(
            m_cam.origin,
            m_cam.center,
            m_cam.up);

        m_mtx_V2C.perspective(
            m_cam.znear,
            m_cam.zfar,
            m_cam.vfov,
            m_cam.aspect);

        m_mtx_C2V = m_mtx_V2C;
        m_mtx_C2V.invert();

        m_mtx_V2W = m_mtx_W2V;
        m_mtx_V2W.invert();

        aten::mat4 mtx_W2C = m_mtx_V2C * m_mtx_W2V;

        dim3 blockPerGrid(((width * height) + 64 - 1) / 64);
        dim3 threadPerBlock(64);

        auto& hitcount = m_compaction.getCount();

        shade << <blockPerGrid, threadPerBlock, 0, m_stream >> > (
            m_reservoirs[m_curReservoirPos].data(),
            m_restir_infos.data(),
            aov_.normal_depth().data(),
            aov_.albedo_meshid().data(),
            mtx_W2C,
            width, height,
            path_host_->paths,
            m_hitidx.data(), hitcount.data(),
            m_isects.data(),
            m_rays.data(),
            sample,
            m_frame,
            bounce, rrBounce,
            ctxt_host_.ctxt,
            ctxt_host_.shapeparam.data(),
            ctxt_host_.mtrlparam.data(),
            ctxt_host_.lightparam.data(),
            ctxt_host_.primparams.data(),
            ctxt_host_.mtxparams.data(),
            m_random.data());

        checkCudaKernel(shade);

        onShadeByShadowRayReSTIR(
            width, height,
            bounce);
    }

    void ReSTIRPathTracing::onShadeByShadowRayReSTIR(
        int32_t width, int32_t height,
        int32_t bounce)
    {
        dim3 blockPerGrid(((width * height) + 64 - 1) / 64);
        dim3 threadPerBlock(64);

        auto& hitcount = m_compaction.getCount();

        EvaluateVisibility << <blockPerGrid, threadPerBlock, 0, m_stream >> > (
            bounce,
            width, height,
            path_host_->paths,
            m_hitidx.data(), hitcount.data(),
            m_reservoirs[m_curReservoirPos].data(),
            m_restir_infos.data(),
            m_shadowRays.data(),
            ctxt_host_.ctxt,
            ctxt_host_.shapeparam.data(),
            ctxt_host_.mtrlparam.data(),
            ctxt_host_.lightparam.data(),
            ctxt_host_.primparams.data(),
            ctxt_host_.mtxparams.data());

        checkCudaKernel(EvaluateVisibility);

        const auto target_idx = computelReuse(
            width, height,
            bounce);

        ComputePixelColor << <blockPerGrid, threadPerBlock, 0, m_stream >> > (
            m_reservoirs[target_idx].data(),
            m_restir_infos.data(),
            path_host_->paths,
            m_hitidx.data(), hitcount.data(),
            aov_.albedo_meshid().data(),
            ctxt_host_.lightparam.data(), ctxt_host_.lightparam.num(),
            ctxt_host_.mtrlparam.data(),
            ctxt_host_.tex.data(),
            m_shadowRays.data());

        checkCudaKernel(ComputePixelColor);
    }
}
