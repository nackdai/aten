#include "hip/hip_runtime.h"
#include "svgf/svgf.h"

#include "kernel/pt_common.h"

#include "cuda/cudadefs.h"
#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

#include "aten4idaten.h"
#include "renderer/svgf/svgf_impl.h"

inline __device__ void computePrevScreenPos(
    int32_t ix, int32_t iy,
    float centerDepth,
    int32_t width, int32_t height,
    aten::vec4* prevPos,
    const aten::mat4* __restrict__ mtxs)
{
    // NOTE
    // Pview = (Xview, Yview, Zview, 1)
    // mtx_V2C = W 0 0  0
    //          0 H 0  0
    //          0 0 A  B
    //          0 0 -1 0
    // mtx_V2C * Pview = (Xclip, Yclip, Zclip, Wclip) = (Xclip, Yclip, Zclip, Zview)
    //  Wclip = Zview = depth
    // Xscr = Xclip / Wclip = Xclip / Zview = Xclip / depth
    // Yscr = Yclip / Wclip = Yclip / Zview = Yclip / depth
    //
    // Xscr * depth = Xclip
    // Xview = mtx_C2V * Xclip

    const aten::mat4 mtx_C2V = mtxs[0];
    const aten::mat4 mtx_V2W = mtxs[1];
    const aten::mat4 mtx_prev_W2V = mtxs[2];
    const aten::mat4 mtx_V2C = mtxs[3];

    float2 uv = make_float2(ix + 0.5, iy + 0.5);
    uv /= make_float2(width - 1, height - 1);    // [0, 1]
    uv = uv * 2.0f - 1.0f;    // [0, 1] -> [-1, 1]

    aten::vec4 pos(uv.x, uv.y, 0, 0);

    // Screen-space -> Clip-space.
    pos.x *= centerDepth;
    pos.y *= centerDepth;

    // Clip-space -> View-space
    pos = mtx_C2V.apply(pos);
    pos.z = -centerDepth;
    pos.w = 1.0;

    pos = mtx_V2W.apply(pos);

    // Reproject previous screen position.
    pos = mtx_prev_W2V.apply(pos);
    *prevPos = mtx_V2C.apply(pos);
    *prevPos /= prevPos->w;

    *prevPos = *prevPos * 0.5 + 0.5;    // [-1, 1] -> [0, 1]
}

__global__ void temporalReprojection(
    const float threshold_normal,
    const float threshold_depth,
    const float4* __restrict__ contributes,
    const aten::CameraParameter camera,
    float4* curAovNormalDepth,
    float4* curAovTexclrMeshid,
    float4* curAovColorVariance,
    float4* curAovMomentTemporalWeight,
    const float4* __restrict__ prevAovNormalDepth,
    const float4* __restrict__ prevAovTexclrMeshid,
    const float4* __restrict__ prevAovColorVariance,
    const float4* __restrict__ prevAovMomentTemporalWeight,
    hipSurfaceObject_t motion_detph_buffer,
    hipSurfaceObject_t dst,
    int32_t width, int32_t height)
{
    int32_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix >= width || iy >= height) {
        return;
    }

    const auto idx = getIdx(ix, iy, width);

    const size_t size = width * height;

    aten::const_span contribs(contributes, size);
    aten::span curr_aov_normal_depth(curAovNormalDepth, size);
    aten::span curr_aov_texclr_meshid(curAovTexclrMeshid, size);
    aten::span curr_aov_color_variance(curAovColorVariance, size);
    aten::span curr_aov_moment_temporalweight(curAovMomentTemporalWeight, size);
    aten::const_span prev_aov_normal_depth(prevAovNormalDepth, size);
    aten::const_span prev_aov_texclr_meshid(prevAovTexclrMeshid, size);
    aten::const_span prev_aov_color_variance(prevAovColorVariance, size);
    aten::const_span prev_aov_moment_temporalweight(prevAovMomentTemporalWeight, size);

    auto extracted_center_pixel = AT_NAME::svgf::ExtractCenterPixel(
        idx,
        contribs,
        curr_aov_normal_depth,
        curr_aov_texclr_meshid);

    const auto center_meshid{ aten::get<2>(extracted_center_pixel) };
    auto curr_color{ aten::get<3>(extracted_center_pixel) };

    auto back_ground_pixel_clr = AT_NAME::svgf::UpdateAOVIfBackgroundPixel(
        idx, curr_color, center_meshid,
        curr_aov_color_variance, curr_aov_moment_temporalweight);
    if (back_ground_pixel_clr) {
        surf2Dwrite(
            back_ground_pixel_clr.value(),
            dst,
            ix * sizeof(float4), iy,
            hipBoundaryModeTrap);
        return;
    }

    const auto center_normal{ aten::get<0>(extracted_center_pixel) };
    const float center_depth{ aten::get<1>(extracted_center_pixel) };

    float weight = 0;

    aten::tie(weight, curr_color) = AT_NAME::svgf::TemporalReprojection(
        ix, iy, width, height,
        threshold_normal, threshold_depth,
        center_normal, center_depth, center_meshid,
        curr_color,
        curr_aov_color_variance, curr_aov_moment_temporalweight,
        prev_aov_normal_depth, prev_aov_texclr_meshid, prev_aov_color_variance,
        motion_detph_buffer);

    AT_NAME::svgf::AccumulateMoments(
        idx, weight,
        aten::const_span<float4>(curr_aov_color_variance),
        curr_aov_moment_temporalweight,
        prev_aov_moment_temporalweight);

    surf2Dwrite(
        curr_color,
        dst,
        ix * sizeof(float4), iy,
        hipBoundaryModeTrap);
}

__global__ void RecomputeTemporalWeightFromSurroundingPixels(
    float4* aovMomentTemporalWeight,
    const float4* __restrict__ aovTexclrMeshid,
    int32_t width, int32_t height)
{
    int32_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix >= width || iy >= height) {
        return;
    }

    auto idx = getIdx(ix, iy, width);

    const size_t size = width * height;

    auto aov_texclr_meshid{ aten::const_span<float4>(aovTexclrMeshid, size) };
    auto aov_moment_temporalweight{ aten::const_span<float4>(aovMomentTemporalWeight, size) };

    auto weight = AT_NAME::svgf::RecomputeTemporalWeightFromSurroundingPixels(
        ix, iy, width, height,
        aov_texclr_meshid,
        aov_moment_temporalweight);

    if (weight) {
        aovMomentTemporalWeight[idx].w = weight.value();
    }
}

namespace idaten
{
    void SVGFPathTracing::onTemporalReprojection(
        hipSurfaceObject_t outputSurf,
        int32_t width, int32_t height)
    {
        dim3 block(BLOCK_SIZE, BLOCK_SIZE);
        dim3 grid(
            (width + block.x - 1) / block.x,
            (height + block.y - 1) / block.y);

        auto& curaov = params_.GetCurrAovBuffer();
        auto& prevaov = params_.GetPrevAovBuffer();

        CudaGLResourceMapper<decltype(params_.motion_depth_buffer)> rscmap(params_.motion_depth_buffer);
        auto motionDepthBuffer = params_.motion_depth_buffer.bind();

        temporalReprojection << <grid, block, 0, m_stream >> > (
        //temporalReprojection << <1, 1 >> > (
            m_nmlThresholdTF,
            m_depthThresholdTF,
            params_.temporary_color_buffer.data(),
            m_cam,
            curaov.get<AT_NAME::SVGFAovBufferType::NormalDepth>().data(),
            curaov.get<AT_NAME::SVGFAovBufferType::AlbedoMeshId>().data(),
            curaov.get<AT_NAME::SVGFAovBufferType::ColorVariance>().data(),
            curaov.get<AT_NAME::SVGFAovBufferType::MomentTemporalWeight>().data(),
            prevaov.get<AT_NAME::SVGFAovBufferType::NormalDepth>().data(),
            prevaov.get<AT_NAME::SVGFAovBufferType::AlbedoMeshId>().data(),
            prevaov.get<AT_NAME::SVGFAovBufferType::ColorVariance>().data(),
            prevaov.get<AT_NAME::SVGFAovBufferType::MomentTemporalWeight>().data(),
            motionDepthBuffer,
            outputSurf,
            width, height);

        checkCudaKernel(temporalReprojection);

        RecomputeTemporalWeightFromSurroundingPixels << <grid, block, 0, m_stream >> > (
            curaov.get<AT_NAME::SVGFAovBufferType::MomentTemporalWeight>().data(),
            curaov.get<AT_NAME::SVGFAovBufferType::AlbedoMeshId>().data(),
            width, height);
        checkCudaKernel(dilateWeight);
    }
}
