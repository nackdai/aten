#include "hip/hip_runtime.h"
#include "kernel/renderer.h"
#include "kernel/device_scene_context.cuh"
#include "kernel/pt_common.h"

#include <hip/hip_runtime.h>
#include <>

#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

#include "aten4idaten.h"

__global__ void textureViewer(
    uint32_t texIdx,
    int32_t width, int32_t height,
    hipTextureObject_t* textures,
    hipSurfaceObject_t outSurface)
{
    const auto ix = blockIdx.x * blockDim.x + threadIdx.x;
    const auto iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix >= width || iy >= height) {
        return;
    }

    float u = ix / (float)width;
    float v = iy / (float)height;

    auto texclr = AT_NAME::sampleTexture(textures[texIdx], u, v, aten::vec4(1.0f));

    surf2Dwrite(
        make_float4(texclr.r, texclr.g, texclr.b, 1.0f),
        outSurface,
        ix * sizeof(float4), iy,
        hipBoundaryModeTrap);
}

namespace idaten
{
    void Renderer::viewTextures(
        uint32_t idx,
        int32_t screenWidth, int32_t screenHeight)
    {
        m_glimg.map();
        auto outputSurf = m_glimg.bind();

        if (!ctxt_host_->texRsc.empty()) {
            std::vector<hipTextureObject_t> tmp;
            for (auto& tex_rsc : ctxt_host_->texRsc) {
                auto cudaTex = tex_rsc.bind();
                tmp.push_back(cudaTex);
            }
            ctxt_host_->tex.writeFromHostToDeviceByNum(&tmp[0], (uint32_t)tmp.size());
        }

        dim3 block(BLOCK_SIZE, BLOCK_SIZE);
        dim3 grid(
            (screenWidth + block.x - 1) / block.x,
            (screenHeight + block.y - 1) / block.y);

        textureViewer << <grid, block >> > (
            idx,
            screenWidth, screenHeight,
            ctxt_host_->tex.data(),
            outputSurf);

        for (auto& tex_rsc : ctxt_host_->texRsc) {
            tex_rsc.unbind();
        }

        m_glimg.unbind();
        m_glimg.unmap();
    }
}
