#include "hip/hip_runtime.h"
#include <algorithm>

#include "kernel/GenMipMaps.h"

#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudamemory.h"
#include "cuda/cudautil.h"

// NOTE
// http://www.cse.uaa.alaska.edu/~ssiewert/a490dmis_code/CUDA/cuda_work/samples/2_Graphics/bindlessTexture/bindlessTexture_kernel.cu

__global__ void genMipmap(
    hipSurfaceObject_t mipOutput,
    hipTextureObject_t mipInput,
    int imageW, int imageH)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    float px = 1.0 / float(imageW);
    float py = 1.0 / float(imageH);

    if ((x < imageW) && (y < imageH))
    {
        // take the average of 4 samples

        // we are using the normalized access to make sure non-power-of-two textures
        // behave well when downsized.
        float4 color =
            (tex2D<float4>(mipInput, (x + 0) * px, (y + 0) * py)) +
            (tex2D<float4>(mipInput, (x + 1) * px, (y + 0) * py)) +
            (tex2D<float4>(mipInput, (x + 1) * px, (y + 1) * py)) +
            (tex2D<float4>(mipInput, (x + 0) * px, (y + 1) * py));


        color /= 4.0f;

        surf2Dwrite(color, mipOutput, x * sizeof(float4), y);
    }
}

namespace idaten {
    void generateMipMaps(
        hipMipmappedArray_t mipmapArray,
        int width, int height,
        int maxLevel)
    {
        int level = 0;

        //while (width != 1 || height != 1)
        while (level + 1 < maxLevel)
        {
            width /= 2;
            height /= 2;

            width = std::max(1, width);
            height = std::max(1, height);

            // Copy from.
            hipArray_t levelFrom;
            checkCudaErrors(hipGetMipmappedArrayLevel(&levelFrom, mipmapArray, level));

            // Copy to.
            hipArray_t levelTo;
            checkCudaErrors(hipGetMipmappedArrayLevel(&levelTo, mipmapArray, level + 1));

            hipExtent levelToSize;
            checkCudaErrors(hipArrayGetInfo(nullptr, &levelToSize, nullptr, levelTo));
            AT_ASSERT(levelToSize.width == width);
            AT_ASSERT(levelToSize.height == height);
            AT_ASSERT(levelToSize.depth == 0);

            // generate texture object for reading
            hipTextureObject_t texInput;
            {
                hipResourceDesc texRes;
                {
                    memset(&texRes, 0, sizeof(hipResourceDesc));

                    texRes.resType = hipResourceTypeArray;
                    texRes.res.array.array = levelFrom;
                }

                hipTextureDesc texDesc;
                {
                    memset(&texDesc, 0, sizeof(hipTextureDesc));

                    texDesc.normalizedCoords = 1;
                    texDesc.filterMode = hipFilterModeLinear;
                    texDesc.addressMode[0] = hipAddressModeClamp;
                    texDesc.addressMode[1] = hipAddressModeClamp;
                    texDesc.addressMode[2] = hipAddressModeClamp;
                    texDesc.readMode = hipReadModeElementType;
                }

                checkCudaErrors(hipCreateTextureObject(&texInput, &texRes, &texDesc, nullptr));
            }

            // generate surface object for writing
            hipSurfaceObject_t surfOutput;
            {
                hipResourceDesc surfRes;
                {
                    memset(&surfRes, 0, sizeof(hipResourceDesc));
                    surfRes.resType = hipResourceTypeArray;
                    surfRes.res.array.array = levelTo;
                }

                checkCudaErrors(hipCreateSurfaceObject(&surfOutput, &surfRes));
            }

            // run mipmap kernel
            dim3 block(16, 16, 1);
            dim3 grid(
                (width + block.x - 1) / block.x,
                (height + block.y - 1) / block.y, 1);

            genMipmap << <grid, block >> > (
                surfOutput,
                texInput,
                width, height);

            checkCudaErrors(hipDeviceSynchronize());
            checkCudaErrors(hipGetLastError());

            checkCudaErrors(hipDestroySurfaceObject(surfOutput));

            checkCudaErrors(hipDestroyTextureObject(texInput));

            level++;
        }
    }
}
