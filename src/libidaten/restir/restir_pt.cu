#include "hip/hip_runtime.h"
#include "restir/restir.h"

#include "aten4idaten.h"
#include "kernel/accelerator.cuh"
#include "kernel/context.cuh"
#include "kernel/intersect.cuh"
#include "kernel/light.cuh"
#include "kernel/material.cuh"
#include "kernel/pt_common.h"
#include "kernel/StreamCompaction.h"
#include "kernel/pt_standard_impl.h"

#include "cuda/cudadefs.h"
#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

#define ENABLE_PERSISTENT_THREAD

__global__ void genPath(
    idaten::TileDomain tileDomain,
    bool isFillAOV,
    idaten::Path* paths,
    aten::ray* rays,
    idaten::ReSTIRInfo* restir_info,
    int width, int height,
    int sample,
    unsigned int frame,
    const aten::CameraParameter* __restrict__ camera,
    const void* samplerValues,
    const unsigned int* __restrict__ random)
{
    auto ix = blockIdx.x * blockDim.x + threadIdx.x;
    auto iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix >= width || iy >= height) {
        return;
    }

    const auto idx = getIdx(ix, iy, width);

    paths->attrib[idx].isHit = false;

    if (paths->attrib[idx].isKill) {
        paths->attrib[idx].isTerminate = true;
        return;
    }

#if IDATEN_SAMPLER == IDATEN_SAMPLER_SOBOL
    auto scramble = random[idx] * 0x1fe3434f;
    paths->sampler[idx].init(frame + sample, 0, scramble, samplerValues);
#elif IDATEN_SAMPLER == IDATEN_SAMPLER_CMJ
    auto rnd = random[idx];
    auto scramble = rnd * 0x1fe3434f
        * (((frame + sample) + 133 * rnd) / (aten::CMJ::CMJ_DIM * aten::CMJ::CMJ_DIM));
    paths->sampler[idx].init(
        (frame + sample) % (aten::CMJ::CMJ_DIM * aten::CMJ::CMJ_DIM),
        0,
        scramble);
#endif

    float r1 = paths->sampler[idx].nextSample();
    float r2 = paths->sampler[idx].nextSample();

    if (isFillAOV) {
        r1 = r2 = 0.5f;
    }

    ix += tileDomain.x;
    iy += tileDomain.y;

    float s = (ix + r1) / (float)(camera->width);
    float t = (iy + r2) / (float)(camera->height);

    AT_NAME::CameraSampleResult camsample;
    AT_NAME::PinholeCamera::sample(&camsample, camera, s, t);

    rays[idx] = camsample.r;

    paths->throughput[idx].throughput = aten::vec3(1);
    paths->throughput[idx].pdfb = 0.0f;
    paths->attrib[idx].isTerminate = false;
    paths->attrib[idx].isSingular = false;

    paths->contrib[idx].samples += 1;

    // Clear restir info.
    restir_info[idx].clear();

    // Accumulate value, so do not reset.
    //path.contrib = aten::vec3(0);
}

__global__ void shadeMiss(
    idaten::TileDomain tileDomain,
    int bounce,
    float4* aovNormalDepth,
    float4* aovTexclrMeshid,
    idaten::Path* paths,
    int width, int height)
{
    auto ix = blockIdx.x * blockDim.x + threadIdx.x;
    auto iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix >= tileDomain.w || iy >= tileDomain.h) {
        return;
    }

    const auto idx = getIdx(ix, iy, tileDomain.w);

    if (!paths->attrib[idx].isTerminate && !paths->attrib[idx].isHit) {
        // TODO
        auto bg = aten::vec3(0);

        if (bounce == 0) {
            paths->attrib[idx].isKill = true;

            ix += tileDomain.x;
            iy += tileDomain.y;
            const auto _idx = getIdx(ix, iy, width);

            // Export bg color to albedo buffer.
            aovTexclrMeshid[_idx] = make_float4(bg.x, bg.y, bg.z, -1);
            aovNormalDepth[_idx].w = -1;

            // For exporting separated albedo.
            bg = aten::vec3(1, 1, 1);
        }

        auto contrib = paths->throughput[idx].throughput * bg;
        paths->contrib[idx].contrib += make_float3(contrib.x, contrib.y, contrib.z);

        paths->attrib[idx].isTerminate = true;
    }
}

__global__ void shadeMissWithEnvmap(
    idaten::TileDomain tileDomain,
    int offsetX, int offsetY,
    int bounce,
    const aten::CameraParameter* __restrict__ camera,
    float4* aovNormalDepth,
    float4* aovTexclrMeshid,
    hipTextureObject_t* textures,
    int envmapIdx,
    real envmapAvgIllum,
    real envmapMultiplyer,
    idaten::Path* paths,
    const aten::ray* __restrict__ rays,
    int width, int height)
{
    auto ix = blockIdx.x * blockDim.x + threadIdx.x;
    auto iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix >= tileDomain.w || iy >= tileDomain.h) {
        return;
    }

    const auto idx = getIdx(ix, iy, tileDomain.w);

    if (!paths->attrib[idx].isTerminate && !paths->attrib[idx].isHit) {
        aten::vec3 dir = rays[idx].dir;

        if (bounce == 0) {
            // Suppress jittering envrinment map.
            // So, re-sample ray without random.

            // TODO
            // More efficient way...

            float s = (ix + offsetX) / (float)(width);
            float t = (iy + offsetY) / (float)(height);

            AT_NAME::CameraSampleResult camsample;
            AT_NAME::PinholeCamera::sample(&camsample, camera, s, t);

            dir = camsample.r.dir;
        }

        auto uv = AT_NAME::envmap::convertDirectionToUV(dir);

        auto bg = tex2D<float4>(textures[envmapIdx], uv.x, uv.y);
        auto emit = aten::vec3(bg.x, bg.y, bg.z);

        float misW = 1.0f;
        if (bounce == 0
            || (bounce == 1 && paths->attrib[idx].isSingular))
        {
            paths->attrib[idx].isKill = true;

            ix += tileDomain.x;
            iy += tileDomain.y;
            const auto _idx = getIdx(ix, iy, width);

            // Export envmap to albedo buffer.
            aovTexclrMeshid[_idx] = make_float4(emit.x, emit.y, emit.z, -1);
            aovNormalDepth[_idx].w = -1;
        }
        else {
            auto pdfLight = AT_NAME::ImageBasedLight::samplePdf(emit, envmapAvgIllum);
            misW = paths->throughput[idx].pdfb / (pdfLight + paths->throughput[idx].pdfb);

            emit *= envmapMultiplyer;
        }

        auto contrib = paths->throughput[idx].throughput * misW * emit;
        paths->contrib[idx].contrib += make_float3(contrib.x, contrib.y, contrib.z);

        paths->attrib[idx].isTerminate = true;
    }
}

__global__ void shade(
    idaten::TileDomain tileDomain,
    float4* aovNormalDepth,
    float4* aovTexclrMeshid,
    aten::mat4 mtxW2C,
    int width, int height,
    idaten::Path* paths,
    const int* __restrict__ hitindices,
    int* hitnum,
    const aten::Intersection* __restrict__ isects,
    aten::ray* rays,
    int sample,
    int frame,
    int bounce, int rrBounce,
    const aten::GeomParameter* __restrict__ shapes, int geomnum,
    const aten::MaterialParameter* __restrict__ mtrls,
    const aten::LightParameter* __restrict__ lights, int lightnum,
    const aten::PrimitiveParamter* __restrict__ prims,
    hipTextureObject_t vtxPos,
    hipTextureObject_t vtxNml,
    const aten::mat4* __restrict__ matrices,
    hipTextureObject_t* textures,
    unsigned int* random,
    idaten::ShadowRay* shadowRays)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= *hitnum) {
        return;
    }

    idaten::Context ctxt;
    {
        ctxt.geomnum = geomnum;
        ctxt.shapes = shapes;
        ctxt.mtrls = mtrls;
        ctxt.lightnum = lightnum;
        ctxt.lights = lights;
        ctxt.prims = prims;
        ctxt.vtxPos = vtxPos;
        ctxt.vtxNml = vtxNml;
        ctxt.matrices = matrices;
        ctxt.textures = textures;
    }

    idx = hitindices[idx];

    __shared__ idaten::ShadowRay shShadowRays[64];
    __shared__ aten::MaterialParameter shMtrls[64];

    const auto ray = rays[idx];

#if IDATEN_SAMPLER == IDATEN_SAMPLER_SOBOL
    auto scramble = random[idx] * 0x1fe3434f;
    paths->sampler[idx].init(frame + sample, 4 + bounce * 300, scramble);
#elif IDATEN_SAMPLER == IDATEN_SAMPLER_CMJ
    auto rnd = random[idx];
    auto scramble = rnd * 0x1fe3434f
        * (((frame + sample) + 331 * rnd) / (aten::CMJ::CMJ_DIM * aten::CMJ::CMJ_DIM));
    paths->sampler[idx].init(
        (frame + sample) % (aten::CMJ::CMJ_DIM * aten::CMJ::CMJ_DIM),
        4 + bounce * 300,
        scramble);
#endif

    aten::hitrecord rec;

    const auto& isect = isects[idx];

    auto obj = &ctxt.shapes[isect.objid];
    evalHitResult(&ctxt, obj, ray, &rec, &isect);

    bool isBackfacing = dot(rec.normal, -ray.dir) < 0.0f;

    // 交差位置の法線.
    // 物体からのレイの入出を考慮.
    aten::vec3 orienting_normal = rec.normal;

    if (rec.mtrlid >= 0) {
        shMtrls[threadIdx.x] = ctxt.mtrls[rec.mtrlid];

        if (rec.isVoxel) {
            // Replace to lambert.
            const auto& albedo = ctxt.mtrls[rec.mtrlid].baseColor;
            shMtrls[threadIdx.x] = aten::MaterialParameter(aten::MaterialType::Lambert, MaterialAttributeLambert);
            shMtrls[threadIdx.x].baseColor = albedo;
        }

        if (shMtrls[threadIdx.x].type != aten::MaterialType::Layer) {
            shMtrls[threadIdx.x].albedoMap = (int)(shMtrls[threadIdx.x].albedoMap >= 0 ? ctxt.textures[shMtrls[threadIdx.x].albedoMap] : -1);
            shMtrls[threadIdx.x].normalMap = (int)(shMtrls[threadIdx.x].normalMap >= 0 ? ctxt.textures[shMtrls[threadIdx.x].normalMap] : -1);
            shMtrls[threadIdx.x].roughnessMap = (int)(shMtrls[threadIdx.x].roughnessMap >= 0 ? ctxt.textures[shMtrls[threadIdx.x].roughnessMap] : -1);
        }
    }
    else {
        // TODO
        shMtrls[threadIdx.x] = aten::MaterialParameter(aten::MaterialType::Lambert, MaterialAttributeLambert);
        shMtrls[threadIdx.x].baseColor = aten::vec3(1.0f);
    }

    auto albedo = AT_NAME::sampleTexture(shMtrls[threadIdx.x].albedoMap, rec.u, rec.v, aten::vec4(1), bounce);

    // Apply normal map.
    int normalMap = shMtrls[threadIdx.x].normalMap;
    if (shMtrls[threadIdx.x].type == aten::MaterialType::Layer) {
        // 最表層の NormalMap を適用.
        auto* topmtrl = &ctxt.mtrls[shMtrls[threadIdx.x].layer[0]];
        normalMap = (int)(topmtrl->normalMap >= 0 ? ctxt.textures[topmtrl->normalMap] : -1);
    }
    AT_NAME::applyNormalMap(normalMap, orienting_normal, orienting_normal, rec.u, rec.v);

    if (bounce == 0) {
        // Store AOV.
        int ix = idx % tileDomain.w;
        int iy = idx / tileDomain.w;

        ix += tileDomain.x;
        iy += tileDomain.y;

        const auto _idx = getIdx(ix, iy, width);

        // World coordinate to Clip coordinate.
        aten::vec4 pos = aten::vec4(rec.p, 1);
        pos = mtxW2C.apply(pos);

        aovNormalDepth[_idx] = make_float4(orienting_normal.x, orienting_normal.y, orienting_normal.z, pos.w);
        aovTexclrMeshid[_idx] = make_float4(albedo.x, albedo.y, albedo.z, isect.mtrlid);
    }

    // Implicit conection to light.
    if (shMtrls[threadIdx.x].attrib.isEmissive) {
        if (!isBackfacing) {
            float weight = 1.0f;

            if (bounce > 0 && !paths->attrib[idx].isSingular) {
                auto cosLight = dot(orienting_normal, -ray.dir);
                auto dist2 = aten::squared_length(rec.p - ray.org);

                if (cosLight >= 0) {
                    auto pdfLight = 1 / rec.area;

                    // Convert pdf area to sradian.
                    // http://kagamin.net/hole/edubpt/edubpt_v100.pdf
                    // p31 - p35
                    pdfLight = pdfLight * dist2 / cosLight;

                    weight = paths->throughput[idx].pdfb / (pdfLight + paths->throughput[idx].pdfb);
                }
            }

            auto contrib = paths->throughput[idx].throughput * weight * static_cast<aten::vec3>(shMtrls[threadIdx.x].baseColor);
            paths->contrib[idx].contrib += make_float3(contrib.x, contrib.y, contrib.z);
        }

        // When ray hit the light, tracing will finish.
        paths->attrib[idx].isTerminate = true;
        return;
    }

    if (!shMtrls[threadIdx.x].attrib.isTranslucent && isBackfacing) {
        orienting_normal = -orienting_normal;
    }

    shShadowRays[threadIdx.x].isActive = false;

    // Explicit conection to light.
    if (!(shMtrls[threadIdx.x].attrib.isSingular || shMtrls[threadIdx.x].attrib.isTranslucent))
    {
        auto shadowRayOrg = rec.p + AT_MATH_EPSILON * orienting_normal;

        aten::LightSampleResult sampleres;
        aten::LightParameter light;

        auto lightidx = aten::cmpMin<int>(paths->sampler[idx].nextSample() * lightnum, lightnum - 1);
        auto lightSelectPdf = 1.0f / lightnum;

        light.pos = ((aten::vec4*)ctxt.lights)[lightidx * aten::LightParameter_float4_size + 0];
        light.dir = ((aten::vec4*)ctxt.lights)[lightidx * aten::LightParameter_float4_size + 1];
        light.le = ((aten::vec4*)ctxt.lights)[lightidx * aten::LightParameter_float4_size + 2];
        light.v0 = ((aten::vec4*)ctxt.lights)[lightidx * aten::LightParameter_float4_size + 3];
        light.v1 = ((aten::vec4*)ctxt.lights)[lightidx * aten::LightParameter_float4_size + 4];
        light.v2 = ((aten::vec4*)ctxt.lights)[lightidx * aten::LightParameter_float4_size + 5];
        //auto light = ctxt.lights[lightidx];

        sampleLight(&sampleres, &ctxt, &light, rec.p, orienting_normal, &paths->sampler[idx], bounce);

        if (lightidx >= 0) {
            const auto& posLight = sampleres.pos;
            const auto& nmlLight = sampleres.nml;
            real pdfLight = sampleres.pdf;

            auto dirToLight = normalize(sampleres.dir);
            auto distToLight = length(posLight - rec.p);

            auto tmp = rec.p + dirToLight - shadowRayOrg;
            auto shadowRayDir = normalize(tmp);

            bool isShadowRayActive = false;

            shShadowRays[threadIdx.x].rayorg = shadowRayOrg;
            shShadowRays[threadIdx.x].raydir = shadowRayDir;
            shShadowRays[threadIdx.x].targetLightId = lightidx;
            shShadowRays[threadIdx.x].distToLight = distToLight;
            shShadowRays[threadIdx.x].lightcontrib = aten::vec3(0);
            {
                auto cosShadow = dot(orienting_normal, dirToLight);

                real pdfb = samplePDF(&ctxt, &shMtrls[threadIdx.x], orienting_normal, ray.dir, dirToLight, rec.u, rec.v);
                auto bsdf = sampleBSDF(&ctxt, &shMtrls[threadIdx.x], orienting_normal, ray.dir, dirToLight, rec.u, rec.v, albedo);

                bsdf *= paths->throughput[idx].throughput;

                // Get light color.
                auto emit = sampleres.finalColor;

                if (light.attrib.isInfinite || light.attrib.isSingular) {
                    if (pdfLight > real(0) && cosShadow >= 0) {
                        auto misW = light.attrib.isSingular
                            ? 1.0f
                            : AT_NAME::computeBalanceHeuristic(pdfLight * lightSelectPdf, pdfb);

                        shShadowRays[threadIdx.x].lightcontrib =
                            (misW * bsdf * emit * cosShadow / pdfLight) / lightSelectPdf;

                        isShadowRayActive = true;
                    }
                }
                else {
                    auto cosLight = dot(nmlLight, -dirToLight);

                    if (cosShadow >= 0 && cosLight >= 0) {
                        auto dist2 = aten::squared_length(sampleres.dir);
                        auto G = cosShadow * cosLight / dist2;

                        if (pdfb > real(0) && pdfLight > real(0)) {
                            // Convert pdf from steradian to area.
                            // http://kagamin.net/hole/edubpt/edubpt_v100.pdf
                            // p31 - p35
                            pdfb = pdfb * cosLight / dist2;

                            auto misW = AT_NAME::computeBalanceHeuristic(pdfLight * lightSelectPdf, pdfb);

                            shShadowRays[threadIdx.x].lightcontrib =
                                (misW * (bsdf * emit * G) / pdfLight) / lightSelectPdf;

                            isShadowRayActive = true;
                        }
                    }
                }
            }

            shShadowRays[threadIdx.x].isActive = isShadowRayActive;
        }
    }

    shadowRays[idx] = shShadowRays[threadIdx.x];

    real russianProb = real(1);

    if (bounce > rrBounce) {
        auto t = normalize(paths->throughput[idx].throughput);
        auto p = aten::cmpMax(t.r, aten::cmpMax(t.g, t.b));

        russianProb = paths->sampler[idx].nextSample();

        if (russianProb >= p) {
            //shPaths[threadIdx.x].contrib = aten::vec3(0);
            paths->attrib[idx].isTerminate = true;
        }
        else {
            russianProb = max(p, 0.01f);
        }
    }

    AT_NAME::MaterialSampling sampling;

    sampleMaterial(
        &sampling,
        &ctxt,
        &shMtrls[threadIdx.x],
        orienting_normal,
        ray.dir,
        rec.normal,
        &paths->sampler[idx],
        rec.u, rec.v,
        albedo);

    auto nextDir = normalize(sampling.dir);
    auto pdfb = sampling.pdf;
    auto bsdf = sampling.bsdf;

    // Get normal to add ray offset.
    // In refraction material case, new ray direction might be computed with inverted normal.
    // For example, when a ray go into the refraction surface, inverted normal is used to compute new ray direction.
    auto rayBasedNormal = (!isBackfacing && shMtrls[threadIdx.x].attrib.isTranslucent)
        ? -orienting_normal
        : orienting_normal;

    real c = 1;
    if (!shMtrls[threadIdx.x].attrib.isSingular) {
        // TODO
        // AMDのはabsしているが....
        //c = aten::abs(dot(orienting_normal, nextDir));
        c = dot(rayBasedNormal, nextDir);
    }

    if (pdfb > 0 && c > 0) {
        paths->throughput[idx].throughput *= bsdf * c / pdfb;
        paths->throughput[idx].throughput /= russianProb;
    }
    else {
        paths->attrib[idx].isTerminate = true;
    }

    // Make next ray.
    rays[idx] = aten::ray(rec.p, nextDir, rayBasedNormal);

    paths->throughput[idx].pdfb = pdfb;
    paths->attrib[idx].isSingular = shMtrls[threadIdx.x].attrib.isSingular;
    paths->attrib[idx].mtrlType = shMtrls[threadIdx.x].type;
}

__global__ void hitShadowRay(
    int bounce,
    idaten::Path* paths,
    int* hitindices,
    int* hitnum,
    const idaten::ShadowRay* __restrict__ shadowRays,
    const aten::GeomParameter* __restrict__ shapes, int geomnum,
    aten::MaterialParameter* mtrls,
    const aten::LightParameter* __restrict__ lights, int lightnum,
    hipTextureObject_t* nodes,
    const aten::PrimitiveParamter* __restrict__ prims,
    hipTextureObject_t vtxPos,
    const aten::mat4* __restrict__ matrices)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= *hitnum) {
        return;
    }

    idaten::Context ctxt;
    {
        ctxt.geomnum = geomnum;
        ctxt.shapes = shapes;
        ctxt.mtrls = mtrls;
        ctxt.lightnum = lightnum;
        ctxt.lights = lights;
        ctxt.nodes = nodes;
        ctxt.prims = prims;
        ctxt.vtxPos = vtxPos;
        ctxt.matrices = matrices;
    }

    idx = hitindices[idx];

    const auto& shadowRay = shadowRays[idx];

    if (!shadowRay.isActive) {
        return;
    }
    auto targetLightId = shadowRay.targetLightId;
    auto distToLight = shadowRay.distToLight;

    auto light = ctxt.lights[targetLightId];
    auto lightobj = (light.objid >= 0 ? &ctxt.shapes[light.objid] : nullptr);

    real distHitObjToRayOrg = AT_MATH_INF;

    // Ray aim to the area light.
    // So, if ray doesn't hit anything in intersectCloserBVH, ray hit the area light.
    const aten::GeomParameter* hitobj = lightobj;

    aten::Intersection isectTmp;

    bool isHit = false;

    aten::ray r(shadowRay.rayorg, shadowRay.raydir);

    // TODO
    bool enableLod = (bounce >= 2);

    isHit = intersectCloser(&ctxt, r, &isectTmp, distToLight - AT_MATH_EPSILON, enableLod);

    if (isHit) {
        hitobj = &ctxt.shapes[isectTmp.objid];
    }

    isHit = AT_NAME::scene::hitLight(
        isHit,
        light.attrib,
        lightobj,
        distToLight,
        distHitObjToRayOrg,
        isectTmp.t,
        hitobj);

    if (isHit) {
        auto contrib = shadowRay.lightcontrib;
        paths->contrib[idx].contrib += make_float3(contrib.x, contrib.y, contrib.z);
    }
}

__global__ void gather(
    idaten::TileDomain tileDomain,
    hipSurfaceObject_t dst,
    const idaten::Path* __restrict__ paths,
    bool enableProgressive,
    int width, int height)
{
    auto ix = blockIdx.x * blockDim.x + threadIdx.x;
    auto iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix >= tileDomain.w || iy >= tileDomain.h) {
        return;
    }

    auto idx = getIdx(ix, iy, tileDomain.w);

    float4 c = paths->contrib[idx].v;
    int sample = c.w;

    float4 contrib = c;

    ix += tileDomain.x;
    iy += tileDomain.y;
    idx = getIdx(ix, iy, width);

    if (enableProgressive) {
        float4 data;
        surf2Dread(&data, dst, ix * sizeof(float4), iy);

        // First data.w value is 0.
        int n = data.w;
        contrib = n * data + make_float4(c.x, c.y, c.z, 0) / sample;
        contrib /= (n + 1);
        contrib.w = n + 1;
    }
    else {
        contrib /= sample;
        contrib.w = 1;
    }

    if (dst) {
        surf2Dwrite(
            contrib,
            dst,
            ix * sizeof(float4), iy,
            hipBoundaryModeTrap);
    }
}

namespace idaten
{
    void ReSTIRPathTracing::onGenPath(
        int sample, int maxBounce,
        int seed,
        hipTextureObject_t texVtxPos,
        hipTextureObject_t texVtxNml)
    {
        dim3 block(BLOCK_SIZE, BLOCK_SIZE);
        dim3 grid(
            (m_tileDomain.w + block.x - 1) / block.x,
            (m_tileDomain.h + block.y - 1) / block.y);

        bool isFillAOV = m_mode == Mode::AOVar;

        genPath << <grid, block, 0, m_stream >> > (
            m_tileDomain,
            isFillAOV,
            m_paths.ptr(),
            m_rays.ptr(),
            m_restir_infos.ptr(),
            m_tileDomain.w, m_tileDomain.h,
            sample,
            m_frame,
            m_cam.ptr(),
            m_sobolMatrices.ptr(),
            m_random.ptr());

        checkCudaKernel(genPath);
    }

    void ReSTIRPathTracing::onHitTest(
        int width, int height,
        int bounce,
        hipTextureObject_t texVtxPos)
    {
        if (bounce == 0 && m_canSSRTHitTest) {
            hitTestOnScreenSpace(
                width, height,
                m_gbuffer,
                texVtxPos);
        }
        else {
            hitTest(
                width, height,
                bounce,
                texVtxPos);
        }
    }

    void ReSTIRPathTracing::onShadeMiss(
        int width, int height,
        int bounce,
        int offsetX/*= -1*/,
        int offsetY/*= -1*/)
    {
        dim3 block(BLOCK_SIZE, BLOCK_SIZE);
        dim3 grid(
            (m_tileDomain.w + block.x - 1) / block.x,
            (m_tileDomain.h + block.y - 1) / block.y);

        offsetX = offsetX < 0 ? m_tileDomain.x : offsetX;
        offsetY = offsetY < 0 ? m_tileDomain.y : offsetY;

        if (m_envmapRsc.idx >= 0) {
            shadeMissWithEnvmap << <grid, block, 0, m_stream >> > (
                m_tileDomain,
                offsetX, offsetY,
                bounce,
                m_cam.ptr(),
                m_aovNormalDepth.ptr(),
                m_aovTexclrMeshid.ptr(),
                m_tex.ptr(),
                m_envmapRsc.idx, m_envmapRsc.avgIllum, m_envmapRsc.multiplyer,
                m_paths.ptr(),
                m_rays.ptr(),
                width, height);
        }
        else {
            shadeMiss << <grid, block, 0, m_stream >> > (
                m_tileDomain,
                bounce,
                m_aovNormalDepth.ptr(),
                m_aovTexclrMeshid.ptr(),
                m_paths.ptr(),
                width, height);
        }

        checkCudaKernel(shadeMiss);
    }

    void ReSTIRPathTracing::onShade(
        hipSurfaceObject_t outputSurf,
        int width, int height,
        int sample,
        int bounce, int rrBounce,
        hipTextureObject_t texVtxPos,
        hipTextureObject_t texVtxNml)
    {
        m_mtxW2V.lookat(
            m_camParam.origin,
            m_camParam.center,
            m_camParam.up);

        m_mtxV2C.perspective(
            m_camParam.znear,
            m_camParam.zfar,
            m_camParam.vfov,
            m_camParam.aspect);

        m_mtxC2V = m_mtxV2C;
        m_mtxC2V.invert();

        m_mtxV2W = m_mtxW2V;
        m_mtxV2W.invert();

        aten::mat4 mtxW2C = m_mtxV2C * m_mtxW2V;

        dim3 blockPerGrid(((m_tileDomain.w * m_tileDomain.h) + 64 - 1) / 64);
        dim3 threadPerBlock(64);

        auto& hitcount = m_compaction.getCount();

        shade << <blockPerGrid, threadPerBlock, 0, m_stream >> > (
            m_tileDomain,
            m_aovNormalDepth.ptr(),
            m_aovTexclrMeshid.ptr(),
            mtxW2C,
            width, height,
            m_paths.ptr(),
            m_hitidx.ptr(), hitcount.ptr(),
            m_isects.ptr(),
            m_rays.ptr(),
            sample,
            m_frame,
            bounce, rrBounce,
            m_shapeparam.ptr(), m_shapeparam.num(),
            m_mtrlparam.ptr(),
            m_lightparam.ptr(), m_lightparam.num(),
            m_primparams.ptr(),
            texVtxPos, texVtxNml,
            m_mtxparams.ptr(),
            m_tex.ptr(),
            m_random.ptr(),
            m_shadowRays.ptr());

        checkCudaKernel(shade);

        onShadeByShadowRay(bounce, texVtxPos);
    }

    void ReSTIRPathTracing::onShadeByShadowRay(
        int bounce,
        hipTextureObject_t texVtxPos)
    {
        dim3 blockPerGrid(((m_tileDomain.w * m_tileDomain.h) + 64 - 1) / 64);
        dim3 threadPerBlock(64);

        auto& hitcount = m_compaction.getCount();

        hitShadowRay << <blockPerGrid, threadPerBlock, 0, m_stream >> > (
            bounce,
            m_paths.ptr(),
            m_hitidx.ptr(), hitcount.ptr(),
            m_shadowRays.ptr(),
            m_shapeparam.ptr(), m_shapeparam.num(),
            m_mtrlparam.ptr(),
            m_lightparam.ptr(), m_lightparam.num(),
            m_nodetex.ptr(),
            m_primparams.ptr(),
            texVtxPos,
            m_mtxparams.ptr());

        checkCudaKernel(hitShadowRay);
    }

    void ReSTIRPathTracing::onGather(
        hipSurfaceObject_t outputSurf,
        int width, int height,
        int maxSamples)
    {
        dim3 block(BLOCK_SIZE, BLOCK_SIZE);
        dim3 grid(
            (m_tileDomain.w + block.x - 1) / block.x,
            (m_tileDomain.h + block.y - 1) / block.y);

        gather << <grid, block, 0, m_stream >> > (
            m_tileDomain,
            outputSurf,
            m_paths.ptr(),
            m_enableProgressive,
            width, height);

        checkCudaKernel(gather);
    }
}
