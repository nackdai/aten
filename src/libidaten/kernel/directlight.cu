#include "hip/hip_runtime.h"
#include "kernel/directlight.h"
#include "kernel/context.cuh"
#include "kernel/light.cuh"
#include "kernel/material.cuh"
#include "kernel/intersect.cuh"
#include "kernel/accelerator.cuh"
#include "kernel/compaction.h"
#include "kernel/pt_common.h"

#include <hip/hip_runtime.h>
#include <>

#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

#include "aten4idaten.h"

__global__ void shade(
	hipSurfaceObject_t outSurface,
	int width, int height,
	idaten::PathTracing::Path* paths,
	int* hitindices,
	int hitnum,
	const aten::Intersection* __restrict__ isects,
	aten::ray* rays,
	int bounce, int rrBounce,
	const aten::GeomParameter* __restrict__ shapes, int geomnum,
	aten::MaterialParameter* mtrls,
	const aten::LightParameter* __restrict__ lights, int lightnum,
	hipTextureObject_t* nodes,
	const aten::PrimitiveParamter* __restrict__ prims,
	hipTextureObject_t vtxPos,
	hipTextureObject_t vtxNml,
	const aten::mat4* __restrict__ matrices,
	hipTextureObject_t* textures)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= hitnum) {
		return;
	}

	Context ctxt;
	{
		ctxt.geomnum = geomnum;
		ctxt.shapes = shapes;
		ctxt.mtrls = mtrls;
		ctxt.lightnum = lightnum;
		ctxt.lights = lights;
		ctxt.nodes = nodes;
		ctxt.prims = prims;
		ctxt.vtxPos = vtxPos;
		ctxt.vtxNml = vtxNml;
		ctxt.matrices = matrices;
		ctxt.textures = textures;
	}

	idx = hitindices[idx];

	auto& path = paths[idx];
	const auto& ray = rays[idx];

	aten::hitrecord rec;

	const auto& isect = isects[idx];

	auto obj = &ctxt.shapes[isect.objid];
	evalHitResult(&ctxt, obj, ray, &rec, &isect);

	aten::MaterialParameter mtrl = ctxt.mtrls[rec.mtrlid];

	// 交差位置の法線.
	// 物体からのレイの入出を考慮.
	aten::vec3 orienting_normal = dot(rec.normal, ray.dir) < 0.0 ? rec.normal : -rec.normal;

	// Apply normal map.
	if (mtrl.type == aten::MaterialType::Layer) {
		// 最表層の NormalMap を適用.
		auto* topmtrl = &ctxt.mtrls[mtrl.layer[0]];
		auto normalMap = (int)(topmtrl->normalMap >= 0 ? ctxt.textures[topmtrl->normalMap] : -1);
		AT_NAME::material::applyNormalMap(normalMap, orienting_normal, orienting_normal, rec.u, rec.v);
	}
	else {
		mtrl.albedoMap = (int)(mtrl.albedoMap >= 0 ? ctxt.textures[mtrl.albedoMap] : -1);
		mtrl.normalMap = (int)(mtrl.normalMap >= 0 ? ctxt.textures[mtrl.normalMap] : -1);
		mtrl.roughnessMap = (int)(mtrl.roughnessMap >= 0 ? ctxt.textures[mtrl.roughnessMap] : -1);

		AT_NAME::material::applyNormalMap(mtrl.normalMap, orienting_normal, orienting_normal, rec.u, rec.v);
	}

	// Implicit conection to light.
	if (mtrl.attrib.isEmissive) {
		float weight = 1.0f;

		if (bounce > 0 && !path.isSingular) {
			auto cosLight = dot(orienting_normal, -ray.dir);
			auto dist2 = aten::squared_length(rec.p - ray.org);

			if (cosLight >= 0) {
				auto pdfLight = 1 / rec.area;

				// Convert pdf area to sradian.
				// http://www.slideshare.net/h013/edubpt-v100
				// p31 - p35
				pdfLight = pdfLight * dist2 / cosLight;

				weight = path.pdfb / (pdfLight + path.pdfb);
			}
		}

		path.contrib += path.throughput * weight * mtrl.baseColor;

		// When ray hit the light, tracing will finish.
		path.isTerminate = true;
		return;
	}

	// Explicit conection to light.
	if (mtrl.attrib.isSingular)
	{
		AT_NAME::MaterialSampling sampling;

		sampleMaterial(
			&sampling,
			&ctxt,
			&mtrl,
			orienting_normal,
			ray.dir,
			rec.normal,
			&path.sampler,
			rec.u, rec.v);

		auto nextDir = normalize(sampling.dir);
		auto pdfb = sampling.pdf;
		auto bsdf = sampling.bsdf;

		if (pdfb > 0) {
			path.throughput *= bsdf / pdfb;
		}
		else {
			path.isTerminate = true;
		}

		// Make next ray.
		rays[idx] = aten::ray(rec.p, nextDir);

		path.pdfb = pdfb;
		path.isSingular = true;
	}
	else
	{
		for (int i = 0; i < lightnum; i++)
		{
			auto light = ctxt.lights[i];

			aten::LightSampleResult sampleres;
			sampleLight(&sampleres, &ctxt, &light, rec.p, orienting_normal, &path.sampler);

			const auto& posLight = sampleres.pos;
			const auto& nmlLight = sampleres.nml;
			real pdfLight = sampleres.pdf;

			auto lightobj = sampleres.obj;

			auto dirToLight = normalize(sampleres.dir);
			auto distToLight = length(posLight - rec.p);

			real distHitObjToRayOrg = AT_MATH_INF;

			// Ray aim to the area light.
			// So, if ray doesn't hit anything in intersectCloserBVH, ray hit the area light.
			auto hitobj = lightobj;

			aten::Intersection isectTmp;
			aten::ray shadowRay(rec.p, dirToLight);

			bool isHit = intersectCloser(&ctxt, shadowRay, &isectTmp, distToLight - AT_MATH_EPSILON);

			if (isHit) {
				hitobj = (void*)&ctxt.shapes[isectTmp.objid];
			}

			isHit = AT_NAME::scene::hitLight(
				isHit,
				light.attrib,
				lightobj,
				distToLight,
				distHitObjToRayOrg,
				isectTmp.t,
				hitobj);

			if (isHit) {
				auto cosShadow = dot(orienting_normal, dirToLight);

				real pdfb = samplePDF(&ctxt, &mtrl, orienting_normal, ray.dir, dirToLight, rec.u, rec.v);
				auto bsdf = sampleBSDF(&ctxt, &mtrl, orienting_normal, ray.dir, dirToLight, rec.u, rec.v);

				// Get light color.
				auto emit = sampleres.finalColor;

				if (light.attrib.isSingular || light.attrib.isInfinite) {
					if (pdfLight > real(0) && cosShadow >= 0) {
						// TODO
						// ジオメトリタームの扱いについて.
						// singular light の場合は、finalColor に距離の除算が含まれている.
						// inifinite light の場合は、無限遠方になり、pdfLightに含まれる距離成分と打ち消しあう？.
						// （打ち消しあうので、pdfLightには距離成分は含んでいない）.
						auto misW = pdfLight / (pdfb + pdfLight);
						path.contrib += (misW * bsdf * path.throughput * emit * cosShadow / pdfLight);
					}
				}
				else {
					auto cosLight = dot(nmlLight, -dirToLight);

					if (cosShadow >= 0 && cosLight >= 0) {
						auto dist2 = aten::squared_length(sampleres.dir);
						auto G = cosShadow * cosLight / dist2;

						if (pdfb > real(0) && pdfLight > real(0)) {
							// Convert pdf from steradian to area.
							// http://www.slideshare.net/h013/edubpt-v100
							// p31 - p35
							pdfb = pdfb * cosLight / dist2;

							auto misW = pdfLight / (pdfb + pdfLight);

							path.contrib += (misW * (bsdf * path.throughput * emit * G) / pdfLight);
						}
					}
				}

				if (!light.attrib.isSingular)
				{
					AT_NAME::MaterialSampling sampling;

					sampleMaterial(
						&sampling,
						&ctxt,
						&mtrl,
						orienting_normal,
						ray.dir,
						rec.normal,
						&path.sampler,
						rec.u, rec.v);

					auto nextDir = normalize(sampling.dir);
					auto pdfb = sampling.pdf;
					auto bsdf = sampling.bsdf;

					auto c = dot(orienting_normal, nextDir);
					aten::vec3 throughput(1, 1, 1);

					if (pdfb > 0 && c > 0) {
						throughput *= bsdf * c / pdfb;
					}

					aten::ray nextRay = aten::ray(rec.p, nextDir);

					aten::Intersection tmpIsect;
					bool isAnyHit = intersectClosest(&ctxt, nextRay, &tmpIsect);

					if (isAnyHit)
					{
						auto tmpObj = &ctxt.shapes[tmpIsect.objid];

						aten::hitrecord tmpRec;
						evalHitResult(&ctxt, tmpObj, nextRay, &tmpRec, &tmpIsect);

						aten::MaterialParameter mtrl = ctxt.mtrls[tmpRec.mtrlid];

						if (mtrl.attrib.isEmissive)
						{
							auto cosLight = dot(orienting_normal, -nextRay.dir);
							auto dist2 = aten::squared_length(tmpRec.p - nextRay.org);

							if (cosLight >= 0) {
								auto pdfLight = 1 / tmpRec.area;

								pdfLight = pdfLight * dist2 / cosLight;

								auto misW = pdfb / (pdfLight + pdfb);

								auto emit = mtrl.baseColor;

								path.contrib += throughput * misW * emit;
							}
						}
					}
					else {
						// TODO
					}
				}
			}
		}

		path.isTerminate = true;
	}
}

namespace idaten {
	void DirectLightRenderer::onShade(
		hipSurfaceObject_t outputSurf,
		int hitcount,
		int width, int height,
		int bounce, int rrBounce,
		hipTextureObject_t texVtxPos,
		hipTextureObject_t texVtxNml)
	{
		dim3 blockPerGrid((hitcount + 64 - 1) / 64);
		dim3 threadPerBlock(64);

		shade<< <blockPerGrid, threadPerBlock >> > (
			outputSurf,
			width, height,
			m_paths.ptr(),
			m_hitidx.ptr(), hitcount,
			m_isects.ptr(),
			m_rays.ptr(),
			bounce, rrBounce,
			m_shapeparam.ptr(), m_shapeparam.num(),
			m_mtrlparam.ptr(),
			m_lightparam.ptr(), m_lightparam.num(),
			m_nodetex.ptr(),
			m_primparams.ptr(),
			texVtxPos, texVtxNml,
			m_mtxparams.ptr(),
			m_tex.ptr());

		checkCudaKernel(shade);
	}
}
