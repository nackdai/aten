#include "hip/hip_runtime.h"
#include <utility>

#include "restir/restir.h"

#include "aten4idaten.h"
#include "kernel/accelerator.cuh"
#include "kernel/context.cuh"
#include "kernel/intersect.cuh"
#include "kernel/light.cuh"
#include "kernel/material.cuh"
#include "kernel/pt_common.h"

#include "cuda/cudadefs.h"
#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

__global__ void shade(
    idaten::TileDomain tileDomain,
    idaten::Reservoir* reservoirs,
    idaten::ReSTIRIntermedidate* intermediates,
    idaten::ReSTIRPathTracing::NormalMaterialStorage* nml_mtrl_buf,
    float4* aovNormalDepth,
    float4* aovTexclrMeshid,
    aten::mat4 mtxW2C,
    int width, int height,
    idaten::Path* paths,
    const int* __restrict__ hitindices,
    int* hitnum,
    const aten::Intersection* __restrict__ isects,
    aten::ray* rays,
    int sample,
    int frame,
    int bounce, int rrBounce,
    const aten::GeomParameter* __restrict__ shapes, int geomnum,
    const aten::MaterialParameter* __restrict__ mtrls,
    const aten::LightParameter* __restrict__ lights, int lightnum,
    const aten::PrimitiveParamter* __restrict__ prims,
    hipTextureObject_t vtxPos,
    hipTextureObject_t vtxNml,
    const aten::mat4* __restrict__ matrices,
    hipTextureObject_t* textures,
    unsigned int* random,
    idaten::ShadowRay* shadowRays)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= *hitnum) {
        return;
    }

    idaten::Context ctxt;
    {
        ctxt.geomnum = geomnum;
        ctxt.shapes = shapes;
        ctxt.mtrls = mtrls;
        ctxt.lightnum = lightnum;
        ctxt.lights = lights;
        ctxt.prims = prims;
        ctxt.vtxPos = vtxPos;
        ctxt.vtxNml = vtxNml;
        ctxt.matrices = matrices;
        ctxt.textures = textures;
    }

    idx = hitindices[idx];

    __shared__ idaten::ShadowRay shShadowRays[64];
    __shared__ aten::MaterialParameter shMtrls[64];
    __shared__ idaten::ReSTIRIntermedidate shIntermediates[64];

    const auto ray = rays[idx];

#if IDATEN_SAMPLER == IDATEN_SAMPLER_SOBOL
    auto scramble = random[idx] * 0x1fe3434f;
    paths->sampler[idx].init(frame + sample, 4 + bounce * 300, scramble);
#elif IDATEN_SAMPLER == IDATEN_SAMPLER_CMJ
    auto rnd = random[idx];
    auto scramble = rnd * 0x1fe3434f
        * (((frame + sample) + 331 * rnd) / (aten::CMJ::CMJ_DIM * aten::CMJ::CMJ_DIM));
    paths->sampler[idx].init(
        (frame + sample) % (aten::CMJ::CMJ_DIM * aten::CMJ::CMJ_DIM),
        4 + bounce * 300,
        scramble);
#endif

    aten::hitrecord rec;

    const auto& isect = isects[idx];

    auto obj = &ctxt.shapes[isect.objid];
    evalHitResult(&ctxt, obj, ray, &rec, &isect);

    bool isBackfacing = dot(rec.normal, -ray.dir) < 0.0f;

    // 交差位置の法線.
    // 物体からのレイの入出を考慮.
    aten::vec3 orienting_normal = rec.normal;

    if (rec.mtrlid >= 0) {
        shMtrls[threadIdx.x] = ctxt.mtrls[rec.mtrlid];

        // This kernel doesn't have to be called except first bounce.
        // And, in that case, hit material should not be voxel.
        // Therefore, we can ignore voxel check at all.
#if 0
        if (rec.isVoxel) {
            // Replace to lambert.
            const auto& albedo = ctxt.mtrls[rec.mtrlid].baseColor;
            shMtrls[threadIdx.x] = aten::MaterialParameter(aten::MaterialType::Lambert, MaterialAttributeLambert);
            shMtrls[threadIdx.x].baseColor = albedo;
        }
#endif

        if (shMtrls[threadIdx.x].type != aten::MaterialType::Layer) {
            shMtrls[threadIdx.x].albedoMap = (int)(shMtrls[threadIdx.x].albedoMap >= 0 ? ctxt.textures[shMtrls[threadIdx.x].albedoMap] : -1);
            shMtrls[threadIdx.x].normalMap = (int)(shMtrls[threadIdx.x].normalMap >= 0 ? ctxt.textures[shMtrls[threadIdx.x].normalMap] : -1);
            shMtrls[threadIdx.x].roughnessMap = (int)(shMtrls[threadIdx.x].roughnessMap >= 0 ? ctxt.textures[shMtrls[threadIdx.x].roughnessMap] : -1);
        }
    }
    else {
        // TODO
        shMtrls[threadIdx.x] = aten::MaterialParameter(aten::MaterialType::Lambert, MaterialAttributeLambert);
        shMtrls[threadIdx.x].baseColor = aten::vec3(1.0f);
    }

    auto albedo = AT_NAME::sampleTexture(shMtrls[threadIdx.x].albedoMap, rec.u, rec.v, aten::vec4(1), bounce);

    // Apply normal map.
    int normalMap = shMtrls[threadIdx.x].normalMap;
    if (shMtrls[threadIdx.x].type == aten::MaterialType::Layer) {
        // 最表層の NormalMap を適用.
        auto* topmtrl = &ctxt.mtrls[shMtrls[threadIdx.x].layer[0]];
        normalMap = (int)(topmtrl->normalMap >= 0 ? ctxt.textures[topmtrl->normalMap] : -1);
    }
    AT_NAME::applyNormalMap(normalMap, orienting_normal, orienting_normal, rec.u, rec.v);

    if (!shMtrls[threadIdx.x].attrib.isTranslucent
        && !shMtrls[threadIdx.x].attrib.isEmissive
        && isBackfacing)
    {
        orienting_normal = -orienting_normal;
    }

    shShadowRays[threadIdx.x].isActive = false;

    shIntermediates[threadIdx.x].clear();

    reservoirs[idx].light_idx = -1;

    if (bounce == 0) {
        // Store AOV.
        int ix = idx % tileDomain.w;
        int iy = idx / tileDomain.w;

        ix += tileDomain.x;
        iy += tileDomain.y;

        const auto _idx = getIdx(ix, iy, width);

        // World coordinate to Clip coordinate.
        aten::vec4 pos = aten::vec4(rec.p, 1);
        pos = mtxW2C.apply(pos);

        aovNormalDepth[_idx] = make_float4(orienting_normal.x, orienting_normal.y, orienting_normal.z, pos.w);
        aovTexclrMeshid[_idx] = make_float4(albedo.x, albedo.y, albedo.z, isect.mtrlid);

        nml_mtrl_buf[idx].normal = orienting_normal;
        nml_mtrl_buf[idx].mtrl_idx = rec.mtrlid;
        nml_mtrl_buf[idx].is_voxel = rec.isVoxel;
        nml_mtrl_buf[idx].is_mtrl_valid = (rec.mtrlid >= 0);
    }

    // Implicit conection to light.
    if (shMtrls[threadIdx.x].attrib.isEmissive) {
        if (!isBackfacing) {
            float weight = 1.0f;

            if (bounce > 0 && !paths->attrib[idx].isSingular) {
                auto cosLight = dot(orienting_normal, -ray.dir);
                auto dist2 = aten::squared_length(rec.p - ray.org);

                if (cosLight >= 0) {
                    auto pdfLight = 1 / rec.area;

                    // Convert pdf area to sradian.
                    // http://www.slideshare.net/h013/edubpt-v100
                    // p31 - p35
                    pdfLight = pdfLight * dist2 / cosLight;

                    weight = paths->throughput[idx].pdfb / (pdfLight + paths->throughput[idx].pdfb);
                }
            }

            auto contrib = paths->throughput[idx].throughput * weight * static_cast<aten::vec3>(shMtrls[threadIdx.x].baseColor);
            paths->contrib[idx].contrib += make_float3(contrib.x, contrib.y, contrib.z);
        }

        // When ray hit the light, tracing will finish.
        paths->attrib[idx].isTerminate = true;
        return;
    }

    ComputeBrdfFunctor compute_brdf_functor(
        ctxt, shMtrls[threadIdx.x], orienting_normal, ray.dir, rec.u, rec.v, albedo);

    // Explicit conection to light.
    if (!(shMtrls[threadIdx.x].attrib.isSingular || shMtrls[threadIdx.x].attrib.isTranslucent))
    {
        aten::LightSampleResult sampleres;
        aten::LightParameter light;

        auto lightidx = sampleLightWithReservoirRIP(
            &sampleres,
            reservoirs[idx],
            &light,
            compute_brdf_functor,
            &ctxt,
            rec.p, orienting_normal,
            &paths->sampler[idx],
            bounce);

        if (lightidx >= 0) {
            const auto& posLight = sampleres.pos;
            const auto& nmlLight = sampleres.nml;
            real pdfLight = sampleres.pdf;

            auto dirToLight = normalize(sampleres.dir);
            auto distToLight = length(posLight - rec.p);

            shShadowRays[threadIdx.x].rayorg = rec.p;
            shShadowRays[threadIdx.x].raydir = dirToLight;
            shShadowRays[threadIdx.x].targetLightId = lightidx;
            shShadowRays[threadIdx.x].distToLight = distToLight;
            shShadowRays[threadIdx.x].lightcontrib = aten::vec3(0);
            shShadowRays[threadIdx.x].isActive = true;

            shIntermediates[threadIdx.x].light_sample_nml = nmlLight;
            shIntermediates[threadIdx.x].light_color = sampleres.finalColor;
            shIntermediates[threadIdx.x].wi = ray.dir;
            shIntermediates[threadIdx.x].mtrl_idx = rec.mtrlid;
            shIntermediates[threadIdx.x].is_voxel = rec.isVoxel;
            shIntermediates[threadIdx.x].is_mtrl_valid = (rec.mtrlid >= 0);
            shIntermediates[threadIdx.x].throughput = paths->throughput[idx].throughput;
            shIntermediates[threadIdx.x].setNml(orienting_normal);
        }
    }

    shadowRays[idx] = shShadowRays[threadIdx.x];
    intermediates[idx] = shIntermediates[threadIdx.x];

    real russianProb = real(1);

    if (bounce > rrBounce) {
        auto t = normalize(paths->throughput[idx].throughput);
        auto p = aten::cmpMax(t.r, aten::cmpMax(t.g, t.b));

        russianProb = paths->sampler[idx].nextSample();

        if (russianProb >= p) {
            //shPaths[threadIdx.x].contrib = aten::vec3(0);
            paths->attrib[idx].isTerminate = true;
        }
        else {
            russianProb = max(p, 0.01f);
        }
    }

    AT_NAME::MaterialSampling sampling;

    sampleMaterial(
        &sampling,
        &ctxt,
        &shMtrls[threadIdx.x],
        orienting_normal,
        ray.dir,
        rec.normal,
        &paths->sampler[idx],
        rec.u, rec.v,
        albedo);

    auto nextDir = normalize(sampling.dir);
    auto pdfb = sampling.pdf;
    auto bsdf = sampling.bsdf;

    // Get normal to add ray offset.
    // In refraction material case, new ray direction might be computed with inverted normal.
    // For example, when a ray go into the refraction surface, inverted normal is used to compute new ray direction.
    auto rayBasedNormal = (!isBackfacing && shMtrls[threadIdx.x].attrib.isTranslucent)
        ? -orienting_normal
        : orienting_normal;

    real c = 1;
    if (!shMtrls[threadIdx.x].attrib.isSingular) {
        // TODO
        // AMDのはabsしているが....
        //c = aten::abs(dot(orienting_normal, nextDir));
        c = dot(rayBasedNormal, nextDir);
    }

    if (pdfb > 0 && c > 0) {
        paths->throughput[idx].throughput *= bsdf * c / pdfb;
        paths->throughput[idx].throughput /= russianProb;
    }
    else {
        paths->attrib[idx].isTerminate = true;
    }

    // Make next ray.
    rays[idx] = aten::ray(rec.p, nextDir, rayBasedNormal);

    paths->throughput[idx].pdfb = pdfb;
    paths->attrib[idx].isSingular = shMtrls[threadIdx.x].attrib.isSingular;
    paths->attrib[idx].mtrlType = shMtrls[threadIdx.x].type;
}

__global__ void hitShadowRay(
    int bounce,
    idaten::Path* paths,
    int* hitindices,
    int* hitnum,
    idaten::Reservoir* reservoirs,
    const idaten::ShadowRay* __restrict__ shadowRays,
    const aten::GeomParameter* __restrict__ shapes, int geomnum,
    aten::MaterialParameter* mtrls,
    const aten::LightParameter* __restrict__ lights, int lightnum,
    hipTextureObject_t* nodes,
    const aten::PrimitiveParamter* __restrict__ prims,
    hipTextureObject_t vtxPos,
    const aten::mat4* __restrict__ matrices)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= *hitnum) {
        return;
    }

    idaten::Context ctxt;
    {
        ctxt.geomnum = geomnum;
        ctxt.shapes = shapes;
        ctxt.mtrls = mtrls;
        ctxt.lightnum = lightnum;
        ctxt.lights = lights;
        ctxt.nodes = nodes;
        ctxt.prims = prims;
        ctxt.vtxPos = vtxPos;
        ctxt.matrices = matrices;
    }

    idx = hitindices[idx];

    const auto& shadowRay = shadowRays[idx];

    if (!shadowRay.isActive) {
        return;
    }
    auto targetLightId = shadowRay.targetLightId;
    auto distToLight = shadowRay.distToLight;

    auto light = ctxt.lights[targetLightId];
    auto lightobj = (light.objid >= 0 ? &ctxt.shapes[light.objid] : nullptr);

    real distHitObjToRayOrg = AT_MATH_INF;

    // Ray aim to the area light.
    // So, if ray doesn't hit anything in intersectCloserBVH, ray hit the area light.
    const aten::GeomParameter* hitobj = lightobj;

    aten::Intersection isectTmp;

    bool isHit = false;

    aten::ray r(shadowRay.rayorg, shadowRay.raydir);

    // TODO
    bool enableLod = (bounce >= 2);

    isHit = intersectCloser(&ctxt, r, &isectTmp, distToLight - AT_MATH_EPSILON, enableLod);

    if (isHit) {
        hitobj = &ctxt.shapes[isectTmp.objid];
    }

    isHit = AT_NAME::scene::hitLight(
        isHit,
        light.attrib,
        lightobj,
        distToLight,
        distHitObjToRayOrg,
        isectTmp.t,
        hitobj);

    if (isHit) {
        reservoirs[idx].w = 0.0f;
        reservoirs[idx].m = 0;
        reservoirs[idx].light_idx = -1;
        reservoirs[idx].light_pdf = 0.0f;
    }
    else {
        reservoirs[idx].light_idx = targetLightId;
    }
}

__global__ void computeShadowRayContribution(
    const idaten::Reservoir* __restrict__ reservoirs,
    const idaten::ReSTIRIntermedidate* __restrict__ intermediates,
    idaten::Path* paths,
    int* hitindices,
    int* hitnum,
    const float4* __restrict__ aovNormalDepth,
    const float4* __restrict__ aovTexclrMeshid,
    const aten::LightParameter* __restrict__ lights, int lightnum,
    const aten::MaterialParameter* __restrict__ mtrls,
    hipTextureObject_t* textures,
    const idaten::ShadowRay* __restrict__ shadowRays)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= *hitnum) {
        return;
    }

    idx = hitindices[idx];

    if (lightnum <= 0) {
        return;
    }

    idaten::Context ctxt;
    {
        ctxt.mtrls = mtrls;
        ctxt.textures = textures;
    }

    __shared__ aten::MaterialParameter shMtrls[64];

    const auto& reservoir = reservoirs[idx];
    const auto& intermediate = intermediates[idx];

    if (intermediate.is_mtrl_valid) {
        shMtrls[threadIdx.x] = ctxt.mtrls[intermediate.mtrl_idx];

        if (intermediate.is_voxel) {
            // Replace to lambert.
            const auto& albedo = ctxt.mtrls[intermediate.mtrl_idx].baseColor;
            shMtrls[threadIdx.x] = aten::MaterialParameter(aten::MaterialType::Lambert, MaterialAttributeLambert);
            shMtrls[threadIdx.x].baseColor = albedo;
        }

        if (shMtrls[threadIdx.x].type != aten::MaterialType::Layer) {
            shMtrls[threadIdx.x].albedoMap = (int)(shMtrls[threadIdx.x].albedoMap >= 0 ? ctxt.textures[shMtrls[threadIdx.x].albedoMap] : -1);
            shMtrls[threadIdx.x].normalMap = (int)(shMtrls[threadIdx.x].normalMap >= 0 ? ctxt.textures[shMtrls[threadIdx.x].normalMap] : -1);
            shMtrls[threadIdx.x].roughnessMap = (int)(shMtrls[threadIdx.x].roughnessMap >= 0 ? ctxt.textures[shMtrls[threadIdx.x].roughnessMap] : -1);
        }
    }
    else {
        // TODO
        shMtrls[threadIdx.x] = aten::MaterialParameter(aten::MaterialType::Lambert, MaterialAttributeLambert);
        shMtrls[threadIdx.x].baseColor = aten::vec3(1.0f);
    }

    if (!(shMtrls[threadIdx.x].attrib.isSingular || shMtrls[threadIdx.x].attrib.isTranslucent))
    {
        auto albedo_meshid = aovTexclrMeshid[idx];

        const aten::vec3 orienting_normal(
            intermediates[idx].nml_x,
            intermediates[idx].nml_y,
            intermediates[idx].nml_z);
        const aten::vec4 albedo(albedo_meshid.x, albedo_meshid.y, albedo_meshid.z, 1.0f);

        int lightidx = reservoir.light_idx;

        if (lightidx >= 0) {
            const auto& light = lights[lightidx];

            auto pdfLight = reservoir.light_pdf;

            auto nmlLight = intermediate.light_sample_nml;
            auto dirToLight = shadowRays[idx].raydir;
            auto distToLight = shadowRays[idx].distToLight;

            aten::vec3 lightcontrib;
            {
                auto cosShadow = dot(orienting_normal, dirToLight);

                // TODO
                // u,v は samplePDF/sampleBSDF 内部では利用されていない
                float u = 0.0f;
                float v = 0.0f;

                real pdfb = samplePDF(&ctxt, &shMtrls[threadIdx.x], orienting_normal, intermediate.wi, dirToLight, u, v);
                auto bsdf = sampleBSDF(&ctxt, &shMtrls[threadIdx.x], orienting_normal, intermediate.wi, dirToLight, u, v, albedo);

                bsdf *= intermediate.throughput;

                // Get light color.
                auto emit = intermediate.light_color;

                if (light.attrib.isSingular || light.attrib.isInfinite) {
                    if (pdfLight > real(0) && cosShadow >= 0) {
                        // TODO
                        // ジオメトリタームの扱いについて.
                        // singular light の場合は、finalColor に距離の除算が含まれている.
                        // inifinite light の場合は、無限遠方になり、pdfLightに含まれる距離成分と打ち消しあう？.
                        // （打ち消しあうので、pdfLightには距離成分は含んでいない）.
                        auto misW = pdfLight / (pdfb + pdfLight);

                        lightcontrib = misW * bsdf * emit * cosShadow / pdfLight;
                    }
                }
                else {
                    auto cosLight = dot(nmlLight, -dirToLight);

                    if (cosShadow >= 0 && cosLight >= 0) {
                        auto dist2 = distToLight * distToLight;
                        auto G = cosShadow * cosLight / dist2;

                        if (pdfb > real(0) && pdfLight > real(0)) {
                            // Convert pdf from steradian to area.
                            // http://www.slideshare.net/h013/edubpt-v100
                            // p31 - p35
                            pdfb = pdfb * cosLight / dist2;

                            auto misW = pdfLight / (pdfb + pdfLight);

                            lightcontrib = misW * (bsdf * emit * G) / pdfLight;
                        }
                    }
                }
            }

            paths->contrib[idx].contrib += make_float3(lightcontrib.x, lightcontrib.y, lightcontrib.z);
        }
    }
}

namespace idaten
{
    void ReSTIRPathTracing::onShadeReSTIR(
        hipSurfaceObject_t outputSurf,
        int width, int height,
        int sample,
        int bounce, int rrBounce,
        hipTextureObject_t texVtxPos,
        hipTextureObject_t texVtxNml)
    {
        m_mtxW2V.lookat(
            m_camParam.origin,
            m_camParam.center,
            m_camParam.up);

        m_mtxV2C.perspective(
            m_camParam.znear,
            m_camParam.zfar,
            m_camParam.vfov,
            m_camParam.aspect);

        m_mtxC2V = m_mtxV2C;
        m_mtxC2V.invert();

        m_mtxV2W = m_mtxW2V;
        m_mtxV2W.invert();

        aten::mat4 mtxW2C = m_mtxV2C * m_mtxW2V;

        dim3 blockPerGrid(((m_tileDomain.w * m_tileDomain.h) + 64 - 1) / 64);
        dim3 threadPerBlock(64);

        auto& hitcount = m_compaction.getCount();

        int curBufNmlMtrlPos = getCurBufNmlMtrlPos();

        shade << <blockPerGrid, threadPerBlock, 0, m_stream >> > (
            m_tileDomain,
            m_reservoirs[0].ptr(),
            m_intermediates[0].ptr(),
            m_bufNmlMtrl[curBufNmlMtrlPos].ptr(),
            m_aovNormalDepth.ptr(),
            m_aovTexclrMeshid.ptr(),
            mtxW2C,
            width, height,
            m_paths.ptr(),
            m_hitidx.ptr(), hitcount.ptr(),
            m_isects.ptr(),
            m_rays.ptr(),
            sample,
            m_frame,
            bounce, rrBounce,
            m_shapeparam.ptr(), m_shapeparam.num(),
            m_mtrlparam.ptr(),
            m_lightparam.ptr(), m_lightparam.num(),
            m_primparams.ptr(),
            texVtxPos, texVtxNml,
            m_mtxparams.ptr(),
            m_tex.ptr(),
            m_random.ptr(),
            m_shadowRays.ptr());

        checkCudaKernel(shade);

        onShadeByShadowRayReSTIR(
            width, height,
            bounce, texVtxPos);
    }

    void ReSTIRPathTracing::onShadeByShadowRayReSTIR(
        int width, int height,
        int bounce,
        hipTextureObject_t texVtxPos)
    {
        dim3 blockPerGrid(((m_tileDomain.w * m_tileDomain.h) + 64 - 1) / 64);
        dim3 threadPerBlock(64);

        auto& hitcount = m_compaction.getCount();

        hitShadowRay << <blockPerGrid, threadPerBlock, 0, m_stream >> > (
            bounce,
            m_paths.ptr(),
            m_hitidx.ptr(), hitcount.ptr(),
            m_reservoirs[0].ptr(),
            m_shadowRays.ptr(),
            m_shapeparam.ptr(), m_shapeparam.num(),
            m_mtrlparam.ptr(),
            m_lightparam.ptr(), m_lightparam.num(),
            m_nodetex.ptr(),
            m_primparams.ptr(),
            texVtxPos,
            m_mtxparams.ptr());

        checkCudaKernel(hitShadowRay);

        const auto target_idx = computelReuse(width, height, bounce);
        const auto reservior_idx = std::get<0>(target_idx);
        const auto intermediate_idx = std::get<1>(target_idx);

        computeShadowRayContribution << <blockPerGrid, threadPerBlock, 0, m_stream >> > (
            m_reservoirs[reservior_idx].ptr(),
            m_intermediates[intermediate_idx].ptr(),
            m_paths.ptr(),
            m_hitidx.ptr(), hitcount.ptr(),
            m_aovNormalDepth.ptr(),
            m_aovTexclrMeshid.ptr(),
            m_lightparam.ptr(), m_lightparam.num(),
            m_mtrlparam.ptr(),
            m_tex.ptr(),
            m_shadowRays.ptr());

        checkCudaKernel(computeShadowRayContribution);
    }
}
