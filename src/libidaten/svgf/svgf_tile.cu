#include "hip/hip_runtime.h"
#include "svgf/svgf.h"

#include "kernel/pt_common.h"
#include "cuda/cudadefs.h"
#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

__global__ void copyBufferForTile(
    idaten::TileDomain tileDomain,
    const idaten::Path paths,
    float4* contribs,
    int32_t width, int32_t height)
{
    int32_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix >= tileDomain.w || iy >= tileDomain.h) {
        return;
    }

    const auto dstIdx = getIdx(
        ix + tileDomain.x,
        iy + tileDomain.y,
        width);

    const auto srcIdx = getIdx(ix, iy, tileDomain.w);

    contribs[dstIdx] = paths.contrib[srcIdx].v;
}

namespace idaten
{
    void SVGFPathTracing::onCopyBufferForTile(int32_t width, int32_t height)
    {
        dim3 block(BLOCK_SIZE, BLOCK_SIZE);
        dim3 grid(
            (m_tileDomain.w + block.x - 1) / block.x,
            (m_tileDomain.h + block.y - 1) / block.y);

        copyBufferForTile << <grid, block, 0, m_stream >> > (
            m_tileDomain,
            m_paths,
            m_tmpBuf.ptr(),
            width, height);

        checkCudaKernel(copyBufferForTile);
    }
}
