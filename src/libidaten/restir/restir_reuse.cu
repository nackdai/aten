#include "hip/hip_runtime.h"
#include "restir/restir.h"

#include "kernel/pt_common.h"
#include "kernel/context.cuh"
#include "kernel/material.cuh"
#include "kernel/light.cuh"

#include "cuda/cudadefs.h"
#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

#include "aten4idaten.h"

#if 0
__device__ float computeEnergyCost(
    idaten::Context& ctxt,
    const aten::vec4 albedo,
    const idaten::Reservoir neighbor_reservoir,
    const idaten::ReSTIRIntermedidate& cur_info,
    const idaten::ReSTIRIntermedidate& neighbor_info)
{
    aten::MaterialParameter mtrl;
    if (cur_info.is_mtrl_valid) {
        mtrl = ctxt.mtrls[cur_info.mtrl_idx];
    }
    else {
        mtrl = aten::MaterialParameter(aten::MaterialType::Lambert, MaterialAttributeLambert);
        mtrl.baseColor = aten::vec3(1.0f);
    }

    const aten::vec3 orienting_normal(
        cur_info.nml_x,
        cur_info.nml_y,
        cur_info.nml_z);

    auto dir_to_light = neighbor_info.light_pos - cur_info.p;
    auto dist_to_light = length(dir_to_light);
    dir_to_light /= dist_to_light;

    auto bsdf = sampleBSDF(
        &ctxt,
        &mtrl,
        orienting_normal,
        cur_info.wi,
        dir_to_light,
        cur_info.u, cur_info.v,
        albedo);

    const auto neighbor_light_idx = neighbor_reservoir.light_idx;

    aten::vec3 energy;
    computeLighting(
        energy,
        ctxt.lights[neighbor_light_idx],
        orienting_normal,
        neighbor_info.light_sample_nml,
        neighbor_reservoir.light_pdf,
        neighbor_info.light_color,
        dir_to_light,
        dist_to_light);

    energy = energy * bsdf;

    auto cost = (energy.x + energy.y + energy.z) / 3;

    return cost;
}

__global__ void computeTemporalReuse(
    idaten::Path* paths,
    const aten::LightParameter* __restrict__ lights,
    const aten::MaterialParameter* __restrict__ mtrls,
    hipTextureObject_t* textures,
    const float4* __restrict__ aovTexclrMeshid,
    const idaten::Reservoir* __restrict__ cur_reservoirs,
    const idaten::Reservoir* __restrict__ prev_reservoirs,
    idaten::Reservoir* dst_reservoirs,
    const idaten::ReSTIRIntermedidate* __restrict__ intermediates,
    idaten::ReSTIRIntermedidate* dst_intermediates,
    const idaten::ReSTIRPathTracing::NormalMaterialStorage* __restrict__ cur_nml_mtrl_buf,
    const idaten::ReSTIRPathTracing::NormalMaterialStorage* __restrict__ prev_nml_mtrl_buf,
    hipSurfaceObject_t motionDetphBuffer,
    int width, int height)
{
    auto ix = blockIdx.x * blockDim.x + threadIdx.x;
    auto iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix >= width || iy >= height) {
        return;
    }

    auto idx = getIdx(ix, iy, width);

    float4 motionDepth;
    surf2Dread(&motionDepth, motionDetphBuffer, ix * sizeof(float4), iy);

    idaten::Context ctxt;
    {
        ctxt.mtrls = mtrls;
        ctxt.lights = lights;
        ctxt.textures = textures;
    }

    int reuse_idx = -1;
    auto new_reservoir = cur_reservoirs[idx];

    const auto& cur_nml = cur_nml_mtrl_buf[idx].normal;
    const auto& cur_mtrl_idx = cur_nml_mtrl_buf[idx].mtrl_idx;

    const auto& albedo_meshid = aovTexclrMeshid[idx];
    const aten::vec4 albedo(albedo_meshid.x, albedo_meshid.y, albedo_meshid.z, 1.0f);

    auto& sampler = paths->sampler[idx];

    // �O�̃t���[���̃X�N���[�����W.
    int px = (int)(ix + motionDepth.x * width);
    int py = (int)(iy + motionDepth.y * height);

    if (AT_MATH_IS_IN_BOUND(px, 0, width - 1)
        && AT_MATH_IS_IN_BOUND(py, 0, height - 1))
    {
        int prev_idx = getIdx(px, py, width);

        const auto& prev_nml = prev_nml_mtrl_buf[prev_idx].normal;
        const auto& prev_mtrl_idx = prev_nml_mtrl_buf[prev_idx].mtrl_idx;

        // TODO
        // Compare normal and material type
        // Even if material index is different, if the material type is same, it's ok.

        {
            const auto& prev_reservoir = prev_reservoirs[prev_idx];

            if (prev_reservoir.light_idx > 0) {
                const auto& cur_info = intermediates[idx];
                const auto& prev_info = intermediates[prev_idx];

                aten::MaterialParameter mtrl;
                if (cur_info.is_mtrl_valid) {
                    mtrl = mtrls[cur_info.mtrl_idx];
                }
                else {
                    mtrl = aten::MaterialParameter(aten::MaterialType::Lambert, MaterialAttributeLambert);
                    mtrl.baseColor = aten::vec3(1.0f);
                }

                const aten::vec3 orienting_normal(
                    cur_info.nml_x,
                    cur_info.nml_y,
                    cur_info.nml_z);

                auto dir_to_light = prev_info.light_pos - cur_info.p;
                auto dist_to_light = length(dir_to_light);
                dir_to_light /= dist_to_light;

                auto bsdf = sampleBSDF(
                    &ctxt,
                    &mtrl,
                    orienting_normal,
                    cur_info.wi,
                    dir_to_light,
                    cur_info.u, cur_info.v,
                    albedo);

                const auto prev_light_idx = prev_reservoir.light_idx;

                aten::vec3 energy;
                computeLighting(
                    energy,
                    lights[prev_light_idx],
                    orienting_normal,
                    prev_info.light_sample_nml,
                    prev_reservoir.light_pdf,
                    prev_info.light_color,
                    dir_to_light,
                    dist_to_light);

                energy = energy * bsdf;

                auto cost = (energy.x + energy.y + energy.z) / 3;

                auto w_sum = new_reservoir.w + cost;

                if (w_sum > 0.0f
                    && sampler.nextSample() <= cost / w_sum)
                {
                    new_reservoir.w = w_sum;
                    new_reservoir.m += min(prev_reservoir.m, 20 * new_reservoir.m);
                    new_reservoir.selected_cost = cost;
                    new_reservoir.pdf = w_sum / (cost * new_reservoir.m);
                    new_reservoir.light_pdf = prev_reservoir.light_pdf;
                    new_reservoir.light_idx = prev_reservoir.light_idx;
                    reuse_idx = prev_idx;
                }
            }
        }
    }

    if (!isfinite(new_reservoir.pdf)) {
        new_reservoir.light_pdf = real(0);
        new_reservoir.light_idx = -1;
    }

    dst_intermediates[idx] = intermediates[idx];

    if (reuse_idx >= 0) {
        dst_reservoirs[idx] = new_reservoir;

        dst_intermediates[idx].light_sample_nml = intermediates[reuse_idx].light_sample_nml;
        dst_intermediates[idx].light_color = intermediates[reuse_idx].light_color;
    }
    else {
        dst_reservoirs[idx] = cur_reservoirs[idx];
    }
}
#endif

#if 0
__host__ void computeSpatialReuse(
    int ix, int iy,
    aten::sampler* samplers,
    const aten::LightParameter* lights,
    const aten::MaterialParameter* mtrls,
    const float4* aovTexclrMeshid,
    const idaten::Reservoir* reservoirs,
    idaten::Reservoir* dst_reservoirs,
    const idaten::ReSTIRInfo* infos,
    int width, int height)
{
    auto idx = getIdx(ix, iy, width);

    idaten::Context ctxt;
    {
        ctxt.mtrls = mtrls;
        ctxt.lights = lights;
    }

    auto& sampler = samplers[idx];

    const auto& albedo_meshid = aovTexclrMeshid[idx];
    const aten::vec4 albedo(albedo_meshid.x, albedo_meshid.y, albedo_meshid.z, 1.0f);

    static const int offset_x[] = {
        -1,  0,  1,
        -1,  1,
        -1,  0,  1,
    };
    static const int offset_y[] = {
        -1, -1, -1,
         0,  0,
         1,  1,  1,
    };

    auto& comibined_reservoir = dst_reservoirs[idx];
    comibined_reservoir.clear();

    const auto& reservoir = reservoirs[idx];
    const auto& self_info = infos[idx];

    float selected_target_density = 0.0f;

    if (reservoir.isValid()) {
        comibined_reservoir = reservoir;
        selected_target_density = reservoir.target_density_;
    }

    const auto& normal = self_info.nml;

#pragma unroll
    for (int i = 0; i < AT_COUNTOF(offset_x); i++) {
        const auto xx = ix + offset_x[i];
        const auto yy = iy + offset_y[i];

        bool is_acceptable = AT_MATH_IS_IN_BOUND(xx, 0, width - 1)
            && AT_MATH_IS_IN_BOUND(yy, 0, height - 1);

        if (is_acceptable)
        {
            auto r = sampler.nextSample();

            auto neighbor_idx = getIdx(xx, yy, width);
            const auto& neighbor_reservoir = reservoirs[neighbor_idx];

            auto m = neighbor_reservoir.m_;

            aten::LightSampleResult lightsample;

            if (neighbor_reservoir.isValid()) {
                const auto& neighbor_info = infos[neighbor_idx];

                const auto light_pos = neighbor_reservoir.light_idx_;

                const auto& light = ctxt.lights[light_pos];

                //sampleLight(&lightsample, ctxt, &light, org, normal, sampler, lod);

                // TODO
                // Only point light
                AT_NAME::PointLight::sample(&light, self_info.p, &sampler, &lightsample);

                aten::vec3 nmlLight = lightsample.nml;
                aten::vec3 dirToLight = normalize(lightsample.dir);

                // TODO
                // Only lambert
                auto pdf = AT_NAME::lambert::pdf(normal, dirToLight);
                auto brdf = AT_NAME::lambert::bsdf(&mtrls[self_info.mtrl_idx], albedo);
                brdf /= pdf;

                auto cosShadow = dot(normal, dirToLight);
                auto cosLight = dot(nmlLight, -dirToLight);
                auto dist2 = aten::squared_length(lightsample.dir);

                auto energy = brdf * lightsample.finalColor;

                cosShadow = aten::abs(cosShadow);

                if (cosShadow > 0 && cosLight > 0) {
                    if (light.attrib.isSingular) {
                        energy = energy * cosShadow * cosLight;
                    }
                    else {
                        energy = energy * cosShadow * cosLight / dist2;
                    }
                }
                else {
                    energy.x = energy.y = energy.z = 0.0f;
                }

                auto target_density = (energy.x + energy.y + energy.z) / 3; // p_hat

                auto weight = target_density * neighbor_reservoir.pdf_ * m;

                if (comibined_reservoir.update(lightsample, light_pos, weight, m, r)) {
                    selected_target_density = target_density;
                }
            }
        }
    }

    if (selected_target_density > 0.0f) {
        comibined_reservoir.target_density_ = selected_target_density;
        // NOTE
        // 1/p_hat(xz) * (1/M * w_sum) = w_sum / (p_hat(xi) * M)
        comibined_reservoir.pdf_ = comibined_reservoir.w_sum_ / (comibined_reservoir.target_density_ * comibined_reservoir.m_);
    }

    if (!isfinite(comibined_reservoir.pdf_)) {
        comibined_reservoir.clear();
    }
}
#else
__global__ void computeSpatialReuse(
    idaten::Path* paths,
    const aten::LightParameter* __restrict__ lights,
    const aten::MaterialParameter* __restrict__ mtrls,
    hipTextureObject_t* textures,
    const float4* __restrict__ aovTexclrMeshid,
    const idaten::Reservoir* __restrict__ reservoirs,
    idaten::Reservoir* dst_reservoirs,
    const idaten::ReSTIRInfo* __restrict__ infos,
    int width, int height)
{
    auto ix = blockIdx.x * blockDim.x + threadIdx.x;
    auto iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix >= width || iy >= height) {
        return;
    }

    auto idx = getIdx(ix, iy, width);

    idaten::Context ctxt;
    {
        ctxt.mtrls = mtrls;
        ctxt.lights = lights;
        ctxt.textures = textures;
    }

    const auto& self_info = infos[idx];
    aten::MaterialParameter mtrl;
    gatherMaterialInfo(
        mtrl,
        &ctxt,
        self_info.mtrl_idx,
        self_info.is_voxel);

    const auto& normal = self_info.nml;

    auto& sampler = paths->sampler[idx];

    const auto& albedo_meshid = aovTexclrMeshid[idx];
    const aten::vec4 albedo(albedo_meshid.x, albedo_meshid.y, albedo_meshid.z, 1.0f);

    static const int offset_x[] = {
        -1,  0,  1,
        -1,  1,
        -1,  0,  1,
    };
    static const int offset_y[] = {
        -1, -1, -1,
         0,  0,
         1,  1,  1,
    };

    auto& comibined_reservoir = dst_reservoirs[idx];
    comibined_reservoir.clear();

    const auto& reservoir = reservoirs[idx];

    float selected_target_density = 0.0f;

    if (reservoir.isValid()) {
        comibined_reservoir = reservoir;
        selected_target_density = reservoir.target_density_;
    }

#pragma unroll
    for (int i = 0; i < AT_COUNTOF(offset_x); i++) {
        const auto xx = ix + offset_x[i];
        const auto yy = iy + offset_y[i];

        bool is_acceptable = AT_MATH_IS_IN_BOUND(xx, 0, width - 1)
            && AT_MATH_IS_IN_BOUND(yy, 0, height - 1);

        if (is_acceptable)
        {
            aten::LightSampleResult lightsample;

            auto neighbor_idx = getIdx(xx, yy, width);
            const auto& neighbor_reservoir = reservoirs[neighbor_idx];

            if (neighbor_reservoir.isValid()) {
                const auto& neighbor_info = infos[neighbor_idx];

                const auto& neighbor_normal = neighbor_info.nml;

                aten::MaterialParameter neightbor_mtrl;
                gatherMaterialInfo(
                    neightbor_mtrl,
                    &ctxt,
                    neighbor_info.mtrl_idx,
                    neighbor_info.is_voxel);

                // Check how close with neighbor pixel.
                is_acceptable = (mtrl.type == neightbor_mtrl.type)
                    && (dot(normal, neighbor_normal) >= 0.95f);

                if (is_acceptable) {
                    const auto light_pos = neighbor_reservoir.light_idx_;

                    const auto& light = ctxt.lights[light_pos];

                    sampleLight(&lightsample, &ctxt, &light, self_info.p, neighbor_normal, &sampler, 0);

                    aten::vec3 nmlLight = lightsample.nml;
                    aten::vec3 dirToLight = normalize(lightsample.dir);

                    auto pdf = samplePDF(
                        &ctxt, &neightbor_mtrl,
                        normal,
                        self_info.wi, dirToLight,
                        self_info.u, self_info.v);
                    auto brdf = sampleBSDF(
                        &ctxt, &neightbor_mtrl,
                        normal,
                        self_info.wi, dirToLight,
                        self_info.u, self_info.v,
                        albedo);
                    brdf /= pdf;

                    auto cosShadow = dot(normal, dirToLight);
                    auto cosLight = dot(nmlLight, -dirToLight);
                    auto dist2 = aten::squared_length(lightsample.dir);

                    auto energy = brdf * lightsample.finalColor;

                    cosShadow = aten::abs(cosShadow);

                    if (cosShadow > 0 && cosLight > 0) {
                        if (light.attrib.isSingular) {
                            energy = energy * cosShadow * cosLight;
                        }
                        else {
                            energy = energy * cosShadow * cosLight / dist2;
                        }
                    }
                    else {
                        energy.x = energy.y = energy.z = 0.0f;
                    }

                    auto target_density = (energy.x + energy.y + energy.z) / 3; // p_hat

                    auto m = neighbor_reservoir.m_;
                    auto weight = target_density * neighbor_reservoir.pdf_ * m;

                    auto r = sampler.nextSample();

                    if (comibined_reservoir.update(lightsample, light_pos, weight, m, r)) {
                        selected_target_density = target_density;
                    }
                }
            }
            else {
                comibined_reservoir.update(lightsample, -1, 0.0f, neighbor_reservoir.m_, 0.0f);
            }
        }
    }

    if (selected_target_density > 0.0f) {
        comibined_reservoir.target_density_ = selected_target_density;
        // NOTE
        // 1/p_hat(xz) * (1/M * w_sum) = w_sum / (p_hat(xi) * M)
        comibined_reservoir.pdf_ = comibined_reservoir.w_sum_ / (comibined_reservoir.target_density_ * comibined_reservoir.m_);
    }

    if (!isfinite(comibined_reservoir.pdf_)) {
        comibined_reservoir.clear();
    }
}
#endif

namespace idaten {
    std::tuple<int, int> ReSTIRPathTracing::computelReuse(
        int width, int height,
        int bounce)
    {
        dim3 block(BLOCK_SIZE, BLOCK_SIZE);
        dim3 grid(
            (m_tileDomain.w + block.x - 1) / block.x,
            (m_tileDomain.h + block.y - 1) / block.y);
#if 0
        constexpr int TEMPORAL_REUSE_RESERVOIR_SRC_IDX = 0;
        constexpr int TEMPORAL_REUSE_RESERVOIR_DST_IDX = 1;
        constexpr int TEMPORAL_REUSE_RESERVOIR_PREV_FRAME_IDX = 2;
        constexpr int SPATIAL_REUSE_RESERVOIR_DST_IDX = 2;

        int spatial_resue_reservoir_src_idx = 0;
        int spatial_resue_reservoir_dst_idx = SPATIAL_REUSE_RESERVOIR_DST_IDX;

        int spatial_resue_intermediate_src_idx = 0;
        int spatial_resue_intermediate_dst_idx = 1;

        if (bounce == 0) {
            if (m_restirMode == ReSTIRMode::ReSTIR) {
                if (m_frame > 1) {
                    int curBufNmlMtrlPos = getCurBufNmlMtrlPos();
                    int prevBufNmlMtrlPos = getPrevBufNmlMtrlPos();

                    CudaGLResourceMapper<decltype(m_motionDepthBuffer)> rscmap(m_motionDepthBuffer);
                    auto motionDepthBuffer = m_motionDepthBuffer.bind();

                    computeTemporalReuse << <grid, block, 0, m_stream >> > (
                        m_paths.ptr(),
                        m_lightparam.ptr(),
                        m_mtrlparam.ptr(),
                        m_tex.ptr(),
                        m_aovTexclrMeshid.ptr(),
                        m_reservoirs[TEMPORAL_REUSE_RESERVOIR_SRC_IDX].ptr(),
                        m_reservoirs[TEMPORAL_REUSE_RESERVOIR_PREV_FRAME_IDX].ptr(),
                        m_reservoirs[TEMPORAL_REUSE_RESERVOIR_DST_IDX].ptr(),
                        m_intermediates[spatial_resue_intermediate_src_idx].ptr(),
                        m_intermediates[spatial_resue_intermediate_dst_idx].ptr(),
                        m_bufNmlMtrl[curBufNmlMtrlPos].ptr(),
                        m_bufNmlMtrl[prevBufNmlMtrlPos].ptr(),
                        motionDepthBuffer,
                        width, height);

                    checkCudaKernel(computeTemporalReuse);

                    updateCurBufNmlMtrlPos();

                    spatial_resue_reservoir_src_idx = TEMPORAL_REUSE_RESERVOIR_DST_IDX;

                    std::swap(
                        spatial_resue_intermediate_src_idx,
                        spatial_resue_intermediate_dst_idx);
                }
            }

            computeSpatialReuse << <grid, block, 0, m_stream >> > (
                m_paths.ptr(),
                m_lightparam.ptr(),
                m_mtrlparam.ptr(),
                m_tex.ptr(),
                m_aovTexclrMeshid.ptr(),
                m_reservoirs[spatial_resue_reservoir_src_idx].ptr(),
                m_reservoirs[spatial_resue_reservoir_dst_idx].ptr(),
                m_intermediates[spatial_resue_intermediate_src_idx].ptr(),
                m_intermediates[spatial_resue_intermediate_dst_idx].ptr(),
                width, height);

            checkCudaKernel(computeSpatialReuse);
        }

        return std::make_tuple(
            spatial_resue_reservoir_dst_idx,
            spatial_resue_intermediate_dst_idx);
#elif 0
        decltype(m_pathSampler)::vector_type samplers;
        m_pathSampler.readFromDeviceToHost(samplers);

        decltype(m_lightparam)::vector_type lights;
        m_lightparam.readFromDeviceToHost(lights);

        decltype(m_mtrlparam)::vector_type mtrls;
        m_mtrlparam.readFromDeviceToHost(mtrls);

        decltype(m_aovTexclrMeshid)::vector_type aov;
        m_aovTexclrMeshid.readFromDeviceToHost(aov);

        decltype(m_reservoirs)::value_type::vector_type reservoirs;
        m_reservoirs[0].readFromDeviceToHost(reservoirs);

        decltype(m_reservoirs)::value_type::vector_type dst_reservoirs;
        dst_reservoirs.resize(reservoirs.size());

        decltype(m_restir_infos)::value_type::vector_type infos;
        m_restir_infos[0].readFromDeviceToHost(infos);

        for (int iy = 0; iy < height; iy++) {
            for (int ix = 0; ix < width; ix++) {
                computeSpatialReuse(
                    ix, iy,
                    samplers.data(),
                    lights.data(),
                    mtrls.data(),
                    aov.data(),
                    reservoirs.data(),
                    dst_reservoirs.data(),
                    infos.data(),
                    width, height);
            }
        }

        return std::make_tuple(0, 0);
#else
        if (bounce == 0) {
            computeSpatialReuse << <grid, block, 0, m_stream >> > (
                m_paths.ptr(),
                m_lightparam.ptr(),
                m_mtrlparam.ptr(),
                m_tex.ptr(),
                m_aovTexclrMeshid.ptr(),
                m_reservoirs[0].ptr(),
                m_reservoirs[1].ptr(),
                m_restir_infos[0].ptr(),
                width, height);

            checkCudaKernel(computeSpatialReuse);

            return std::make_tuple(1, 0);
        }

        return std::make_tuple(0, 0);
#endif
    }
}
