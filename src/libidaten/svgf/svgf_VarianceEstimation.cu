#include "hip/hip_runtime.h"
#include "svgf/svgf.h"

#include "kernel/pt_common.h"

#include "cuda/cudadefs.h"
#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

#include "aten4idaten.h"
#include "renderer/svgf/svgf_impl.h"

__global__ void varianceEstimation(
    hipSurfaceObject_t dst,
    const float4* __restrict__ aovNormalDepth,
    float4* aovMomentTemporalWeight,
    float4* aovColorVariance,
    float4* aovTexclrMeshid,
    aten::mat4 mtx_C2V,
    int32_t width, int32_t height,
    float cameraDistance)
{
    int32_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix >= width || iy >= height) {
        return;
    }

    const size_t size = width * height;

    auto aov_normal_depth{ aten::span<float4>(const_cast<float4*>(aovNormalDepth), size) };
    auto aov_texclr_meshid{ aten::span<float4>(aovTexclrMeshid, size) };
    auto aov_color_variance{ aten::span<float4>(aovColorVariance, size) };
    auto aov_moment_temporalweight{ aten::span<float4>(aovMomentTemporalWeight, size) };

    auto result = AT_NAME::svgf::EstimateVariance(
        ix, iy, width, height,
        mtx_C2V, cameraDistance,
        aov_normal_depth,
        aov_texclr_meshid,
        aov_color_variance,
        aov_moment_temporalweight);

    surf2Dwrite(
        result,
        dst,
        ix * sizeof(float4), iy,
        hipBoundaryModeTrap);
}

namespace idaten
{
    void SVGFPathTracing::onVarianceEstimation(
        hipSurfaceObject_t outputSurf,
        int32_t width, int32_t height)
    {
        dim3 block(BLOCK_SIZE, BLOCK_SIZE);
        dim3 grid(
            (width + block.x - 1) / block.x,
            (height + block.y - 1) / block.y);

        float cameraDistance = AT_NAME::camera::ComputeScreenDistance(m_cam, height);

        int32_t curaov_idx = getCurAovs();
        auto& curaov = aov_[curaov_idx];

        varianceEstimation << <grid, block, 0, m_stream >> > (
            outputSurf,
            curaov.get<AOVBuffer::NormalDepth>().data(),
            curaov.get<AOVBuffer::MomentTemporalWeight>().data(),
            curaov.get<AOVBuffer::ColorVariance>().data(),
            curaov.get<AOVBuffer::AlbedoMeshId>().data(),
            m_mtx_C2V,
            width, height,
            cameraDistance);

        checkCudaKernel(varianceEstimation);
    }
}
