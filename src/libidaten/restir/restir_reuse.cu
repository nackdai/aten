#include "hip/hip_runtime.h"
#include "restir/restir.h"

#include "kernel/pt_common.h"
#include "kernel/device_scene_context.cuh"

#include "cuda/cudadefs.h"
#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

#include "aten4idaten.h"
#include "light/light_impl.h"
#include "renderer/pathtracing/pt_params.h"
#include "renderer/restir/restir_impl.h"

__global__ void computeTemporalReuse(
    idaten::Path paths,
    idaten::context ctxt,
    const aten::ObjectParameter* __restrict__ shapes,
    const aten::MaterialParameter* __restrict__ mtrls,
    const aten::LightParameter* __restrict__ lights,
    const aten::TriangleParameter* __restrict__ prims,
    const aten::mat4* __restrict__ matrices,
    const float4* __restrict__ aovTexclrMeshid,
    idaten::Reservoir* reservoirs,
    const idaten::Reservoir* __restrict__ prev_reservoirs,
    const idaten::ReSTIRInfo* __restrict__ infos,
    hipSurfaceObject_t motionDetphBuffer,
    int32_t width, int32_t height)
{
    int32_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix >= width || iy >= height) {
        return;
    }

    auto idx = getIdx(ix, iy, width);

    if (paths.attrib[idx].isTerminate) {
        return;
    }

    ctxt.shapes = shapes;
    ctxt.mtrls = mtrls;
    ctxt.lights = lights;
    ctxt.prims = prims;
    ctxt.matrices = matrices;

    auto& sampler = paths.sampler[idx];

    const auto size = width * height;

    aten::const_span prev_reservoirs_as_span(prev_reservoirs, size);
    aten::const_span resitr_infos(infos, size);
    aten::const_span aov_texclr_meshid(aovTexclrMeshid, size);

    AT_NAME::restir::ApplyTemporalReuse(
        ix, iy,
        width, height,
        ctxt,
        sampler,
        reservoirs[idx],
        resitr_infos[idx],
        prev_reservoirs_as_span,
        resitr_infos,
        aov_texclr_meshid, motionDetphBuffer);
}

__global__ void computeSpatialReuse(
    idaten::Path paths,
    idaten::context ctxt,
    const aten::ObjectParameter* __restrict__ shapes,
    const aten::MaterialParameter* __restrict__ mtrls,
    const aten::LightParameter* __restrict__ lights,
    const aten::TriangleParameter* __restrict__ prims,
    const aten::mat4* __restrict__ matrices,
    const float4* __restrict__ aovTexclrMeshid,
    const idaten::Reservoir* __restrict__ reservoirs,
    idaten::Reservoir* dst_reservoirs,
    const idaten::ReSTIRInfo* __restrict__ infos,
    int32_t width, int32_t height)
{
    int32_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix >= width || iy >= height) {
        return;
    }

    auto idx = getIdx(ix, iy, width);

    if (paths.attrib[idx].isTerminate) {
        return;
    }

    ctxt.shapes = shapes;
    ctxt.mtrls = mtrls;
    ctxt.lights = lights;
    ctxt.prims = prims;
    ctxt.matrices = matrices;

    auto& sampler = paths.sampler[idx];

    const auto size = width * height;

    aten::const_span reservoirs_as_span(reservoirs, size);
    aten::const_span resitr_infos(infos, size);
    aten::const_span aov_texclr_meshid(aovTexclrMeshid, size);

    AT_NAME::restir::ApplySpatialReuse(
        ix, iy,
        width, height,
        ctxt,
        sampler,
        dst_reservoirs[idx],
        reservoirs_as_span,
        resitr_infos,
        aov_texclr_meshid);
}

namespace idaten {
    int32_t ReSTIRPathTracing::computelReuse(
        int32_t width, int32_t height,
        int32_t bounce)
    {
        dim3 block(BLOCK_SIZE, BLOCK_SIZE);
        dim3 grid(
            (width + block.x - 1) / block.x,
            (height + block.y - 1) / block.y);

        const auto curr_reservoirs_idx = m_reservoirs.GetCurrParamsIdx();
        const auto dst_reservoirs_idx = m_reservoirs.GetDestinationParamsIdxForSpatialReuse();

        if (bounce == 0) {
            // NOTE
            // temporal reuse �ŗ��p���� previous reservoir ��
            // spatial reuse ������O�̂��̂łȂ��Ƃ����Ȃ�
            // spatial reuse �͂����܂ł����݃t���[���ɑ΂��čs������̂�
            // ���t���[���ɉe����^���Ȃ��悤�ɂ���
            // e.g.
            //  - frame 1
            //     cur:0
            //     prev:N/A (�ŏ��Ȃ̂� temporal �� skip)
            //     spatial_dst:1
            //     pos=0 -> pos=1(for next)
            //  - frame 2
            //     cur:1(=pos)
            //     prev:0
            //     spatial_dst:0
            //     pos=1 -> pos=0(for next)
            //     ���̂Ƃ� prev �͑O�t���[���� cur �ƂȂ��Ă���

            auto& cur_reservoirs = m_reservoirs.GetCurrParams();
            auto& prev_reservoirs = m_reservoirs.GetPreviousFrameParamsForTemporalReuse();
            auto& dst_reservoirs = m_reservoirs.GetDestinationParamsForSpatialReuse();
            m_reservoirs.Update();

            if (m_restirMode == ReSTIRMode::ReSTIR
                || m_restirMode == ReSTIRMode::TemporalReuse) {
                if (m_frame > 1) {
                    CudaGLResourceMapper<decltype(m_motionDepthBuffer)> rscmap(m_motionDepthBuffer);
                    auto motionDepthBuffer = m_motionDepthBuffer.bind();

                    computeTemporalReuse << <grid, block, 0, m_stream >> > (
                        path_host_->paths,
                        ctxt_host_.ctxt,
                        ctxt_host_.shapeparam.data(),
                        ctxt_host_.mtrlparam.data(),
                        ctxt_host_.lightparam.data(),
                        ctxt_host_.primparams.data(),
                        ctxt_host_.mtxparams.data(),
                        aov_.albedo_meshid().data(),
                        cur_reservoirs.data(),
                        prev_reservoirs.data(),
                        m_restir_infos.data(),
                        motionDepthBuffer,
                        width, height);

                    checkCudaKernel(computeTemporalReuse);
                }
            }

            if (m_restirMode == ReSTIRMode::ReSTIR
                || m_restirMode == ReSTIRMode::SpatialReuse) {
                computeSpatialReuse << <grid, block, 0, m_stream >> > (
                    path_host_->paths,
                    ctxt_host_.ctxt,
                    ctxt_host_.shapeparam.data(),
                    ctxt_host_.mtrlparam.data(),
                    ctxt_host_.lightparam.data(),
                    ctxt_host_.primparams.data(),
                    ctxt_host_.mtxparams.data(),
                    aov_.albedo_meshid().data(),
                    cur_reservoirs.data(),
                    dst_reservoirs.data(),
                    m_restir_infos.data(),
                    width, height);

                checkCudaKernel(computeSpatialReuse);

                return dst_reservoirs_idx;
            }
        }

        return curr_reservoirs_idx;
    }
}
