#include "hip/hip_runtime.h"
#include "kernel/renderer.h"
#include "kernel/pt_common.h"

#include <hip/hip_runtime.h>
#include <>

#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

#include "aten4idaten.h"

__global__ void textureViewer(
    uint32_t texIdx,
    int width, int height,
    hipTextureObject_t* textures,
    hipSurfaceObject_t outSurface)
{
    const auto ix = blockIdx.x * blockDim.x + threadIdx.x;
    const auto iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix >= width || iy >= height) {
        return;
    }

    float u = ix / (float)width;
    float v = iy / (float)height;

    auto texclr = AT_NAME::sampleTexture(textures[texIdx], u, v, aten::vec3(1.0f));

    surf2Dwrite(
        make_float4(texclr.r, texclr.g, texclr.b, 1.0f),
        outSurface,
        ix * sizeof(float4), iy,
        hipBoundaryModeTrap);
}

namespace idaten
{
    void Renderer::viewTextures(
        uint32_t idx,
        int screenWidth, int screenHeight)
    {
        m_glimg.map();
        auto outputSurf = m_glimg.bind();

        if (!m_texRsc.empty()) {
            std::vector<hipTextureObject_t> tmp;
            for (int i = 0; i < m_texRsc.size(); i++) {
                auto cudaTex = m_texRsc[i].bind();
                tmp.push_back(cudaTex);
            }
            m_tex.writeByNum(&tmp[0], (uint32_t)tmp.size());
        }

        dim3 block(BLOCK_SIZE, BLOCK_SIZE);
        dim3 grid(
            (screenWidth + block.x - 1) / block.x,
            (screenHeight + block.y - 1) / block.y);

        textureViewer << <grid, block >> > (
            idx,
            screenWidth, screenHeight,
            m_tex.ptr(),
            outputSurf);

        for (int i = 0; i < m_texRsc.size(); i++) {
            m_texRsc[i].unbind();
        }

        m_glimg.unbind();
        m_glimg.unmap();
    }
}
