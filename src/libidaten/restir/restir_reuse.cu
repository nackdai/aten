#include "hip/hip_runtime.h"
#include "restir/restir.h"

#include "kernel/pt_common.h"

#include "cuda/cudadefs.h"
#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

#include "aten4idaten.h"

__host__ __device__ void OnComputeSpatialReuse(
    int idx,
    aten::sampler* sampler,
    const idaten::Reservoir* reservoirs,
    idaten::Reservoir* dst_reservoirs,
    const idaten::ReSTIRIntermedidate* intermediates,
    idaten::ReSTIRIntermedidate* dst_intermediates,
    int width, int height)
{
    int ix = idx % width;
    int iy = idx / width;

    static const int pos_x[] = {
        -1,
         0,
         1,

        -1,
         1,

        -1,
         0,
         1,
    };

    static const int pos_y[] = {
        -1,
        -1,
        -1,

         0,
         0,

         1,
         1,
         1,
    };

    int reuse_idx = -1;
    auto new_reservoir = reservoirs[idx];

    auto r = sampler->nextSample();

#pragma unroll
    for (int i = 0; i < AT_COUNTOF(pos_x); i++) {
        const auto x = ix + pos_x[i];
        const auto y = iy + pos_y[i];

        if (AT_MATH_IS_IN_BOUND(x, 0, width - 1)
            && AT_MATH_IS_IN_BOUND(y, 0, height - 1))
        {
            auto new_idx = getIdx(x, y, width);
            const auto& reservoir = reservoirs[new_idx];

            if (reservoir.w > 0.0f) {
                new_reservoir.w += reservoir.w;
                new_reservoir.m += reservoir.m;

                if (r <= reservoir.w / new_reservoir.w) {
                    new_reservoir.light_pdf = reservoir.light_pdf;
                    new_reservoir.light_idx = reservoir.light_idx;
                    reuse_idx = new_idx;
                }
            }
        }
    }

    if (reuse_idx >= 0) {
        dst_reservoirs[idx] = new_reservoir;

        dst_intermediates[idx].light_sample_nml = intermediates[reuse_idx].light_sample_nml;
        dst_intermediates[idx].light_color = intermediates[reuse_idx].light_color;
    }
    else {
        dst_reservoirs[idx] = reservoirs[idx];
        dst_intermediates[idx] = intermediates[idx];
    }
}

__global__ void computeSpatialReuse(
    idaten::Path* paths,
    const idaten::Reservoir* __restrict__ reservoirs,
    idaten::Reservoir* dst_reservoirs,
    const idaten::ReSTIRIntermedidate* __restrict__ intermediates,
    idaten::ReSTIRIntermedidate* dst_intermediates,
    int width, int height)
{
    auto ix = blockIdx.x * blockDim.x + threadIdx.x;
    auto iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix >= width || iy >= height) {
        return;
    }

    auto idx = getIdx(ix, iy, width);

    OnComputeSpatialReuse(
        idx,
        &paths->sampler[idx],
        reservoirs,
        dst_reservoirs,
        intermediates,
        dst_intermediates,
        width, height
    );
}

namespace idaten {
    int ReSTIRPathTracing::computelReuse(
        int width, int height,
        int bounce)
    {
        dim3 blockPerGrid(((m_tileDomain.w * m_tileDomain.h) + 64 - 1) / 64);
        dim3 threadPerBlock(64);

        int target_idx = 0;

        if (bounce == 0) {
            computeSpatialReuse << <blockPerGrid, threadPerBlock, 0, m_stream >> > (
                m_paths.ptr(),
                m_reservoirs[0].ptr(),
                m_reservoirs[1].ptr(),
                m_intermediates[0].ptr(),
                m_intermediates[1].ptr(),
                width, height);

            checkCudaKernel(computeSpatialReuse);

            target_idx = 1;
        }

        return target_idx;
    }
}
