#include "hip/hip_runtime.h"
#include "svgf/svgf.h"

#include "kernel/pt_common.h"

#include "cuda/cudadefs.h"
#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

#include "aten4idaten.h"

//#define ENABLE_MEDIAN_FILTER

inline __device__ void computePrevScreenPos(
    int32_t ix, int32_t iy,
    float centerDepth,
    int32_t width, int32_t height,
    aten::vec4* prevPos,
    const aten::mat4* __restrict__ mtxs)
{
    // NOTE
    // Pview = (Xview, Yview, Zview, 1)
    // mtxV2C = W 0 0  0
    //          0 H 0  0
    //          0 0 A  B
    //          0 0 -1 0
    // mtxV2C * Pview = (Xclip, Yclip, Zclip, Wclip) = (Xclip, Yclip, Zclip, Zview)
    //  Wclip = Zview = depth
    // Xscr = Xclip / Wclip = Xclip / Zview = Xclip / depth
    // Yscr = Yclip / Wclip = Yclip / Zview = Yclip / depth
    //
    // Xscr * depth = Xclip
    // Xview = mtxC2V * Xclip

    const aten::mat4 mtxC2V = mtxs[0];
    const aten::mat4 mtxV2W = mtxs[1];
    const aten::mat4 mtxPrevW2V = mtxs[2];
    const aten::mat4 mtxV2C = mtxs[3];

    float2 uv = make_float2(ix + 0.5, iy + 0.5);
    uv /= make_float2(width - 1, height - 1);    // [0, 1]
    uv = uv * 2.0f - 1.0f;    // [0, 1] -> [-1, 1]

    aten::vec4 pos(uv.x, uv.y, 0, 0);

    // Screen-space -> Clip-space.
    pos.x *= centerDepth;
    pos.y *= centerDepth;

    // Clip-space -> View-space
    pos = mtxC2V.apply(pos);
    pos.z = -centerDepth;
    pos.w = 1.0;

    pos = mtxV2W.apply(pos);

    // Reproject previous screen position.
    pos = mtxPrevW2V.apply(pos);
    *prevPos = mtxV2C.apply(pos);
    *prevPos /= prevPos->w;

    *prevPos = *prevPos * 0.5 + 0.5;    // [-1, 1] -> [0, 1]
}

inline __device__ int32_t getLinearIdx(int32_t x, int32_t y, int32_t w, int32_t h)
{
    int32_t max_buffer_size = w * h;
    return clamp(y * w + x, 0, max_buffer_size - 1);
}

// Bilinear sampler
inline __device__ float4 sampleBilinear(
    const float4* buffer,
    float uvx, float uvy,
    int32_t w, int32_t h)
{
    float2 uv = make_float2(uvx, uvy) * make_float2(w, h) - make_float2(0.5f, 0.5f);

    int32_t x = floor(uv.x);
    int32_t y = floor(uv.y);

    float2 uv_ratio = uv - make_float2(x, y);
    float2 uv_inv = make_float2(1.f, 1.f) - uv_ratio;

    int32_t x1 = clamp(x + 1, 0, w - 1);
    int32_t y1 = clamp(y + 1, 0, h - 1);

    float4 r = (buffer[getLinearIdx(x, y, w, h)] * uv_inv.x + buffer[getLinearIdx(x1, y, w, h)] * uv_ratio.x) * uv_inv.y +
        (buffer[getLinearIdx(x, y1, w, h)] * uv_inv.x + buffer[getLinearIdx(x1, y1, w, h)] * uv_ratio.x) * uv_ratio.y;

    return r;
}

__global__ void temporalReprojection(
    idaten::TileDomain tileDomain,
    const float nThreshold,
    const float zThreshold,
    const float4* __restrict__ contribs,
    const aten::CameraParameter* __restrict__ camera,
    float4* curAovNormalDepth,
    float4* curAovTexclrMeshid,
    float4* curAovColorVariance,
    float4* curAovMomentTemporalWeight,
    const float4* __restrict__ prevAovNormalDepth,
    const float4* __restrict__ prevAovTexclrMeshid,
    const float4* __restrict__ prevAovColorVariance,
    const float4* __restrict__ prevAovMomentTemporalWeight,
    hipSurfaceObject_t motionDetphBuffer,
    hipSurfaceObject_t dst,
    int32_t width, int32_t height)
{
    int32_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix >= tileDomain.w || iy >= tileDomain.h) {
        return;
    }

    ix += tileDomain.x;
    iy += tileDomain.y;

    const auto idx = getIdx(ix, iy, width);

    auto nmlDepth = curAovNormalDepth[idx];
    auto texclrMeshId = curAovTexclrMeshid[idx];

    const float centerDepth = nmlDepth.w;
    const int32_t centerMeshId = (int32_t)texclrMeshId.w;

    // 今回のフレームのピクセルカラー.
    auto contrib = contribs[idx];
    float4 curColor = make_float4(contrib.x, contrib.y, contrib.z, 1.0f) / contrib.w;
    //curColor.w = 1;

    if (centerMeshId < 0) {
        // 背景なので、そのまま出力して終わり.
        surf2Dwrite(
            curColor,
            dst,
            ix * sizeof(float4), iy,
            hipBoundaryModeTrap);

        curAovColorVariance[idx] = curColor;
        curAovMomentTemporalWeight[idx] = make_float4(1, 1, 1, curAovMomentTemporalWeight[idx].w);

        return;
    }

    float3 centerNormal = make_float3(nmlDepth.x, nmlDepth.y, nmlDepth.z);

    float4 sum = make_float4(0);
    float weight = 0.0f;

    aten::vec4 centerPrevPos;

#pragma unroll
    for (int32_t y = -1; y <= 1; y++) {
        for (int32_t x = -1; x <= 1; x++) {
            int32_t xx = clamp(ix + x, 0, width - 1);
            int32_t yy = clamp(iy + y, 0, height - 1);

            float4 motionDepth;
            surf2Dread(&motionDepth, motionDetphBuffer, ix * sizeof(float4), iy);

            // 前のフレームのスクリーン座標.
            int32_t px = (int32_t)(xx + motionDepth.x * width);
            int32_t py = (int32_t)(yy + motionDepth.y * height);

            px = clamp(px, 0, width - 1);
            py = clamp(py, 0, height - 1);

            int32_t pidx = getIdx(px, py, width);

            nmlDepth = prevAovNormalDepth[pidx];
            texclrMeshId = prevAovTexclrMeshid[pidx];

            const float prevDepth = nmlDepth.w;
            const int32_t prevMeshId = (int32_t)texclrMeshId.w;
            float3 prevNormal = make_float3(nmlDepth.x, nmlDepth.y, nmlDepth.z);

            // TODO
            // 同じメッシュ上でもライトのそばの明るくなったピクセルを拾ってしまう場合の対策が必要.

            float Wz = clamp((zThreshold - abs(1 - centerDepth / prevDepth)) / zThreshold, 0.0f, 1.0f);
            float Wn = clamp((dot(centerNormal, prevNormal) - nThreshold) / (1.0f - nThreshold), 0.0f, 1.0f);
            float Wm = centerMeshId == prevMeshId ? 1.0f : 0.0f;

            // 前のフレームのピクセルカラーを取得.
            float4 prev = prevAovColorVariance[pidx];
            //float4 prev = sampleBilinear(prevAovColorVariance, prevPos.x, prevPos.y, width, height);

            float W = Wz * Wn * Wm;
            sum += prev * W;
            weight += W;
        }
    }

    if (weight > 0.0f) {
        sum /= weight;
        weight /= 9;
#if 0
        auto w = min(0.8f, weight);
        curColor = (1.0f - w) * curColor + w * sum;
#elif 1
        curColor = 0.2 * curColor + 0.8 * sum;
#else
        curColor = (1.0f - weight) * curColor + weight * sum;
#endif
    }

    curAovMomentTemporalWeight[idx].w = weight;

#ifdef ENABLE_MEDIAN_FILTER
    curAovColorVariance[idx].x = curColor.x;
    curAovColorVariance[idx].y = curColor.y;
    curAovColorVariance[idx].z = curColor.z;
#else
    curAovColorVariance[idx].x = curColor.x;
    curAovColorVariance[idx].y = curColor.y;
    curAovColorVariance[idx].z = curColor.z;

    // TODO
    // 現フレームと過去フレームが同率で加算されるため、どちらかに強い影響がでると影響が弱まるまでに非常に時間がかかる.
    // ex)
    // f0 = 100, f1 = 0, f2 = 0
    // avg = (f0 + f1 + f2) / 3 = 33.3 <- 非常に大きい値が残り続ける.

    // accumulate moments.
    {
        float lum = AT_NAME::color::luminance(curColor.x, curColor.y, curColor.z);
        float3 centerMoment = make_float3(lum * lum, lum, 0);

        // 積算フレーム数のリセット.
        int32_t frame = 1;

        if (weight > 0.0f) {
            auto momentTemporalWeight = prevAovMomentTemporalWeight[idx];;
            float3 prevMoment = make_float3(momentTemporalWeight.x, momentTemporalWeight.y, momentTemporalWeight.z);

            // 積算フレーム数を１増やす.
            frame = (int32_t)prevMoment.z + 1;

            centerMoment += prevMoment;
        }

        centerMoment.z = frame;

        curAovMomentTemporalWeight[idx].x = centerMoment.x;
        curAovMomentTemporalWeight[idx].y = centerMoment.y;
        curAovMomentTemporalWeight[idx].z = centerMoment.z;
    }
#endif

    surf2Dwrite(
        curColor,
        dst,
        ix * sizeof(float4), iy,
        hipBoundaryModeTrap);
}

__global__ void dilateWeight(
    idaten::TileDomain tileDomain,
    float4* aovMomentTemporalWeight,
    const float4* __restrict__ aovTexclrMeshid,
    int32_t width, int32_t height)
{
    int32_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix >= tileDomain.w || iy >= tileDomain.h) {
        return;
    }

    ix += tileDomain.x;
    iy += tileDomain.y;

    auto idx = getIdx(ix, iy, width);

    const int32_t centerMeshId = (int32_t)aovTexclrMeshid[idx].w;

    if (centerMeshId < 0) {
        // This pixel is background, so nothing is done.
        return;
    }

    float temporalWeight = aovMomentTemporalWeight[idx].w;

    for (int32_t y = -1; y <= 1; y++) {
        for (int32_t x = -1; x <= 1; x++) {
            int32_t xx = ix + x;
            int32_t yy = iy + y;

            if ((0 <= xx) && (xx < width)
                && (0 <= yy) && (yy < height))
            {
                int32_t pidx = getIdx(xx, yy, width);
                float w = aovMomentTemporalWeight[pidx].w;
                temporalWeight = min(temporalWeight, w);
            }
        }
    }

    aovMomentTemporalWeight[idx].w = temporalWeight;
}

inline __device__ float3 min(float3 a, float3 b)
{
    return make_float3(
        min(a.x, b.x),
        min(a.y, b.y),
        min(a.z, b.z));
}

inline __device__ float3 max(float3 a, float3 b)
{
    return make_float3(
        max(a.x, b.x),
        max(a.y, b.y),
        max(a.z, b.z));
}

// Macro for sorting.
#define s2(a, b)                temp = a; a = min(a, b); b = max(temp, b);
#define mn3(a, b, c)            s2(a, b); s2(a, c);
#define mx3(a, b, c)            s2(b, c); s2(a, c);

#define mnmx3(a, b, c)            mx3(a, b, c); s2(a, b);                                   // 3 exchanges
#define mnmx4(a, b, c, d)        s2(a, b); s2(c, d); s2(a, c); s2(b, d);                   // 4 exchanges
#define mnmx5(a, b, c, d, e)    s2(a, b); s2(c, d); mn3(a, c, e); mx3(b, d, e);           // 6 exchanges
#define mnmx6(a, b, c, d, e, f) s2(a, d); s2(b, e); s2(c, f); mn3(a, b, c); mx3(d, e, f); // 7 exchanges

inline __device__ float3 medianFilter(
    int32_t ix, int32_t iy,
    const float4* src,
    int32_t width, int32_t height)
{
    float3 v[9];

    int32_t pos = 0;

    for (int32_t y = -1; y <= 1; y++) {
        for (int32_t x = -1; x <= 1; x++) {
            int32_t xx = clamp(ix + x, 0, width - 1);
            int32_t yy = clamp(iy + y, 0, height - 1);

            int32_t pidx = getIdx(xx, yy, width);

            auto s = src[pidx];
            v[pos] = make_float3(s.x, s.y, s.z);

            pos++;
        }
    }

    // Sort
    float3 temp;
    mnmx6(v[0], v[1], v[2], v[3], v[4], v[5]);
    mnmx5(v[1], v[2], v[3], v[4], v[6]);
    mnmx4(v[2], v[3], v[4], v[7]);
    mnmx3(v[3], v[4], v[8]);

    return v[4];
}

__global__ void medianFilter(
    hipSurfaceObject_t dst,
    float4* curAovColorVariance,
    float4* curAovMomentTemporalWeight,
    const float4* __restrict__ curAovTexclrMeshid,
    const float4* __restrict__ prevAovMomentTemporalWeight,
    int32_t width, int32_t height)
{
    int32_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix >= width || iy >= height) {
        return;
    }

    auto idx = getIdx(ix, iy, width);

    const int32_t centerMeshId = curAovTexclrMeshid[idx].w;

    if (centerMeshId < 0) {
        // This pixel is background, so nothing is done.
        return;
    }

    auto curColor = medianFilter(ix, iy, curAovColorVariance, width, height);

    curAovColorVariance[idx].x = curColor.x;
    curAovColorVariance[idx].y = curColor.y;
    curAovColorVariance[idx].z = curColor.z;

    // accumulate moments.
    {
        float lum = AT_NAME::color::luminance(curColor.x, curColor.y, curColor.z);
        float3 centerMoment = make_float3(lum * lum, lum, 0);

        // 積算フレーム数のリセット.
        int32_t frame = 1;

        auto momentTemporalWeight = prevAovMomentTemporalWeight[idx];;
        float3 prevMoment = make_float3(momentTemporalWeight.x, momentTemporalWeight.y, momentTemporalWeight.z);

        // 積算フレーム数を１増やす.
        frame = (int32_t)prevMoment.z + 1;

        centerMoment += prevMoment;

        centerMoment.z = frame;

        curAovMomentTemporalWeight[idx].x = centerMoment.x;
        curAovMomentTemporalWeight[idx].y = centerMoment.y;
        curAovMomentTemporalWeight[idx].z = centerMoment.z;
    }

    surf2Dwrite(
        make_float4(curColor, 0),
        dst,
        ix * sizeof(float4), iy,
        hipBoundaryModeTrap);
}

namespace idaten
{
    void SVGFPathTracing::onTemporalReprojection(
        hipSurfaceObject_t outputSurf,
        int32_t width, int32_t height)
    {
        dim3 block(BLOCK_SIZE, BLOCK_SIZE);
        dim3 grid(
            (m_tileDomain.w + block.x - 1) / block.x,
            (m_tileDomain.h + block.y - 1) / block.y);

        int32_t curaov_idx = getCurAovs();
        auto& curaov = aov_[curaov_idx];

        int32_t prevaov_idx = getPrevAovs();
        auto& prevaov = aov_[prevaov_idx];

        CudaGLResourceMapper<decltype(m_motionDepthBuffer)> rscmap(m_motionDepthBuffer);
        auto motionDepthBuffer = m_motionDepthBuffer.bind();

        temporalReprojection << <grid, block, 0, m_stream >> > (
        //temporalReprojection << <1, 1 >> > (
            m_tileDomain,
            m_nmlThresholdTF,
            m_depthThresholdTF,
            m_tmpBuf.ptr(),
            m_cam.ptr(),
            curaov.get<AOVBuffer::NormalDepth>().ptr(),
            curaov.get<AOVBuffer::AlbedoMeshId>().ptr(),
            curaov.get<AOVBuffer::ColorVariance>().ptr(),
            curaov.get<AOVBuffer::MomentTemporalWeight>().ptr(),
            prevaov.get<AOVBuffer::NormalDepth>().ptr(),
            prevaov.get<AOVBuffer::AlbedoMeshId>().ptr(),
            prevaov.get<AOVBuffer::ColorVariance>().ptr(),
            prevaov.get<AOVBuffer::MomentTemporalWeight>().ptr(),
            motionDepthBuffer,
            outputSurf,
            width, height);

        checkCudaKernel(temporalReprojection);

#ifdef ENABLE_MEDIAN_FILTER
        medianFilter << <grid, block, 0, m_stream >> > (
            outputSurf,
            m_aovColorVariance[curaov].ptr(),
            m_aovMomentTemporalWeight[curaov].ptr(),
            m_aovTexclrMeshid[curaov].ptr(),
            m_aovMomentTemporalWeight[prevaov].ptr(),
            width, height);

        checkCudaKernel(medianFilter);
#endif

        dilateWeight << <grid, block, 0, m_stream >> > (
            m_tileDomain,
            curaov.get<AOVBuffer::MomentTemporalWeight>().ptr(),
            curaov.get<AOVBuffer::AlbedoMeshId>().ptr(),
            width, height);
        checkCudaKernel(dilateWeight);
    }
}
