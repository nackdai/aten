#include "hip/hip_runtime.h"
#include "svgf/svgf.h"

#include "kernel/pt_common.h"

#include "cuda/cudadefs.h"
#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

#include "aten4idaten.h"

__global__ void varianceEstimation(
    idaten::TileDomain tileDomain,
    hipSurfaceObject_t dst,
    const float4* __restrict__ aovNormalDepth,
    float4* aovMomentTemporalWeight,
    float4* aovColorVariance,
    float4* aovTexclrMeshid,
    aten::mat4 mtxC2V,
    int width, int height,
    float cameraDistance)
{
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix >= tileDomain.w || iy >= tileDomain.h) {
        return;
    }

    ix += tileDomain.x;
    iy += tileDomain.y;

    const int idx = getIdx(ix, iy, width);

    auto normalDepth = aovNormalDepth[idx];
    auto texclrMeshid = aovTexclrMeshid[idx];
    auto momentTemporalWeight = aovMomentTemporalWeight[idx];
    auto centerColor = aovColorVariance[idx];

    float centerDepth = aovNormalDepth[idx].w;
    int centerMeshId = (int)texclrMeshid.w;

    if (centerMeshId < 0) {
        // 背景なので、分散はゼロ.
        aovMomentTemporalWeight[idx].x = 0;
        aovMomentTemporalWeight[idx].y = 0;
        aovMomentTemporalWeight[idx].z = 1;

        surf2Dwrite(
            make_float4(0),
            dst,
            ix * sizeof(float4), iy,
            hipBoundaryModeTrap);
    }

    float pixelDistanceRatio = (centerDepth / cameraDistance) * height;

    float3 centerMoment = make_float3(momentTemporalWeight.x, momentTemporalWeight.y, momentTemporalWeight.z);

    int frame = (int)centerMoment.z;

    centerMoment /= centerMoment.z;

    float var = 0.0f;
    float4 color = centerColor;

    if (frame < 4) {
        // 積算フレーム数が４未満 or Disoccludedされている.
        // 7x7birateral filterで輝度を計算.

        float3 centerNormal = make_float3(normalDepth.x, normalDepth.y, normalDepth.z);

        float3 momentSum = make_float3(centerMoment.x, centerMoment.y, centerMoment.z);
        float weight = 1.0f;

        float radius = frame > 1 ? 2 : 3;

        for (int v = -radius; v <= radius; v++)
        {
            for (int u = -radius; u <= radius; u++)
            {
                if (u != 0 || v != 0) {
                    int xx = clamp(ix + u, 0, width - 1);
                    int yy = clamp(iy + v, 0, height - 1);

                    int pidx = getIdx(xx, yy, width);
                    normalDepth = aovNormalDepth[pidx];
                    texclrMeshid = aovTexclrMeshid[pidx];
                    momentTemporalWeight = aovMomentTemporalWeight[pidx];

                    float3 sampleNml = make_float3(normalDepth.x, normalDepth.y, normalDepth.z);
                    float sampleDepth = normalDepth.w;
                    int sampleMeshId = (int)texclrMeshid.w;
                    auto sampleColor = aovColorVariance[pidx];

                    float3 moment = make_float3(momentTemporalWeight.x, momentTemporalWeight.y, momentTemporalWeight.z);
                    moment /= moment.z;

                    float Wz = aten::abs(sampleDepth - centerDepth) / (pixelDistanceRatio * length(make_float2(u, v)) + 1e-2);
                    float Wn = aten::pow(aten::cmpMax(0.0f, dot(sampleNml, centerNormal)), 128.0f);

                    float Wm = centerMeshId == sampleMeshId ? 1.0f : 0.0f;

                    float W = exp(-Wz) * Wn * Wm;

                    momentSum += moment * W;
                    color += sampleColor * W;
                    weight += W;
                }
            }
        }

        momentSum /= weight;
        color /= weight;

        var = 1.0f + 3.0f * (1.0f - frame / 4.0f) * max(0.0, momentSum.y - momentSum.x * momentSum.x);
    }
    else {
        var = max(0.0f, centerMoment.x - centerMoment.y * centerMoment.y);
    }

    color.w = var;
    aovColorVariance[idx] = color;

    surf2Dwrite(
        make_float4(var, var, var, 1),
        dst,
        ix * sizeof(float4), iy,
        hipBoundaryModeTrap);
}

namespace idaten
{
    void SVGFPathTracing::onVarianceEstimation(
        hipSurfaceObject_t outputSurf,
        int width, int height)
    {
        dim3 block(BLOCK_SIZE, BLOCK_SIZE);
        dim3 grid(
            (m_tileDomain.w + block.x - 1) / block.x,
            (m_tileDomain.h + block.y - 1) / block.y);

        float cameraDistance = height / (2.0f * aten::tan(0.5f * m_camParam.vfov));

        int curaov_idx = getCurAovs();
        auto& curaov = aov_[curaov_idx];

        varianceEstimation << <grid, block, 0, m_stream >> > (
        //varianceEstimation << <1, 1 >> > (
            m_tileDomain,
            outputSurf,
            curaov.get<AOVBuffer::NormalDepth>().ptr(),
            curaov.get<AOVBuffer::MomentTemporalWeight>().ptr(),
            curaov.get<AOVBuffer::ColorVariance>().ptr(),
            curaov.get<AOVBuffer::AlbedoMeshId>().ptr(),
            m_mtxC2V,
            width, height,
            cameraDistance);

        checkCudaKernel(varianceEstimation);
    }
}
