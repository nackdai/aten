#include "hip/hip_runtime.h"
#include "kernel/raytracing.h"
#include "kernel/context.cuh"
#include "kernel/light.cuh"
#include "kernel/material.cuh"
#include "kernel/intersect.cuh"
#include "kernel/accelerator.cuh"

#include <hip/hip_runtime.h>
#include <>

#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

#include "aten4idaten.h"

struct ShadowRay : public aten::ray {
	real distToLight;
	int targetLightId;

	struct {
		uint32_t isActive : 1;
	};
};

struct Path {
	aten::ray ray;
	aten::vec3 throughput;
	aten::Intersection isect;
	bool isHit;
	bool isTerminate;
};

__global__ void genPathRayTracing(
	Path* paths,
	int width, int height,
	aten::CameraParameter* camera)
{
	const auto ix = blockIdx.x * blockDim.x + threadIdx.x;
	const auto iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= width && iy >= height) {
		return;
	}

	const auto idx = iy * camera->width + ix;

	float s = (ix + 0.5f) / (float)(camera->width - 1);
	float t = (iy + 0.5f) / (float)(camera->height - 1);

	AT_NAME::CameraSampleResult camsample;
	AT_NAME::PinholeCamera::sample(&camsample, camera, s, t);

	auto& path = paths[idx];

	path.ray = camsample.r;
	path.throughput = aten::vec3(1);
	path.isHit = false;
	path.isTerminate = false;
}

__global__ void hitTestRayTracing(
	Path* paths,
	int width, int height,
	aten::GeomParameter* shapes, int geomnum,
	aten::MaterialParameter* mtrls,
	aten::LightParameter* lights, int lightnum,
	hipTextureObject_t* nodes,
	aten::PrimitiveParamter* prims,
	hipTextureObject_t vtxPos,
	aten::mat4* matrices)
{
	const auto ix = blockIdx.x * blockDim.x + threadIdx.x;
	const auto iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= width && iy >= height) {
		return;
	}

	const auto idx = iy * width + ix;

	auto& path = paths[idx];
	path.isHit = false;

	if (path.isTerminate) {
		return;
	}

	Context ctxt;
	{
		ctxt.geomnum = geomnum;
		ctxt.shapes = shapes;
		ctxt.mtrls = mtrls;
		ctxt.lightnum = lightnum;
		ctxt.lights = lights;
		ctxt.nodes = nodes;
		ctxt.prims = prims;
		ctxt.vtxPos = vtxPos;
		ctxt.matrices = matrices;
	}

	aten::Intersection isect;

	bool isHit = intersectClosest(&ctxt, path.ray, &isect);

	path.isect.t = isect.t;
	path.isect.objid = isect.objid;
	path.isect.mtrlid = isect.mtrlid;
	path.isect.primid = isect.primid;
	path.isect.a = isect.a;
	path.isect.b = isect.b;

	path.isHit = isHit;
}

__global__ void raytracing(
	hipSurfaceObject_t outSurface,
	Path* paths,
	ShadowRay* shadowRays,
	int width, int height,
	aten::GeomParameter* shapes, int geomnum,
	aten::MaterialParameter* mtrls,
	aten::LightParameter* lights, int lightnum,
	hipTextureObject_t* nodes,
	aten::PrimitiveParamter* prims,
	hipTextureObject_t vtxPos,
	hipTextureObject_t vtxNml,
	aten::mat4* matrices)
{
	const auto ix = blockIdx.x * blockDim.x + threadIdx.x;
	const auto iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= width && iy >= height) {
		return;
	}

	Context ctxt;
	{
		ctxt.geomnum = geomnum;
		ctxt.shapes = shapes;
		ctxt.mtrls = mtrls;
		ctxt.lightnum = lightnum;
		ctxt.lights = lights;
		ctxt.nodes = nodes;
		ctxt.prims = prims;
		ctxt.vtxPos = vtxPos;
		ctxt.vtxNml = vtxNml;
		ctxt.matrices = matrices;
	}

	const auto idx = iy * width + ix;

	auto& path = paths[idx];

	shadowRays[idx].isActive = false;

	if (!path.isHit) {
		return;
	}
	if (path.isTerminate) {
		return;
	}

	aten::vec3 contrib = aten::vec3(0);

	const aten::MaterialParameter* mtrl = &ctxt.mtrls[path.isect.mtrlid];

	if (mtrl->attrib.isEmissive) {
		contrib = path.throughput * mtrl->baseColor;

		path.isTerminate = true;
		//p[idx] = make_float4(contrib.x, contrib.y, contrib.z, 1);
		surf2Dwrite(make_float4(contrib.x, contrib.y, contrib.z, 1), outSurface, ix * sizeof(float4), iy, hipBoundaryModeTrap);

		return;
	}

	aten::hitrecord rec;

	auto obj = &ctxt.shapes[path.isect.objid];
	evalHitResult(&ctxt, obj, path.ray, &rec, &path.isect);

	// 交差位置の法線.
	// 物体からのレイの入出を考慮.
	const aten::vec3 orienting_normal = dot(rec.normal, path.ray.dir) < 0.0 ? rec.normal : -rec.normal;

	if (mtrl->attrib.isSingular || mtrl->attrib.isTranslucent) {
		AT_NAME::MaterialSampling sampling;

		sampleMaterial(
			&sampling,
			&ctxt,
			mtrl,
			orienting_normal,
			path.ray.dir,
			rec.normal,
			nullptr,
			rec.u, rec.v);

		auto nextDir = normalize(sampling.dir);

		path.throughput *= sampling.bsdf;

		// Make next ray.
		path.ray = aten::ray(rec.p, nextDir);
	}
	else {
		// TODO
		int lightidx = 0;

		auto light = lights[lightidx];

		aten::LightSampleResult sampleres;
		sampleLight(&sampleres, &ctxt, &light, rec.p, orienting_normal, nullptr);

		aten::vec3 dirToLight = sampleres.dir;
		auto len = dirToLight.length();

		dirToLight = normalize(dirToLight);

		shadowRays[idx].isActive = true;
		shadowRays[idx].org = rec.p;
		shadowRays[idx].dir = dirToLight;
		shadowRays[idx].distToLight = len;
		shadowRays[idx].targetLightId = lightidx;

		aten::hitrecord tmpRec;

		if (light.attrib.isInfinite) {
			len = 1.0f;
		}

		const auto c0 = max(0.0f, dot(orienting_normal, dirToLight));
		float c1 = 1.0f;

		if (!light.attrib.isSingular) {
			c1 = max(0.0f, dot(sampleres.nml, -dirToLight));
		}

		auto G = c0 * c1 / (len * len);

		path.throughput = path.throughput * (mtrl->baseColor * sampleres.finalColor) * G;
	}
}

__global__ void hitShadowRay(
	hipSurfaceObject_t outSurface,
	Path* paths,
	ShadowRay* shadowRays,
	int width, int height,
	aten::GeomParameter* shapes, int geomnum,
	aten::MaterialParameter* mtrls,
	aten::LightParameter* lights, int lightnum,
	hipTextureObject_t* nodes,
	aten::PrimitiveParamter* prims,
	hipTextureObject_t vtxPos,
	aten::mat4* matrices)
{
	const auto ix = blockIdx.x * blockDim.x + threadIdx.x;
	const auto iy = blockIdx.y * blockDim.y + threadIdx.y;

	if (ix >= width && iy >= height) {
		return;
	}

	const auto idx = iy * width + ix;

	auto& path = paths[idx];

	if (path.isTerminate) {
		return;
	}

	Context ctxt;
	{
		ctxt.geomnum = geomnum;
		ctxt.shapes = shapes;
		ctxt.mtrls = mtrls;
		ctxt.lightnum = lightnum;
		ctxt.lights = lights;
		ctxt.nodes = nodes;
		ctxt.prims = prims;
		ctxt.vtxPos = vtxPos;
		ctxt.matrices = matrices;
	}

	auto& shadowRay = shadowRays[idx];

	if (shadowRay.isActive) {
		auto& path = paths[idx];

		aten::Intersection isect;
		bool isHit = intersectClosest(&ctxt, shadowRay, &isect);

		real distHitObjToRayOrg = AT_MATH_INF;
		const aten::GeomParameter* hitobj = nullptr;

		aten::hitrecord rec;

		if (isHit) {
			hitobj = &ctxt.shapes[isect.objid];

#if 0
			evalHitResult(&ctxt, hitobj, shadowRay, &rec, &isect);

			distHitObjToRayOrg = (rec.p - shadowRay.org).length();
#endif
		}

		auto light = &ctxt.lights[shadowRay.targetLightId];
		auto lightobj = (light->objid >= 0 ? &ctxt.shapes[light->objid] : nullptr);

		shadowRay.isActive = AT_NAME::scene::hitLight(
			isHit,
			light->attrib,
			lightobj,
			shadowRay.distToLight,
			distHitObjToRayOrg,
			isect.t,
			hitobj);

		if (shadowRay.isActive) {
			path.isTerminate = true;

			auto contrib = path.throughput;

			//p[idx] = make_float4(contrib.x, contrib.y, contrib.z, 1);
			surf2Dwrite(
				make_float4(contrib.x, contrib.y, contrib.z, 1), outSurface, ix * sizeof(float4), iy, hipBoundaryModeTrap);
		}
	}
}


namespace idaten {
	void RayTracing::prepare()
	{
	}

	void RayTracing::render(
		int width, int height,
		int maxSamples,
		int maxBounce)
	{
		dim3 block(16, 16);
		dim3 grid(
			(width + block.x - 1) / block.x,
			(height + block.y - 1) / block.y);

		int bounce = 0;

		idaten::TypedCudaMemory<Path> paths;
		paths.init(width * height);

		CudaGLResourceMap rscmap(&m_glimg);
		auto outputSurf = m_glimg.bind();

		auto vtxTexPos = m_vtxparamsPos.bind();
		auto vtxTexNml = m_vtxparamsNml.bind();

		std::vector<hipTextureObject_t> tmp;
		for (int i = 0; i < m_nodeparam.size(); i++) {
			auto nodeTex = m_nodeparam[i].bind();
			tmp.push_back(nodeTex);
		}
		m_nodetex.writeByNum(&tmp[0], tmp.size());

		idaten::TypedCudaMemory<ShadowRay> shadowRays;
		shadowRays.init(width * height);

		genPathRayTracing << <grid, block >> > (
			paths.ptr(),
			width, height,
			m_cam.ptr());

		//checkCudaErrors(hipDeviceSynchronize());

		while (bounce < 5) {
			hitTestRayTracing << <grid, block >> > (
			//hitTestRayTracing << <1, 1 >> > (
				paths.ptr(),
				width, height,
				m_shapeparam.ptr(), m_shapeparam.num(),
				m_mtrlparam.ptr(),
				m_lightparam.ptr(), m_lightparam.num(),
				m_nodetex.ptr(),
				m_primparams.ptr(),
				vtxTexPos,
				m_mtxparams.ptr());

			auto err = hipGetLastError();
			if (err != hipSuccess) {
				AT_PRINTF("Cuda Kernel Err(hitTest) [%s]\n", hipGetErrorString(err));
			}

			//checkCudaErrors(hipDeviceSynchronize());

			raytracing << <grid, block >> > (
				outputSurf,
				paths.ptr(),
				shadowRays.ptr(),
				width, height,
				m_shapeparam.ptr(), m_shapeparam.num(),
				m_mtrlparam.ptr(),
				m_lightparam.ptr(), m_lightparam.num(),
				m_nodetex.ptr(),
				m_primparams.ptr(),
				vtxTexPos, vtxTexNml,
				m_mtxparams.ptr());

			err = hipGetLastError();
			if (err != hipSuccess) {
				AT_PRINTF("Cuda Kernel Err(raytracing) [%s]\n", hipGetErrorString(err));
			}

			hitShadowRay << <grid, block >> > (
				//hitShadowRay << <1, 1 >> > (
				outputSurf,
				paths.ptr(),
				shadowRays.ptr(),
				width, height,
				m_shapeparam.ptr(), m_shapeparam.num(),
				m_mtrlparam.ptr(),
				m_lightparam.ptr(), m_lightparam.num(),
				m_nodetex.ptr(),
				m_primparams.ptr(),
				vtxTexPos,
				m_mtxparams.ptr());

			err = hipGetLastError();
			if (err != hipSuccess) {
				AT_PRINTF("Cuda Kernel Err(hitShadowRay) [%s]\n", hipGetErrorString(err));
			}

			//checkCudaErrors(hipDeviceSynchronize());

			bounce++;
		}

		checkCudaErrors(hipDeviceSynchronize());

		m_vtxparamsPos.unbind();
		m_vtxparamsNml.unbind();
		for (int i = 0; i < m_nodeparam.size(); i++) {
			m_nodeparam[i].unbind();
		}
		m_nodetex.reset();

		//dst.read(image, sizeof(aten::vec4) * width * height);
	}
}
