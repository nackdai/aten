#include "hip/hip_runtime.h"
#include "restir/restir.h"

#include "kernel/pt_common.h"
#include "kernel/device_scene_context.cuh"

#include "cuda/cudadefs.h"
#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

#include "aten4idaten.h"
#include "light/light_impl.h"
#include "renderer/pathtracing/pt_params.h"
#include "renderer/restir/restir_impl.h"

__global__ void computeTemporalReuse(
    idaten::Path paths,
    idaten::context ctxt,
    const aten::ObjectParameter* __restrict__ shapes,
    const aten::MaterialParameter* __restrict__ mtrls,
    const aten::LightParameter* __restrict__ lights,
    const aten::TriangleParameter* __restrict__ prims,
    const aten::mat4* __restrict__ matrices,
    const float4* __restrict__ aovTexclrMeshid,
    idaten::Reservoir* reservoirs,
    const idaten::Reservoir* __restrict__ prev_reservoirs,
    const idaten::ReSTIRInfo* __restrict__ infos,
    hipSurfaceObject_t motionDetphBuffer,
    int32_t width, int32_t height)
{
    int32_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix >= width || iy >= height) {
        return;
    }

    auto idx = getIdx(ix, iy, width);

    if (paths.attrib[idx].isTerminate) {
        return;
    }

    ctxt.shapes = shapes;
    ctxt.mtrls = mtrls;
    ctxt.lights = lights;
    ctxt.prims = prims;
    ctxt.matrices = matrices;

    auto& sampler = paths.sampler[idx];

    const auto size = width * height;

    aten::const_span prev_reservoirs_as_span(prev_reservoirs, size);
    aten::const_span resitr_infos(infos, size);
    aten::const_span aov_texclr_meshid(aovTexclrMeshid, size);

    AT_NAME::restir::ApplyTemporalReuse(
        ix, iy,
        width, height,
        ctxt,
        sampler,
        reservoirs[idx],
        resitr_infos[idx],
        prev_reservoirs_as_span,
        resitr_infos,
        aov_texclr_meshid, motionDetphBuffer);
}

__global__ void computeSpatialReuse(
    idaten::Path paths,
    idaten::context ctxt,
    const aten::ObjectParameter* __restrict__ shapes,
    const aten::MaterialParameter* __restrict__ mtrls,
    const aten::LightParameter* __restrict__ lights,
    const aten::TriangleParameter* __restrict__ prims,
    const aten::mat4* __restrict__ matrices,
    const float4* __restrict__ aovTexclrMeshid,
    const idaten::Reservoir* __restrict__ reservoirs,
    idaten::Reservoir* dst_reservoirs,
    const idaten::ReSTIRInfo* __restrict__ infos,
    int32_t width, int32_t height)
{
    int32_t ix = blockIdx.x * blockDim.x + threadIdx.x;
    int32_t iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix >= width || iy >= height) {
        return;
    }

    auto idx = getIdx(ix, iy, width);

    if (paths.attrib[idx].isTerminate) {
        return;
    }

    ctxt.shapes = shapes;
    ctxt.mtrls = mtrls;
    ctxt.lights = lights;
    ctxt.prims = prims;
    ctxt.matrices = matrices;

    auto& sampler = paths.sampler[idx];

    const auto size = width * height;

    aten::const_span reservoirs_as_span(reservoirs, size);
    aten::span dst_reservoirs_as_span(dst_reservoirs, size);
    aten::const_span resitr_infos(infos, size);
    aten::const_span aov_texclr_meshid(aovTexclrMeshid, size);

    AT_NAME::restir::ApplySpatialReuse(
        ix, iy,
        width, height,
        ctxt,
        sampler,
        reservoirs_as_span,
        dst_reservoirs_as_span,
        resitr_infos,
        aov_texclr_meshid);
}

namespace idaten {
    int32_t ReSTIRPathTracing::computelReuse(
        int32_t width, int32_t height,
        int32_t bounce)
    {
        dim3 block(BLOCK_SIZE, BLOCK_SIZE);
        dim3 grid(
            (width + block.x - 1) / block.x,
            (height + block.y - 1) / block.y);

        if (bounce == 0) {
            // NOTE
            // temporal reuse �ŗ��p���� previous reservoir ��
            // spatial reuse ������O�̂��̂łȂ��Ƃ����Ȃ�
            // spatial reuse �͂����܂ł����݃t���[���ɑ΂��čs������̂�
            // ���t���[���ɉe����^���Ȃ��悤�ɂ���
            // e.g.
            //  - frame 1
            //     cur:0
            //     prev:N/A (�ŏ��Ȃ̂� temporal �� skip)
            //     spatial_dst:1
            //     pos=0 -> pos=1(for next)
            //  - frame 2
            //     cur:1(=pos)
            //     prev:0
            //     spatial_dst:0
            //     pos=1 -> pos=0(for next)
            //     ���̂Ƃ� prev �͑O�t���[���� cur �ƂȂ��Ă���

            const auto cur_idx = m_curReservoirPos;
            const auto prev_idx = (m_curReservoirPos + 1) & 0x01;
            const auto dst_idx = (m_curReservoirPos + 1) & 0x01;

            m_curReservoirPos = (m_curReservoirPos + 1) & 0x01;
            if (m_restirMode == ReSTIRMode::ReSTIR
                || m_restirMode == ReSTIRMode::TemporalReuse) {
                if (m_frame > 1) {
                    CudaGLResourceMapper<decltype(m_motionDepthBuffer)> rscmap(m_motionDepthBuffer);
                    auto motionDepthBuffer = m_motionDepthBuffer.bind();

                    computeTemporalReuse << <grid, block, 0, m_stream >> > (
                        path_host_->paths,
                        ctxt_host_.ctxt,
                        ctxt_host_.shapeparam.data(),
                        ctxt_host_.mtrlparam.data(),
                        ctxt_host_.lightparam.data(),
                        ctxt_host_.primparams.data(),
                        ctxt_host_.mtxparams.data(),
                        aov_.albedo_meshid().data(),
                        m_reservoirs[cur_idx].data(),
                        m_reservoirs[prev_idx].data(),
                        m_restir_infos.data(),
                        motionDepthBuffer,
                        width, height);

                    checkCudaKernel(computeTemporalReuse);
                }
            }

            if (m_restirMode == ReSTIRMode::ReSTIR
                || m_restirMode == ReSTIRMode::SpatialReuse) {
                computeSpatialReuse << <grid, block, 0, m_stream >> > (
                    path_host_->paths,
                    ctxt_host_.ctxt,
                    ctxt_host_.shapeparam.data(),
                    ctxt_host_.mtrlparam.data(),
                    ctxt_host_.lightparam.data(),
                    ctxt_host_.primparams.data(),
                    ctxt_host_.mtxparams.data(),
                    aov_.albedo_meshid().data(),
                    m_reservoirs[cur_idx].data(),
                    m_reservoirs[dst_idx].data(),
                    m_restir_infos.data(),
                    width, height);

                checkCudaKernel(computeSpatialReuse);

                return dst_idx;
            }
        }

        return m_curReservoirPos;
    }
}
