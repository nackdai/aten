#include "hip/hip_runtime.h"
#include "asvgf/asvgf.h"

#include "kernel/StreamCompaction.h"

#include "kernel/context.cuh"
#include "kernel/light.cuh"
#include "kernel/material.cuh"
#include "kernel/intersect.cuh"
#include "kernel/accelerator.cuh"
#include "kernel/pt_common.h"

#include "cuda/cudadefs.h"
#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

#include "aten4idaten.h"

__global__ void shadeASVGF(
    idaten::TileDomain tileDomain,
    float4* aovNormalDepth,
    float4* aovTexclrMeshid,
    aten::mat4 mtxW2C,
    int width, int height,
    idaten::SVGFPathTracing::Path* paths,
    const int* __restrict__ hitindices,
    int* hitnum,
    const aten::Intersection* __restrict__ isects,
    aten::ray* rays,
    int sample,
    int frame,
    int bounce, int rrBounce,
    const aten::GeomParameter* __restrict__ shapes, int geomnum,
    const aten::MaterialParameter* __restrict__ mtrls,
    const aten::LightParameter* __restrict__ lights, int lightnum,
    const aten::PrimitiveParamter* __restrict__ prims,
    hipTextureObject_t vtxPos,
    hipTextureObject_t vtxNml,
    const aten::mat4* __restrict__ matrices,
    hipTextureObject_t* textures,
    unsigned int* random,
    hipTextureObject_t blueNoise,
    idaten::SVGFPathTracing::ShadowRay* shadowRays)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= *hitnum) {
        return;
    }

    Context ctxt;
    {
        ctxt.geomnum = geomnum;
        ctxt.shapes = shapes;
        ctxt.mtrls = mtrls;
        ctxt.lightnum = lightnum;
        ctxt.lights = lights;
        ctxt.prims = prims;
        ctxt.vtxPos = vtxPos;
        ctxt.vtxNml = vtxNml;
        ctxt.matrices = matrices;
        ctxt.textures = textures;
    }

    idx = hitindices[idx];

    __shared__ idaten::SVGFPathTracing::ShadowRay shShadowRays[64 * idaten::SVGFPathTracing::ShadowRayNum];
    __shared__ aten::MaterialParameter shMtrls[64];

    const auto ray = rays[idx];

#if IDATEN_SAMPLER == IDATEN_SAMPLER_SOBOL
    auto scramble = random[idx] * 0x1fe3434f;
    paths->sampler[idx].init(frame + sample, 4 + bounce * 300, scramble);
#elif IDATEN_SAMPLER == IDATEN_SAMPLER_CMJ
    auto rnd = random[idx];
    auto scramble = rnd * 0x1fe3434f
        * (((frame + sample) + 331 * rnd) / (aten::CMJ::CMJ_DIM * aten::CMJ::CMJ_DIM));
    paths->sampler[idx].init(
        (frame + sample) % (aten::CMJ::CMJ_DIM * aten::CMJ::CMJ_DIM),
        4 + bounce * 300,
        scramble);
#elif IDATEN_SAMPLER == IDATEN_SAMPLER_BLUENOISE
    // Not need to do.
#endif

    aten::hitrecord rec;

    const auto& isect = isects[idx];

    auto obj = &ctxt.shapes[isect.objid];
    evalHitResult(&ctxt, obj, ray, &rec, &isect);

    bool isBackfacing = dot(rec.normal, -ray.dir) < 0.0f;

    // 交差位置の法線.
    // 物体からのレイの入出を考慮.
    aten::vec3 orienting_normal = rec.normal;

    if (rec.mtrlid >= 0) {
        shMtrls[threadIdx.x] = ctxt.mtrls[rec.mtrlid];

#if 1
        if (rec.isVoxel)
        {
            // Replace to lambert.
            const auto& albedo = ctxt.mtrls[rec.mtrlid].baseColor;
            shMtrls[threadIdx.x] = aten::MaterialParameter(aten::MaterialType::Lambert, MaterialAttributeLambert);
            shMtrls[threadIdx.x].baseColor = albedo;
        }
#endif

        if (shMtrls[threadIdx.x].type != aten::MaterialType::Layer) {
            shMtrls[threadIdx.x].albedoMap = (int)(shMtrls[threadIdx.x].albedoMap >= 0 ? ctxt.textures[shMtrls[threadIdx.x].albedoMap] : -1);
            shMtrls[threadIdx.x].normalMap = (int)(shMtrls[threadIdx.x].normalMap >= 0 ? ctxt.textures[shMtrls[threadIdx.x].normalMap] : -1);
            shMtrls[threadIdx.x].roughnessMap = (int)(shMtrls[threadIdx.x].roughnessMap >= 0 ? ctxt.textures[shMtrls[threadIdx.x].roughnessMap] : -1);
        }
    }
    else {
        // TODO
        shMtrls[threadIdx.x] = aten::MaterialParameter(aten::MaterialType::Lambert, MaterialAttributeLambert);
        shMtrls[threadIdx.x].baseColor = aten::vec3(1.0f);
    }


    // Render AOVs.
    // NOTE
    // 厳密に法線をAOVに保持するなら、法線マップ適用後するべき.
    // しかし、temporal reprojection、atrousなどのフィルタ適用時に法線を参照する際に、法線マップが細かすぎてはじかれてしまうことがある.
    // それにより、フィルタがおもったようにかからずフィルタの品質が下がってしまう問題が発生する.
    if (bounce == 0) {
        int ix = idx % tileDomain.w;
        int iy = idx / tileDomain.w;

        ix += tileDomain.x;
        iy += tileDomain.y;

        const auto _idx = getIdx(ix, iy, width);

        // World coordinate to Clip coordinate.
        aten::vec4 pos = aten::vec4(rec.p, 1);
        pos = mtxW2C.apply(pos);

        // normal, depth
        aovNormalDepth[_idx] = make_float4(orienting_normal.x, orienting_normal.y, orienting_normal.z, pos.w);

        // texture color, meshid.
        auto texcolor = AT_NAME::sampleTexture(shMtrls[threadIdx.x].albedoMap, rec.u, rec.v, aten::vec3(1.0f));
#if 0
        aovTexclrMeshid[_idx] = make_float4(texcolor.x, texcolor.y, texcolor.z, isect.meshid);
#else
        aovTexclrMeshid[_idx] = make_float4(texcolor.x, texcolor.y, texcolor.z, isect.mtrlid);
#endif

        // For exporting separated albedo.
        shMtrls[threadIdx.x].albedoMap = -1;
    }
    // TODO
    // How to deal Refraction?
    else if (bounce == 1 && paths->attrib[idx].mtrlType == aten::MaterialType::Specular) {
        int ix = idx % tileDomain.w;
        int iy = idx / tileDomain.w;

        ix += tileDomain.x;
        iy += tileDomain.y;

        const auto _idx = getIdx(ix, iy, width);

        // World coordinate to Clip coordinate.
        aten::vec4 pos = aten::vec4(rec.p, 1);
        pos = mtxW2C.apply(pos);

        // normal, depth
        aovNormalDepth[_idx] = make_float4(orienting_normal.x, orienting_normal.y, orienting_normal.z, pos.w);

        // texture color.
        auto texcolor = AT_NAME::sampleTexture(shMtrls[threadIdx.x].albedoMap, rec.u, rec.v, aten::vec3(1.0f));
#if 0
        aovTexclrMeshid[_idx] = make_float4(texcolor.x, texcolor.y, texcolor.z, isect.meshid);
#else
        aovTexclrMeshid[_idx] = make_float4(texcolor.x, texcolor.y, texcolor.z, isect.mtrlid);
#endif

        // For exporting separated albedo.
        shMtrls[threadIdx.x].albedoMap = -1;
    }

    // Implicit conection to light.
    if (shMtrls[threadIdx.x].attrib.isEmissive) {
        if (!isBackfacing) {
            float weight = 1.0f;

            if (bounce > 0 && !paths->attrib[idx].isSingular) {
                auto cosLight = dot(orienting_normal, -ray.dir);
                auto dist2 = aten::squared_length(rec.p - ray.org);

                if (cosLight >= 0) {
                    auto pdfLight = 1 / rec.area;

                    // Convert pdf area to sradian.
                    // http://www.slideshare.net/h013/edubpt-v100
                    // p31 - p35
                    pdfLight = pdfLight * dist2 / cosLight;

                    weight = paths->throughput[idx].pdfb / (pdfLight + paths->throughput[idx].pdfb);
                }
            }

            auto contrib = paths->throughput[idx].throughput * weight * shMtrls[threadIdx.x].baseColor;
            paths->contrib[idx].contrib += make_float3(contrib.x, contrib.y, contrib.z);
        }

        // When ray hit the light, tracing will finish.
        paths->attrib[idx].isTerminate = true;
        return;
    }

    if (!shMtrls[threadIdx.x].attrib.isTranslucent && isBackfacing) {
        orienting_normal = -orienting_normal;
    }

    // Apply normal map.
    int normalMap = shMtrls[threadIdx.x].normalMap;
    if (shMtrls[threadIdx.x].type == aten::MaterialType::Layer) {
        // 最表層の NormalMap を適用.
        auto* topmtrl = &ctxt.mtrls[shMtrls[threadIdx.x].layer[0]];
        normalMap = (int)(topmtrl->normalMap >= 0 ? ctxt.textures[topmtrl->normalMap] : -1);
    }
    AT_NAME::applyNormalMap(normalMap, orienting_normal, orienting_normal, rec.u, rec.v);

    auto albedo = AT_NAME::sampleTexture(shMtrls[threadIdx.x].albedoMap, rec.u, rec.v, aten::vec3(1), bounce);

#if 1
#pragma unroll
    for (int i = 0; i < idaten::SVGFPathTracing::ShadowRayNum; i++) {
        shShadowRays[threadIdx.x * idaten::SVGFPathTracing::ShadowRayNum + i].isActive = false;
    }

    // Explicit conection to light.
    if (!(shMtrls[threadIdx.x].attrib.isSingular || shMtrls[threadIdx.x].attrib.isTranslucent))
    {
        auto shadowRayOrg = rec.p + AT_MATH_EPSILON * orienting_normal;

        for (int i = 0; i < idaten::SVGFPathTracing::ShadowRayNum; i++) {
            real lightSelectPdf = 1;
            aten::LightSampleResult sampleres;

            // TODO
            // Importance sampling.
            int lightidx = aten::cmpMin<int>(paths->sampler[idx].nextSample() * lightnum, lightnum - 1);
            lightSelectPdf = 1.0f / lightnum;

            aten::LightParameter light;
            light.pos = ((aten::vec4*)ctxt.lights)[lightidx * aten::LightParameter_float4_size + 0];
            light.dir = ((aten::vec4*)ctxt.lights)[lightidx * aten::LightParameter_float4_size + 1];
            light.le = ((aten::vec4*)ctxt.lights)[lightidx * aten::LightParameter_float4_size + 2];
            light.v0 = ((aten::vec4*)ctxt.lights)[lightidx * aten::LightParameter_float4_size + 3];
            light.v1 = ((aten::vec4*)ctxt.lights)[lightidx * aten::LightParameter_float4_size + 4];
            light.v2 = ((aten::vec4*)ctxt.lights)[lightidx * aten::LightParameter_float4_size + 5];
            //auto light = ctxt.lights[lightidx];

            sampleLight(&sampleres, &ctxt, &light, rec.p, orienting_normal, &paths->sampler[idx], bounce);

            const auto& posLight = sampleres.pos;
            const auto& nmlLight = sampleres.nml;
            real pdfLight = sampleres.pdf;

            auto dirToLight = normalize(sampleres.dir);
            auto distToLight = length(posLight - rec.p);

            auto tmp = rec.p + dirToLight - shadowRayOrg;
            auto shadowRayDir = normalize(tmp);

            shShadowRays[threadIdx.x * idaten::SVGFPathTracing::ShadowRayNum + i].isActive = true;
            shShadowRays[threadIdx.x * idaten::SVGFPathTracing::ShadowRayNum + i].rayorg = shadowRayOrg;
            shShadowRays[threadIdx.x * idaten::SVGFPathTracing::ShadowRayNum + i].raydir = shadowRayDir;
            shShadowRays[threadIdx.x * idaten::SVGFPathTracing::ShadowRayNum + i].targetLightId = lightidx;
            shShadowRays[threadIdx.x * idaten::SVGFPathTracing::ShadowRayNum + i].distToLight = distToLight;
            shShadowRays[threadIdx.x * idaten::SVGFPathTracing::ShadowRayNum + i].lightcontrib = aten::vec3(0);
            {
                auto cosShadow = dot(orienting_normal, dirToLight);

                real pdfb = samplePDF(&ctxt, &shMtrls[threadIdx.x], orienting_normal, ray.dir, dirToLight, rec.u, rec.v);
                auto bsdf = sampleBSDF(&ctxt, &shMtrls[threadIdx.x], orienting_normal, ray.dir, dirToLight, rec.u, rec.v, albedo);

                bsdf *= paths->throughput[idx].throughput;

                // Get light color.
                auto emit = sampleres.finalColor;

                if (light.attrib.isSingular || light.attrib.isInfinite) {
                    if (pdfLight > real(0) && cosShadow >= 0) {
                        // TODO
                        // ジオメトリタームの扱いについて.
                        // singular light の場合は、finalColor に距離の除算が含まれている.
                        // inifinite light の場合は、無限遠方になり、pdfLightに含まれる距離成分と打ち消しあう？.
                        // （打ち消しあうので、pdfLightには距離成分は含んでいない）.
                        auto misW = pdfLight / (pdfb + pdfLight);

                        shShadowRays[threadIdx.x * idaten::SVGFPathTracing::ShadowRayNum + i].lightcontrib =
                            (misW * bsdf * emit * cosShadow / pdfLight) / lightSelectPdf / (float)idaten::SVGFPathTracing::ShadowRayNum;
                    }
                }
                else {
                    auto cosLight = dot(nmlLight, -dirToLight);

                    if (cosShadow >= 0 && cosLight >= 0) {
                        auto dist2 = aten::squared_length(sampleres.dir);
                        auto G = cosShadow * cosLight / dist2;

                        if (pdfb > real(0) && pdfLight > real(0)) {
                            // Convert pdf from steradian to area.
                            // http://www.slideshare.net/h013/edubpt-v100
                            // p31 - p35
                            pdfb = pdfb * cosLight / dist2;

                            auto misW = pdfLight / (pdfb + pdfLight);

                            shShadowRays[threadIdx.x * idaten::SVGFPathTracing::ShadowRayNum + i].lightcontrib =
                                (misW * (bsdf * emit * G) / pdfLight) / lightSelectPdf / (float)idaten::SVGFPathTracing::ShadowRayNum;;
                        }
                    }
                }
            }
        }
    }
#endif

    real russianProb = real(1);

    if (bounce > rrBounce) {
        auto t = normalize(paths->throughput[idx].throughput);
        auto p = aten::cmpMax(t.r, aten::cmpMax(t.g, t.b));

        russianProb = paths->sampler[idx].nextSample();

        if (russianProb >= p) {
            //shPaths[threadIdx.x].contrib = aten::vec3(0);
            paths->attrib[idx].isTerminate = true;
        }
        else {
            russianProb = max(p, 0.01f);
        }
    }

    AT_NAME::MaterialSampling sampling;

    sampleMaterial(
        &sampling,
        &ctxt,
        &shMtrls[threadIdx.x],
        orienting_normal,
        ray.dir,
        rec.normal,
        &paths->sampler[idx],
        rec.u, rec.v,
        albedo);

    auto nextDir = normalize(sampling.dir);
    auto pdfb = sampling.pdf;
    auto bsdf = sampling.bsdf;

    real c = 1;
    if (!shMtrls[threadIdx.x].attrib.isSingular) {
        // TODO
        // AMDのはabsしているが....
        c = aten::abs(dot(orienting_normal, nextDir));
        //c = dot(orienting_normal, nextDir);
    }

    if (pdfb > 0 && c > 0) {
        paths->throughput[idx].throughput *= bsdf * c / pdfb;
        paths->throughput[idx].throughput /= russianProb;
    }
    else {
        paths->attrib[idx].isTerminate = true;
    }

    // Make next ray.
    rays[idx] = aten::ray(rec.p, nextDir);

    paths->throughput[idx].pdfb = pdfb;
    paths->attrib[idx].isSingular = shMtrls[threadIdx.x].attrib.isSingular;
    paths->attrib[idx].mtrlType = shMtrls[threadIdx.x].type;

#pragma unroll
    for (int i = 0; i < idaten::SVGFPathTracing::ShadowRayNum; i++) {
        shadowRays[idx * idaten::SVGFPathTracing::ShadowRayNum + i] = shShadowRays[threadIdx.x * idaten::SVGFPathTracing::ShadowRayNum + i];
    }
}

namespace idaten
{
    void AdvancedSVGFPathTracing::onShade(
        hipSurfaceObject_t outputSurf,
        int width, int height,
        int sample,
        int bounce, int rrBounce,
        hipTextureObject_t texVtxPos,
        hipTextureObject_t texVtxNml)
    {
        m_mtxW2V.lookat(
            m_camParam.origin,
            m_camParam.center,
            m_camParam.up);

        m_mtxV2C.perspective(
            m_camParam.znear,
            m_camParam.zfar,
            m_camParam.vfov,
            m_camParam.aspect);

        m_mtxC2V = m_mtxV2C;
        m_mtxC2V.invert();

        m_mtxV2W = m_mtxW2V;
        m_mtxV2W.invert();

        aten::mat4 mtxW2C = m_mtxV2C * m_mtxW2V;

        dim3 blockPerGrid(((m_tileDomain.w * m_tileDomain.h) + 64 - 1) / 64);
        dim3 threadPerBlock(64);

        auto& hitcount = m_compaction.getCount();

        int curaov = getCurAovs();

        auto blueNoise = m_bluenoise.bind();

        shadeASVGF << <blockPerGrid, threadPerBlock, 0, m_stream >> > (
            m_tileDomain,
            m_aovNormalDepth[curaov].ptr(),
            m_aovTexclrMeshid[curaov].ptr(),
            mtxW2C,
            width, height,
            m_paths.ptr(),
            m_hitidx.ptr(), hitcount.ptr(),
            m_isects.ptr(),
            m_rays.ptr(),
            sample,
            m_frame,
            bounce, rrBounce,
            m_shapeparam.ptr(), m_shapeparam.num(),
            m_mtrlparam.ptr(),
            m_lightparam.ptr(), m_lightparam.num(),
            m_primparams.ptr(),
            texVtxPos, texVtxNml,
            m_mtxparams.ptr(),
            m_tex.ptr(),
            m_random.ptr(),
            blueNoise,
            m_shadowRays.ptr());

        checkCudaKernel(shade);

        onShadeByShadowRay(bounce, texVtxPos);

        m_bluenoise.unbind();
    }
}
