#include "hip/hip_runtime.h"
#include "npr/npr_pathtracing.h"

#include "kernel/accelerator.cuh"
#include "kernel/device_scene_context.cuh"
#include "kernel/intersect.cuh"
#include "kernel/pt_common.h"

#include "cuda/cudadefs.h"
#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

#include "renderer/pathtracing/pt_params.h"
#include "renderer/npr/npr_impl.h"

namespace npr_kernel {
    __global__ void GenerateSampleRay(
        idaten::NPRPathTracing::SampleRayInfo* sample_ray_infos,
        idaten::Path paths,
        const aten::ray* __restrict__ rays,
        const int32_t* __restrict__ hitindices,
        int32_t* hitnum,
        real feature_line_width,
        real pixel_width)
    {
        int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx >= *hitnum) {
            return;
        }

        idx = hitindices[idx];

        auto& sample_ray_info = sample_ray_infos[idx];
        const auto& ray = rays[idx];

        AT_NAME::npr::GenerateSampleRayAndDiscPerQueryRay<idaten::NPRPathTracing::SampleRayNum>(
            sample_ray_info.descs, sample_ray_info.disc,
            ray, paths.sampler[idx],
            feature_line_width, pixel_width);
    }

    __global__ void shadeSampleRay(
        aten::vec3 line_color,  // TODO
        real feature_line_width,
        real pixel_width,
        idaten::NPRPathTracing::SampleRayInfo* sample_ray_infos,
        int32_t depth,
        const int32_t* __restrict__ hitindices,
        int32_t* hitnum,
        idaten::Path paths,
        const aten::CameraParameter camera,
        const aten::Intersection* __restrict__ isects,
        const aten::ray* __restrict__ rays,
        idaten::context ctxt,
        const aten::ObjectParameter* __restrict__ shapes,
        const aten::MaterialParameter* __restrict__ mtrls,
        const aten::LightParameter* __restrict__ lights,
        const aten::TriangleParameter* __restrict__ prims,
        const aten::mat4* __restrict__ matrices)
    {
        int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;

        if (idx >= *hitnum) {
            return;
        }

        idx = hitindices[idx];

        if (paths.attrib[idx].isKill || paths.attrib[idx].isTerminate) {
            paths.attrib[idx].isTerminate = true;
            return;
        }

        ctxt.shapes = shapes;
        ctxt.mtrls = mtrls;
        ctxt.lights = lights;
        ctxt.prims = prims;
        ctxt.matrices = matrices;

        aten::hitrecord hrec_query;

        const auto& isect = isects[idx];

        const auto& query_ray = rays[idx];
        auto* sampler = &paths.sampler[idx];

        const auto& cam_org = camera.origin;

        constexpr auto SampleRayNum = aten::array_size<decltype(idaten::NPRPathTracing::SampleRayInfo::descs)>::size;

        // TODO: These value should be configurable.
        constexpr real albedo_threshold = 0.1f;
        constexpr real normal_threshold = 0.1f;

        auto& sample_ray_info = sample_ray_infos[idx];
        auto& sample_ray_descs = sample_ray_info.descs;
        auto& disc = sample_ray_info.disc;

        // Current closest distance to feature line point.
        auto closest_feature_line_point_distance = std::numeric_limits<real>::max();

        // Whether the feature line point has been found.
        bool is_found_feature_line_point = false;

        real hit_point_distance = 0;

        const auto& obj = ctxt.GetObject(static_cast<uint32_t>(isect.objid));
        AT_NAME::evaluate_hit_result(hrec_query, obj, ctxt, query_ray, isect);

        const auto distance_query_ray_hit = length(hrec_query.p - query_ray.org);

        // disc.center��query_ray.org�Ɉ�v����.
        // �������A�ŏ������́Aquery_ray.org�̓J�������_�ɂȂ��Ă��邪�A
        // accumulated_distance�ŃJ������disc�̋��������łɊ܂܂�Ă���.
        hit_point_distance = length(hrec_query.p - disc.center);

        const auto prev_disc = disc;
        disc = AT_NAME::npr::FeatureLine::ComputeDiscAtQueryRayHitPoint(
            hrec_query.p,
            query_ray.dir,
            prev_disc.radius,
            hit_point_distance,
            disc.accumulated_distance);

        for (size_t i = 0; i < SampleRayNum; i++) {
            if (sample_ray_info.descs[i].is_terminated) {
                continue;
            }

            auto sample_ray = AT_NAME::npr::GetSampleRay(
                depth,
                sample_ray_descs[i],
                prev_disc, disc);
            if (sample_ray_descs[i].is_terminated) {
                continue;
            }

            aten::Intersection isect_sample_ray;

            auto is_hit = intersectClosest(&ctxt, sample_ray, &isect_sample_ray);
            if (is_hit) {
                // Query ray hits and then sample ray hits.
                aten::tie(is_found_feature_line_point, closest_feature_line_point_distance) = AT_NAME::npr::EvaluateQueryAndSampleRayHit(
                    sample_ray_descs[i],
                    ctxt, cam_org,
                    query_ray, hrec_query, distance_query_ray_hit,
                    isect_sample_ray,
                    disc,
                    is_found_feature_line_point,
                    closest_feature_line_point_distance,
                    feature_line_width, pixel_width,
                    albedo_threshold, normal_threshold);
            }
            else {
                // Query ray hits but sample ray doesn't hit anything.
                aten::tie(is_found_feature_line_point, closest_feature_line_point_distance) = AT_NAME::npr::EvaluateQueryRayHitButSampleRayNotHit(
                    sample_ray_descs[i],
                    query_ray, hrec_query, distance_query_ray_hit,
                    sample_ray, disc,
                    is_found_feature_line_point,
                    closest_feature_line_point_distance,
                    feature_line_width, pixel_width);
            }

            const auto mtrl = ctxt.GetMaterial(hrec_query.mtrlid);
            if (!mtrl.attrib.isGlossy) {
                // In non glossy material case, sample ray doesn't bounce anymore.
                // TODO
                // Even if material is glossy, how glossy depends on parameter (e.g. roughness etc).
                // So, I need to consider how to indetify if sample ray bounce is necessary based on material.
                sample_ray_descs[i].is_terminated = true;
            }
        }

        if (is_found_feature_line_point) {
            AT_NAME::npr::ComputeFeatureLineContribution<SampleRayNum>(
                closest_feature_line_point_distance, paths, idx, line_color);
        }

        disc.accumulated_distance += hit_point_distance;
    }

    __global__ void shadeMissSampleRay(
        int32_t width, int32_t height,
        aten::vec3 line_color,  // TODO
        real feature_line_width,
        real pixel_width,
        idaten::NPRPathTracing::SampleRayInfo* sample_ray_infos,
        int32_t depth,
        const int32_t* __restrict__ hitindices,
        int32_t* hitnum,
        idaten::Path paths,
        const aten::CameraParameter camera,
        const aten::Intersection* __restrict__ isects,
        const aten::ray* __restrict__ rays,
        idaten::context ctxt,
        const aten::ObjectParameter* __restrict__ shapes,
        const aten::MaterialParameter* __restrict__ mtrls,
        const aten::LightParameter* __restrict__ lights,
        const aten::TriangleParameter* __restrict__ prims,
        const aten::mat4* __restrict__ matrices)
    {
        auto ix = blockIdx.x * blockDim.x + threadIdx.x;
        auto iy = blockIdx.y * blockDim.y + threadIdx.y;

        if (ix >= width || iy >= height) {
            return;
        }

        const auto idx = getIdx(ix, iy, width);

        if (paths.attrib[idx].isTerminate || paths.attrib[idx].isHit) {
            return;
        }

        ctxt.shapes = shapes;
        ctxt.mtrls = mtrls;
        ctxt.lights = lights;
        ctxt.prims = prims;
        ctxt.matrices = matrices;

        // Query ray doesn't hit anything, but evaluate a possibility that sample ray might hit something.

        const auto& isect = isects[idx];

        const auto& query_ray = rays[idx];
        auto* sampler = &paths.sampler[idx];

        const auto& cam_org = camera.origin;

        constexpr auto SampleRayNum = aten::array_size<decltype(idaten::NPRPathTracing::SampleRayInfo::descs)>::size;

        // TODO: These value should be configurable.
        constexpr real albedo_threshold = 0.1f;
        constexpr real normal_threshold = 0.1f;

        auto& sample_ray_info = sample_ray_infos[idx];
        auto& sample_ray_descs = sample_ray_info.descs;
        auto& disc = sample_ray_info.disc;

        auto closest_feature_line_point_distance = std::numeric_limits<real>::max();
        bool is_found_feature_line_point = false;
        real hit_point_distance = 0;

        // NOTE:
        // In order to compute sample ray, previous disc and next disc are necessary.
        // In first bounce, initial point is camera original.
        // So, previous disc is not necessary.

        AT_NAME::npr::FeatureLine::Disc prev_disc;
        hit_point_distance = CreateNextDiscByDummyQueryRayHitPoint(depth, hit_point_distance, query_ray, prev_disc, disc);

        for (size_t i = 0; i < SampleRayNum; i++) {
            if (sample_ray_descs[i].is_terminated) {
                continue;
            }

            auto sample_ray = AT_NAME::npr::GetSampleRay(
                depth,
                sample_ray_descs[i],
                prev_disc, disc);
            if (sample_ray_descs[i].is_terminated) {
                continue;
            }

            aten::Intersection isect_sample_ray;

            auto is_hit = intersectClosest(&ctxt, sample_ray, &isect_sample_ray);
            if (is_hit) {
                // Query ray doesn't hit, but sample ray hits.
                aten::tie(is_found_feature_line_point, closest_feature_line_point_distance) = AT_NAME::npr::EvaluateQueryRayNotHitButSampleRayHit(
                    ctxt, query_ray,
                    isect_sample_ray,
                    disc,
                    is_found_feature_line_point,
                    closest_feature_line_point_distance,
                    feature_line_width, pixel_width);
            }
            else {
                // Sample ray doesn't hit anything. It means sample ray causes hit miss.
                // So, traversing sample ray is terminated.
                sample_ray_descs[i].is_terminated = true;
                break;
            }
        }

        if (is_found_feature_line_point) {
            AT_NAME::npr::ComputeFeatureLineContribution<SampleRayNum>(
                closest_feature_line_point_distance, paths, idx, line_color);
        }
    }
}

namespace idaten {
    void NPRPathTracing::onShade(
        hipSurfaceObject_t outputSurf,
        int32_t width, int32_t height,
        int32_t sample,
        int32_t bounce, int32_t rrBounce)
    {
        if (is_enable_feature_line_) {
            dim3 blockPerGrid(((width * height) + 64 - 1) / 64);
            dim3 threadPerBlock(64);

            if (sample_ray_infos_.empty()) {
                sample_ray_infos_.resize(width * height);
            }

            auto& hitcount = m_compaction.getCount();

            const auto pixel_width = AT_NAME::camera::computePixelWidthAtDistance(m_cam, 1);

            if (bounce == 0) {
                npr_kernel::GenerateSampleRay << <blockPerGrid, threadPerBlock, 0, m_stream >> > (
                    sample_ray_infos_.data(),
                    path_host_->paths,
                    m_rays.data(),
                    m_hitidx.data(),
                    hitcount.data(),
                    feature_line_width_,
                    pixel_width);
                checkCudaKernel(GenerateSampleRay);
            }

            npr_kernel::shadeSampleRay << <blockPerGrid, threadPerBlock, 0, m_stream >> > (
                aten::vec3(0, 1, 0),
                feature_line_width_,
                pixel_width,
                sample_ray_infos_.data(),
                bounce,
                m_hitidx.data(),
                hitcount.data(),
                path_host_->paths,
                m_cam,
                m_isects.data(),
                m_rays.data(),
                ctxt_host_.ctxt,
                ctxt_host_.shapeparam.data(),
                ctxt_host_.mtrlparam.data(),
                ctxt_host_.lightparam.data(),
                ctxt_host_.primparams.data(),
                ctxt_host_.mtxparams.data());
            checkCudaKernel(shadeSampleRay);
        }

        PathTracing::onShade(
            outputSurf,
            width, height,
            sample,
            bounce, rrBounce);
    }

    void NPRPathTracing::missShade(
        int32_t width, int32_t height,
        int32_t bounce)
    {
        if (is_enable_feature_line_) {
            dim3 block(BLOCK_SIZE, BLOCK_SIZE);
            dim3 grid(
                (width + block.x - 1) / block.x,
                (height + block.y - 1) / block.y);

            if (sample_ray_infos_.empty()) {
                sample_ray_infos_.resize(width * height);
            }

            auto& hitcount = m_compaction.getCount();

            const auto pixel_width = AT_NAME::camera::computePixelWidthAtDistance(m_cam, 1);

            // Sample ray hit miss never happen at 1st bounce.
            npr_kernel::shadeMissSampleRay << <grid, block, 0, m_stream >> > (
                width, height,
                aten::vec3(0, 1, 0),
                feature_line_width_,
                pixel_width,
                sample_ray_infos_.data(),
                bounce,
                m_hitidx.data(),
                hitcount.data(),
                path_host_->paths,
                m_cam,
                m_isects.data(),
                m_rays.data(),
                ctxt_host_.ctxt,
                ctxt_host_.shapeparam.data(),
                ctxt_host_.mtrlparam.data(),
                ctxt_host_.lightparam.data(),
                ctxt_host_.primparams.data(),
                ctxt_host_.mtxparams.data());
            checkCudaKernel(shadeMissSampleRay);
        }

        PathTracing::missShade(width, height, bounce);
    }
}
