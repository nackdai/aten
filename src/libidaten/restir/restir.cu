#include "hip/hip_runtime.h"
#include <utility>

#include "restir/restir.h"
#include "restir/restir_sample_light.cuh"

#include "aten4idaten.h"
#include "kernel/accelerator.cuh"
#include "kernel/device_scene_context.cuh"
#include "kernel/intersect.cuh"
#include "kernel/pt_common.h"

#include "cuda/cudadefs.h"
#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

#include "renderer/pathtracing/pathtracing_impl.h"

__global__ void initReSTIRParameters(
    int32_t width, int32_t height,
    idaten::Reservoir* reservoirs,
    idaten::ReSTIRInfo* restir_infos)
{
    auto ix = blockIdx.x * blockDim.x + threadIdx.x;
    auto iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix >= width || iy >= height) {
        return;
    }

    const auto idx = getIdx(ix, iy, width);

    reservoirs[idx].clear();
    restir_infos[idx].clear();
}

__global__ void shade(
    idaten::Reservoir* reservoirs,
    idaten::ReSTIRInfo* restir_infos,
    float4* aovNormalDepth,
    float4* aovTexclrMeshid,
    aten::mat4 mtxW2C,
    int32_t width, int32_t height,
    idaten::Path paths,
    const int32_t* __restrict__ hitindices,
    int32_t* hitnum,
    const aten::Intersection* __restrict__ isects,
    aten::ray* rays,
    int32_t sample,
    int32_t frame,
    int32_t bounce, int32_t rrBounce,
    idaten::context ctxt,
    uint32_t* random,
    idaten::ShadowRay* shadowRays)
{
    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= *hitnum) {
        return;
    }

    idx = hitindices[idx];

    __shared__ aten::MaterialParameter shMtrls[64];

    const auto ray = rays[idx];

#if IDATEN_SAMPLER == IDATEN_SAMPLER_SOBOL
    auto scramble = random[idx] * 0x1fe3434f;
    paths.sampler[idx].init(frame + sample, 4 + bounce * 300, scramble);
#elif IDATEN_SAMPLER == IDATEN_SAMPLER_CMJ
    auto rnd = random[idx];
    auto scramble = rnd * 0x1fe3434f
        * (((frame + sample) + 331 * rnd) / (aten::CMJ::CMJ_DIM * aten::CMJ::CMJ_DIM));
    paths.sampler[idx].init(
        (frame + sample) % (aten::CMJ::CMJ_DIM * aten::CMJ::CMJ_DIM),
        4 + bounce * 300,
        scramble);
#endif

    aten::hitrecord rec;

    const auto& isect = isects[idx];

    const auto& obj = ctxt.GetObject(static_cast<uint32_t>(isect.objid));
    AT_NAME::evaluate_hit_result(rec, obj, ctxt, ray, isect);

    bool isBackfacing = dot(rec.normal, -ray.dir) < 0.0f;

    // 交差位置の法線.
    // 物体からのレイの入出を考慮.
    aten::vec3 orienting_normal = rec.normal;

    AT_NAME::FillMaterial(
        shMtrls[threadIdx.x],
        ctxt,
        rec.mtrlid,
        rec.isVoxel);

    auto albedo = AT_NAME::sampleTexture(shMtrls[threadIdx.x].albedoMap, rec.u, rec.v, aten::vec4(1), bounce);

    // Apply normal map.
    int32_t normalMap = shMtrls[threadIdx.x].normalMap;
    const auto pre_sampled_r = AT_NAME::material::applyNormal(
        &shMtrls[threadIdx.x],
        normalMap,
        orienting_normal, orienting_normal,
        rec.u, rec.v,
        ray.dir,
        &paths.sampler[idx]);

    if (!shMtrls[threadIdx.x].attrib.isTranslucent
        && !shMtrls[threadIdx.x].attrib.isEmissive
        && isBackfacing)
    {
        orienting_normal = -orienting_normal;
    }

    shadowRays[idx].isActive = false;

    auto& restir_info = restir_infos[idx];
    {
        restir_info.clear();
        restir_info.nml = orienting_normal;
        restir_info.is_voxel = rec.isVoxel;
        restir_info.mtrl_idx = rec.mtrlid;
        restir_info.throughput = paths.throughput[idx].throughput;
        restir_info.wi = ray.dir;
        restir_info.u = rec.u;
        restir_info.v = rec.v;
        restir_info.p = rec.p;
        restir_info.pre_sampled_r = pre_sampled_r;
    }

    if (bounce == 0) {
        // Store AOV.
        int32_t ix = idx % width;
        int32_t iy = idx / width;

        const auto _idx = getIdx(ix, iy, width);

        // World coordinate to Clip coordinate.
        aten::vec4 pos = aten::vec4(rec.p, 1);
        pos = mtxW2C.apply(pos);

        aovNormalDepth[_idx] = make_float4(orienting_normal.x, orienting_normal.y, orienting_normal.z, pos.w);
        aovTexclrMeshid[_idx] = make_float4(albedo.x, albedo.y, albedo.z, isect.mtrlid);
    }

    // Implicit conection to light.
    auto is_hit_implicit_light = AT_NAME::HitImplicitLight(
        isBackfacing,
        bounce,
        paths.contrib[idx], paths.attrib[idx], paths.throughput[idx],
        ray,
        rec.p, orienting_normal,
        rec.area,
        shMtrls[threadIdx.x]);
    if (is_hit_implicit_light) {
        return;
    }

    // Explicit conection to light.
    if (!(shMtrls[threadIdx.x].attrib.isSingular || shMtrls[threadIdx.x].attrib.isTranslucent))
    {
        auto& reservoir = reservoirs[idx];

        auto lightidx = sampleLightWithReservoirRIP(
            reservoir,
            shMtrls[threadIdx.x],
            &ctxt,
            rec.p, orienting_normal,
            ray.dir,
            rec.u, rec.v, albedo,
            &paths.sampler[idx],
            bounce);

        if (lightidx >= 0) {
            const auto& light = ctxt.lights[lightidx];

            const auto& posLight = reservoir.light_sample_.pos;
            const auto& nmlLight = reservoir.light_sample_.nml;

            auto lightSelectPdf = reservoir.pdf_;

            auto dirToLight = normalize(reservoir.light_sample_.dir);
            auto distToLight = length(posLight - rec.p);

            aten::Intersection isectTmp;

            auto shadowRayOrg = rec.p + AT_MATH_EPSILON * orienting_normal;
            auto tmp = rec.p + dirToLight - shadowRayOrg;
            auto shadowRayDir = normalize(tmp);

            bool isShadowRayActive = false;

            shadowRays[idx].rayorg = shadowRayOrg;
            shadowRays[idx].raydir = shadowRayDir;
            shadowRays[idx].targetLightId = lightidx;
            shadowRays[idx].distToLight = distToLight;
            shadowRays[idx].lightcontrib = aten::vec3(0);
            {
                auto cosShadow = dot(orienting_normal, dirToLight);
                cosShadow = aten::abs(cosShadow);

                if (light.attrib.isInfinite || light.attrib.isSingular) {
                    if (cosShadow >= 0) {
                        isShadowRayActive = true;
                    }
                }
                else {
                    auto cosLight = dot(nmlLight, -dirToLight);

                    if (cosShadow >= 0 && cosLight >= 0) {
                        isShadowRayActive = true;
                    }
                }

                shadowRays[idx].isActive = isShadowRayActive;
            }
        }
    }

    const auto russianProb = AT_NAME::ComputeRussianProbability(
        bounce, rrBounce,
        paths.attrib[idx],
        paths.throughput[idx],
        paths.sampler[idx]);

    AT_NAME::MaterialSampling sampling;

    AT_NAME::material::sampleMaterialWithExternalAlbedo(
        &sampling,
        &shMtrls[threadIdx.x],
        orienting_normal,
        ray.dir,
        rec.normal,
        &paths.sampler[idx],
        pre_sampled_r,
        rec.u, rec.v,
        albedo);

    auto nextDir = normalize(sampling.dir);
    auto pdfb = sampling.pdf;
    auto bsdf = sampling.bsdf;

    // Get normal to add ray offset.
    // In refraction material case, new ray direction might be computed with inverted normal.
    // For example, when a ray go into the refraction surface, inverted normal is used to compute new ray direction.
    auto rayBasedNormal = (!isBackfacing && shMtrls[threadIdx.x].attrib.isTranslucent)
        ? -orienting_normal
        : orienting_normal;

    auto c = dot(orienting_normal, nextDir);

    if (pdfb > 0 && c > 0) {
        paths.throughput[idx].throughput *= bsdf * c / pdfb;
        paths.throughput[idx].throughput /= russianProb;
    }
    else {
        paths.attrib[idx].isTerminate = true;
        return;
    }

    // Make next ray.
    rays[idx] = aten::ray(rec.p, nextDir, rayBasedNormal);

    paths.throughput[idx].pdfb = pdfb;
    paths.attrib[idx].isSingular = shMtrls[threadIdx.x].attrib.isSingular;
    paths.attrib[idx].mtrlType = shMtrls[threadIdx.x].type;
}

__global__ void hitShadowRay(
    int32_t bounce,
    idaten::Path paths,
    int32_t* hitindices,
    int32_t* hitnum,
    idaten::Reservoir* reservoirs,
    const idaten::ShadowRay* __restrict__ shadowRays,
    idaten::context ctxt)
{
    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= *hitnum) {
        return;
    }

    idx = hitindices[idx];

    auto isHit = AT_NAME::HitShadowRay(idx, bounce, ctxt, paths, shadowRays);

    if (!isHit) {
        reservoirs[idx].w_sum_ = 0.0f;
        reservoirs[idx].pdf_ = 0.0f;
        reservoirs[idx].target_density_ = 0.0f;
        reservoirs[idx].light_idx_ = -1;
    }
}

__global__ void computeShadowRayContribution(
    const idaten::Reservoir* __restrict__ reservoirs,
    const idaten::ReSTIRInfo* __restrict__ restir_infos,
    idaten::Path paths,
    int32_t* hitindices,
    int32_t* hitnum,
    const float4* __restrict__ aovNormalDepth,
    const float4* __restrict__ aovTexclrMeshid,
    const aten::LightParameter* __restrict__ lights, int32_t lightnum,
    const aten::MaterialParameter* __restrict__ mtrls,
    hipTextureObject_t* textures,
    const idaten::ShadowRay* __restrict__ shadowRays)
{
    int32_t idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= *hitnum) {
        return;
    }

    idx = hitindices[idx];

    if (paths.attrib[idx].isTerminate) {
        return;
    }

    if (lightnum <= 0) {
        return;
    }

    idaten::context ctxt;
    {
        ctxt.mtrls = mtrls;
        ctxt.textures = textures;
    }

    __shared__ aten::MaterialParameter shMtrls[64];

    const auto& reservoir = reservoirs[idx];
    const auto& restir_info = restir_infos[idx];

    AT_NAME::FillMaterial(
        shMtrls[threadIdx.x],
        ctxt,
        restir_info.mtrl_idx,
        restir_info.is_voxel);

    if (!(shMtrls[threadIdx.x].attrib.isSingular || shMtrls[threadIdx.x].attrib.isTranslucent))
    {
        if (reservoir.isValid()) {
            const auto& orienting_normal = restir_info.nml;

            const auto& albedo_meshid = aovTexclrMeshid[idx];
            const aten::vec4 albedo(albedo_meshid.x, albedo_meshid.y, albedo_meshid.z, 1.0f);

            const auto& light = lights[reservoir.light_idx_];

            const auto& nmlLight = reservoir.light_sample_.nml;
            const auto& dirToLight = shadowRays[idx].raydir;
            const auto& distToLight = shadowRays[idx].distToLight;

            aten::vec3 lightcontrib;
            {
                auto cosShadow = dot(orienting_normal, dirToLight);

                // TODO
                // 計算済みのalbedoを与えているため
                // u,v は samplePDF/sampleBSDF 内部では利用されていない
                float u = 0.0f;
                float v = 0.0f;

                auto bsdf = AT_NAME::material::sampleBSDFWithExternalAlbedo(
                    &shMtrls[threadIdx.x],
                    orienting_normal,
                    restir_info.wi,
                    dirToLight,
                    u, v,
                    albedo,
                    restir_info.pre_sampled_r);

                bsdf *= restir_info.throughput;

                // Get light color.
                auto emit = reservoir.light_sample_.finalColor;

                cosShadow = aten::abs(cosShadow);

                if (light.attrib.isInfinite || light.attrib.isSingular) {
                    if (cosShadow >= 0) {
                        lightcontrib = bsdf * emit * cosShadow * reservoir.pdf_;
                    }
                }
                else {
                    auto cosLight = dot(nmlLight, -dirToLight);

                    if (cosShadow >= 0 && cosLight >= 0) {
                        auto dist2 = distToLight * distToLight;
                        auto G = cosShadow * cosLight / dist2;

                        lightcontrib = (bsdf * emit * G) * reservoir.pdf_;
                    }
                }
            }

            paths.contrib[idx].contrib += make_float3(lightcontrib.x, lightcontrib.y, lightcontrib.z);
        }
    }
}

namespace idaten
{
    void ReSTIRPathTracing::initReSTIR(int32_t width, int32_t height)
    {
        dim3 block(BLOCK_SIZE, BLOCK_SIZE);
        dim3 grid(
            (width + block.x - 1) / block.x,
            (height + block.y - 1) / block.y);

        initReSTIRParameters << < grid, block, 0, m_stream >> > (
            width, height,
            m_reservoirs[m_curReservoirPos].data(),
            m_restir_infos.data());

        checkCudaKernel(initReSTIRParameters);
    }

    void ReSTIRPathTracing::onShadeReSTIR(
        hipSurfaceObject_t outputSurf,
        int32_t width, int32_t height,
        int32_t sample,
        int32_t bounce, int32_t rrBounce)
    {
        m_mtxW2V.lookat(
            m_cam.origin,
            m_cam.center,
            m_cam.up);

        m_mtxV2C.perspective(
            m_cam.znear,
            m_cam.zfar,
            m_cam.vfov,
            m_cam.aspect);

        m_mtxC2V = m_mtxV2C;
        m_mtxC2V.invert();

        m_mtxV2W = m_mtxW2V;
        m_mtxV2W.invert();

        aten::mat4 mtxW2C = m_mtxV2C * m_mtxW2V;

        dim3 blockPerGrid(((width * height) + 64 - 1) / 64);
        dim3 threadPerBlock(64);

        auto& hitcount = m_compaction.getCount();

        shade << <blockPerGrid, threadPerBlock, 0, m_stream >> > (
            m_reservoirs[m_curReservoirPos].data(),
            m_restir_infos.data(),
            aov_.normal_depth().data(),
            aov_.albedo_meshid().data(),
            mtxW2C,
            width, height,
            path_host_->paths,
            m_hitidx.data(), hitcount.data(),
            m_isects.data(),
            m_rays.data(),
            sample,
            m_frame,
            bounce, rrBounce,
            ctxt_host_.ctxt,
            m_random.data(),
            m_shadowRays.data());

        checkCudaKernel(shade);

        onShadeByShadowRayReSTIR(
            width, height,
            bounce);
    }

    void ReSTIRPathTracing::onShadeByShadowRayReSTIR(
        int32_t width, int32_t height,
        int32_t bounce)
    {
        dim3 blockPerGrid(((width * height) + 64 - 1) / 64);
        dim3 threadPerBlock(64);

        auto& hitcount = m_compaction.getCount();

        hitShadowRay << <blockPerGrid, threadPerBlock, 0, m_stream >> > (
            bounce,
            path_host_->paths,
            m_hitidx.data(), hitcount.data(),
            m_reservoirs[m_curReservoirPos].data(),
            m_shadowRays.data(),
            ctxt_host_.ctxt);

        checkCudaKernel(hitShadowRay);

        const auto target_idx = computelReuse(
            width, height,
            bounce);

        computeShadowRayContribution << <blockPerGrid, threadPerBlock, 0, m_stream >> > (
            m_reservoirs[target_idx].data(),
            m_restir_infos.data(),
            path_host_->paths,
            m_hitidx.data(), hitcount.data(),
            aov_.normal_depth().data(),
            aov_.albedo_meshid().data(),
            ctxt_host_.lightparam.data(), ctxt_host_.lightparam.num(),
            ctxt_host_.mtrlparam.data(),
            ctxt_host_.tex.data(),
            m_shadowRays.data());

        checkCudaKernel(computeShadowRayContribution);
    }
}
